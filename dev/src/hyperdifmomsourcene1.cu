#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdmne1.cuh"

__global__ void hyperdifmomsourcene6_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
      if(i<((p->n[0])) && j<((p->n[1])))
     #endif

                        //if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                              - sign here same as vac maybe a +
                             //wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode3_hdmne1(p,iia,mom1+ii0)]; 
                             wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdmne1(p,iia,mom1+ii0)];
                             wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdmne1(p,iia,energy)]; 

                         }
              //  }	
}
  __syncthreads();


  



}



__global__ void hyperdifmomsourcene5_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif

{
 
    dwn1[fencode3_hdmne1(p,iia,mom1+ii0)]=(grad13_hdmne1(wtemp,p,iia,tmp7,ii));


  }
}
 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
      if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif

     dwn1[fencode3_hdmne1(p,iia,energy)]=(grad13_hdmne1(wtemp,p,iia,tmp8,ii));



}
 __syncthreads();


  



}

__global__ void hyperdifmomsourcene4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
	{		               
     wtemp[fencode3_hdmne1(p,iia,tmp7)]=wtemp[fencode3_hdmne1(p,iia,tmp1)]*wtemp[fencode3_hdmne1(p,iia,tmp6)];

     wtemp[fencode3_hdmne1(p,iia,tmp8)]=wtemp[fencode3_hdmne1(p,iia,tmp6)]*wmod[(shift)+fencode3_hdmne1(p,iia,mom1+ii0)];





   }
}
 __syncthreads();




  



}


__global__ void hyperdifmomsourcene3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
  {

     wtemp[fencode3_hdmne1(p,iia,tmp6)]=wtemp[fencode3_hdmne1(p,iia,tmp5)]*((wd[fencode3_hdmne1(p,iia,hdnur)]+wd[fencode3_hdmne1(p,iia,hdnul)]))/4.0;





   }
}
__syncthreads();




  



}

__global__ void hyperdifmomsourcene2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
      wtemp[fencode3_hdmne1(p,iia,tmp5)]=(grad13_hdmne1(wtemp,p,iia,tmp4,dim));

}
__syncthreads();

}


__global__ void hyperdifmomsourcene1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];



  

   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;



  //init rhol and rhor
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode3_hdmne1(p,iia,f)]=0.0;

     dwn1[fencode3_hdmne1(p,iia,energy)]=0.0;
     dwn1[fencode3_hdmne1(p,iia,mom1+ii0)]=0.0;

   }
}
 __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
  {

     #ifdef ADIABHYDRO
;
    #else
     wtemp[fencode3_hdmne1(p,iia,tmp1)]=wmod[(shift)+fencode3_hdmne1(p,iia,rho)]+wmod[(shift)+fencode3_hdmne1(p,iia,rhob)];

     wtemp[fencode3_hdmne1(p,iia,tmp4)]=wmod[(shift)+fencode3_hdmne1(p,iia,mom1+field)]/(wmod[(shift)+fencode3_hdmne1(p,iia,rho)]+wmod[(shift)+fencode3_hdmne1(p,iia,rhob)]);
    #endif



   }
}
__syncthreads();


}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdmne1ne(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsourcene1(struct params **p, struct params **d_p, real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   //hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifmomsourcene1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene5_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene6_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();



}







