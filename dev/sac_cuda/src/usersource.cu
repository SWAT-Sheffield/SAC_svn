#include "hip/hip_runtime.h"


__device__ __host__
int addsourceterms2_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,xc3;
   real xxmax,yymax;
   real dx,dy,dz;
   real aa;
   real s_period;
   real tdep;

   real vx,vy;

   real exp_x,exp_y,exp_z,exp_xyz;


   real x,y,z;
   int i,j,k;
 	  
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

     xc1=0.1e6;
    xc2=1.0e6;
    xc3=1.0e6;

          x=(p->xmin[1])+(j*(p->dx[1]))-xc2;
          z=(p->xmin[0])+(i*(p->dx[0]))-xc1;
          y=(p->xmin[2])+(k*(p->dx[2]))-xc3;
     // xx=x(ix_1,ix_2,ix_3,2)-xc2
     // yy=x(ix_1,ix_2,ix_3,3)-xc3
     // zz=x(ix_1,ix_2,ix_3,1)-xc1  
  


    xxmax=2.0e6;
    yymax=2.0e6;

    dx=0.1e6;
    dy=0.1e6;
    dz=0.05e6;

    aa=10000.0;
    s_period=30.0;
    tdep=1.00;


        //exp_z=exp(-zz**2.d0/(delta_z**2.d0))
        //exp_x=exp(-xx**2.d0/(delta_x**2.d0))
        //exp_y=exp(-yy**2.d0/(delta_y**2.d0))       
        //exp_xyz=exp_x*exp_y*exp_z
        exp_z=exp(-z*z/(dz*dz));
        exp_x=exp(-x*x/(dx*dx));
        exp_y=exp(-y*y/(dy*dy));       
        exp_xyz=exp_x*exp_y*exp_z;

        //vvx(ix_1,ix_2,ix_3)=AA*yy/yymax*exp_xyz*tdep    
        //vvy(ix_1,ix_2,ix_3)=-AA*xx/xxmax*exp_xyz*tdep 
        vx=aa*y/yymax*exp_xyz*tdep;    
        vy=-aa*x/xxmax*exp_xyz*tdep; 
 
 switch(field)
  {

    case mom2:
                           dw[fencode3_MODID(p,ii,field)]=dw[fencode3_MODID(p,ii,field)]-vx*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
    break;
    case mom3:     
                           dw[fencode3_MODID(p,ii,field)]=dw[fencode3_MODID(p,ii,field)]-vy*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
    break;
    case energy:
                          dw[fencode3_MODID(p,ii,field)]=dw[fencode3_MODID(p,ii,field)]-(vx*vx+vy*vy)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;
    break;
   } 

  return ( status);
}

__device__ __host__
int addsourceterms1_MODID(real *dw, real *wd, real *w, struct params *p, struct state *s,int *ii,int field,int dir) {

  int direction;
  int status=0;

   real xc1,xc2,xc3;
   real xxmax,yymax;
   real delx,dely,delz;
   real aa;
   real s_period;
   real tdep;

   real vx,vy;

   real exp_x,exp_y,exp_z,exp_xyz;


   real x,y,z;
   int i,j,k;
 	  
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

     xc1=0.1e6;
    xc2=1.0e6;
    xc3=1.0e6;

          x=(p->xmin[1])+(j*(p->dx[1]))-xc2;
          z=(p->xmin[0])+(i*(p->dx[0]))-xc1;
          y=(p->xmin[2])+(k*(p->dx[2]))-xc3;
     // xx=x(ix_1,ix_2,ix_3,2)-xc2
     // yy=x(ix_1,ix_2,ix_3,3)-xc3
     // zz=x(ix_1,ix_2,ix_3,1)-xc1  
  


    xxmax=2.0e6;
    yymax=2.0e6;

    delx=0.1e6;
    dely=0.1e6;
    delz=0.05e6;

    aa=10000.0;
    s_period=30.0;
    tdep=1.00;


        //exp_z=exp(-zz**2.d0/(delta_z**2.d0))
        //exp_x=exp(-xx**2.d0/(delta_x**2.d0))
        //exp_y=exp(-yy**2.d0/(delta_y**2.d0))       
        //exp_xyz=exp_x*exp_y*exp_z
        exp_z=exp(-z*z/(delz*delz));
        exp_x=exp(-x*x/(delx*delx));
        exp_y=exp(-y*y/(dely*dely));       
        exp_xyz=exp_x*exp_y*exp_z;

        //vvx(ix_1,ix_2,ix_3)=AA*yy/yymax*exp_xyz*tdep    
        //vvy(ix_1,ix_2,ix_3)=-AA*xx/xxmax*exp_xyz*tdep 
        vx=aa*y/yymax*exp_xyz*tdep;    
        vy=-aa*x/xxmax*exp_xyz*tdep; 
 
 switch(field)
  {

    case mom2:
                           dw[fencode3_MODID(p,ii,field)]=dw[fencode3_MODID(p,ii,field)]-vx*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
    break;
    case mom3:     
                           dw[fencode3_MODID(p,ii,field)]=dw[fencode3_MODID(p,ii,field)]-vy*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)]);
    break;
    case energy:
                          dw[fencode3_MODID(p,ii,field)]=dw[fencode3_MODID(p,ii,field)]-(vx*vx+vy*vy)*(w[fencode3_MODID(p,ii,rho)]+w[fencode3_MODID(p,ii,rhob)])/2.0;
    break;
   }
 
   


  return ( status);
}

