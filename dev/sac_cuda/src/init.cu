#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_i.cuh"
#include "../include/init_user_i.cuh"


//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ip,jp;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));

   
 #ifdef USE_SAC_3D
   int kp;
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
/*   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#else
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif */ 

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
		//b[i+j*(p->n[0])]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read

	    for(int f=0; f<NVAR; f++)
            { 		         
                          for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=0;
	    }



//	 __syncthreads();

			}

        	
	 __syncthreads();






     #ifdef USE_SAC_3D
      // if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[2]>1 && ii[0]<(p->n[0])-1 && ii[1]<(p->n[1])-1 && ii[2]<(p->n[2])-1)
         if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1])   && ii[2]<(p->n[2])) 
     #else
      // if((p->readini==0) && ii[0]>2 && ii[1]>2 && ii[0]<(p->n[0])-3 && ii[1]<(p->n[1])-3)  //this form for OZT test???? 
     
     
     //if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[0]<(p->n[0])-1 && ii[1]<(p->n[1])-1)  //this form for OZT test???? 
        if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1]))  //this form for BW test  //still issue here
     #endif
	{


            #ifdef ADIABHYDRO
		    if(i> (((p->n[0])/2)-2) && i<(((p->n[0])/2)+2) && j>(((p->n[1])/2)-2) && j<(((p->n[1])/2)+2) ) 
				w[fencode3_i(p,ii,rho)]=1.3;
            #else
                   // init_alftest (real *w, struct params *p,int i, int j)
                   // init_alftest(w,p,i,j);
                   // init_ozttest (real *w, struct params *p,int i, int j)
                   // init_ozttest(w,p,i,j);
                   // init_bwtest(w,p,i,j);
                   init_user_i(w,p,ii);
           #endif

	

        }
	
	 __syncthreads();


       





     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
        /*for(int f=energyb; f<NVAR; f++)
             if(f != rhob)
                      w[fencode3_i(p,ii,f)]=0.0;*/
        //w[fencode3_i(p,ii,b2b)]=w[fencode3_i(p,ii,b3b)];
        for(int f=rho; f<NVAR; f++)
        {               
                  //wmod[fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];
                  //wmod[  (((3*(1+(p->rkon)))-1)*NVAR*dimp)+fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];              
                  dwn1[fencode3_i(p,ii,f)]=0;
                  for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                  {
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*dimp]=w[fencode3_i(p,ii,f)];
                              //wmod[fencode3_i(p,ii,b2b)+ord*NVAR*dimp]=w[fencode3_i(p,ii,b3b)];
                  }
                            
        }

        for(int f=tmp1; f<NTEMP; f++)
                 wtemp[fencode3_i(p,ii,f)]=0;


}

 __syncthreads();


     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
     
               for(int f=vel1; f<NDERV; f++)
                    wd[fencode3_i(p,ii,f)]=0.0;
     

 __syncthreads(); 
}

__device__ __host__
int encodempiw (struct params *p,int ix, int iy, int iz, int field,int bound,int dim) {
  #ifdef USE_SAC_3D
    return (dim*(    4*NVAR*(         ((p->n[0])*(p->n[1]))+((p->n[1])*(p->n[2]))+((p->n[0])*(p->n[2]))   )           )+4*field*(         ((p->n[0])*(p->n[1]))+((p->n[1])*(p->n[2]))+((p->n[0])*(p->n[2]))   )+
bound*(         (dim==2)*((p->n[0])*(p->n[1]))   +  (dim==0)*((p->n[1])*(p->n[2]))  +   (dim==1)*((p->n[0])*(p->n[2]))    )+   (  (ix+iz*(p->n[0]))*(dim==1)+(iy+iz*(p->n[1]))*(dim==0)+(iz+ix*(p->n[2]))*(dim==2)    ));
  #else
    return (dim*(4*NVAR*((p->n[0])+(p->n[1])))+4*field*((p->n[0])+(p->n[1]))+bound*((dim==1)*(p->n[0])+(dim==0)*(p->n[1]))  +   (ix*(dim==1)+iy*(dim==0)));
  #endif
}

__device__ __host__
int encodempivisc (struct params *p,int ix, int iy, int iz, int bound,int dim) {
  #ifdef USE_SAC_3D
    return (dim*(    2*(         (((p->n[0])+2)*((p->n[1])+2))+(((p->n[1])+2)*((p->n[2])+2))+(((p->n[0])+2)*((p->n[2])+2))   )           )+
bound*(         (dim==2)*(((p->n[0])+2)*((p->n[1])+2))   +  (dim==0)*(((p->n[1])+2)*((p->n[2])+2))  +   (dim==1)*(((p->n[0])+2)*((p->n[2])+2))    )+   (  (ix+iz*((p->n[0])+2))*(dim==1)+(iy+iz*((p->n[1])+2))*(dim==0)+(iz+ix*((p->n[2])+2))*(dim==2)    ));
  #else
    return (   dim*(2*(  ((p->n[0])+2)+((p->n[1])+2)   ))      +bound*(    (dim==1)*((p->n[0])+2)+(dim==0)*((p->n[1])+2)  )  +   (ix*(dim==1)+iy*(dim==0))     );
  #endif
}



     __device__ __host__ void mpiwtogpu(struct params *p,real *d_w,real *d_wmod,real *d_mpiw,real *d_mpiwmod,int *ii, int var, int dim)
    {

             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=1+(p->n[0])-i;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=1+(p->n[1])-j;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=1+(p->n[0])-k;
                    d_w[encode3_i(p,i,j,k,var)]=d_mpiw[encodempiw(p,i,j,k,var,bound,dim)];
                    d_wmod[encode3_i(p,i,j,k,var)]=d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)];              
                }

     #endif
 


    }

__device__ __host__ void   mpivisctogpu(struct params *p,real *d_wtemp2,real *d_gmpivisc,int *ii,  int dim)
{
                                
               int i,j,k,bound,var;
              var=0;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 ) && dim==0)
                {              
                    bound=i;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                    
                }
                else if((( i==((p->n[0])+1)   ))  && dim==0)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }

              

                if((j==0) && dim==1)              
                {              
                    bound=j;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }            
                 else if((( j==((p->n[1])+1)   ))  && dim==1)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 ) && dim==2)              
                {              
                    bound=k;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }        
                 else if(((k==((p->n[2])+1)   ))  && dim==2)               
                {
                    bound=1;
                    d_wtemp2[encode3p2_i(p,i,j,k,var)]=d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)];
                }

     #endif
                               
                                
}

__device__ __host__ void   gputompivisc(struct params *p,real *d_wtemp2,real *d_gmpivisc,int *ii,  int dim)
{
                                
              int i,j,k,bound,var;
              var=0;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 ) && dim==0)
                {              
                    bound=i;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                    
                }
                else if((( i==((p->n[0])+1)   ))  && dim==0)               
                {
                    bound=1;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }

              

                if((j==0) && dim==1)              
                {              
                    bound=j;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }            
                 else if((( j==((p->n[1])+1)   ))  && dim==1)               
                {
                    bound=1;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
             
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 ) && dim==2)              
                {              
                    bound=k;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }        
                 else if(((k==((p->n[2])+1)   ))  && dim==2)               
                {
                    bound=1;
                    d_gmpivisc[encodempivisc(p,i,j,k,bound,dim)]=d_wtemp2[encode3p2_i(p,i,j,k,var)];
                }

     #endif
                               
                                
}

     __device__ __host__ void gputompiw(struct params *p,real *d_w,real *d_wmod,real *d_mpiw,real *d_mpiwmod,int *ii, int var, int dim)
    {
             int i,j,k,bound;
i=ii[0];
j=ii[1];
k=0;
 
 
                if((i==0 || i==1) && dim==0)
                {              
                    bound=i;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];              
                }
                else if((( i>=((p->n[0])-2)   ))  && dim==0)               
                {
                    bound=1+(p->n[0])-i;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];               
                }

              

                if((j==0 || j==1) && dim==1)              
                {              
                    bound=j;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];              
                }            
                 else if((( j>=((p->n[1])-2)   ))  && dim==1)               
                {
                    bound=1+(p->n[1])-j;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];               
                }

       #ifdef USE_SAC_3D
               k=ii[2];
                if((k==0 || k==1) && dim==2)              
                {              
                    bound=k;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];              
                }        
                 else if((( k>=((p->n[2])-2)   ))  && dim==2)               
                {
                    bound=1+(p->n[0])-k;
                    d_mpiw[encodempiw(p,i,j,k,var,bound,dim)]=d_w[encode3_i(p,i,j,k,var)];
                    d_mpiwmod[encodempiw(p,i,j,k,var,bound,dim)]=d_wmod[encode3_i(p,i,j,k,var)];               
                }

     #endif
 
 }

__global__ void  mpiwtogpu_parallel(struct params *p,real *d_w, real *d_wmod, real *d_mpiw, real *d_mpiwmod)
{

int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

#ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      for(int dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b3; f++)
     #else
     for(int dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b2; f++)
     #endif     
         #ifdef USE_SAC_3D
           if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
         #else
           if(i<((p->n[0])) && j<((p->n[1])))
         #endif           
                      mpiwtogpu(p,d_w,d_wmod,d_mpiw,d_mpiwmod,iia,f,dim);


 __syncthreads();

           
               
}


     __global__ void gputompiw_parallel(struct params *p,real *d_w,real *d_wmod,real *d_mpiw,real *d_mpiwmod,int order)
    {

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
      for(dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b3; f++)
     #else
           for(dim=0; dim<NDIM;dim++)
           for( f=rho; f<=b2; f++)
     #endif
             {
            
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{

 

                  gputompiw(p,d_w,d_wmod,d_mpiw,d_mpiwmod,iia,f,dim);

	}

               }

 __syncthreads();

}



     __global__ void gputompivisc_parallel(struct params *p,real *d_wtemp2,real *d_gmpivisc)
     {
               
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/((nj+2)*(ni+2));
   jp=(iindex-(kp*((nj+2)*(ni+2))))/(ni+2);
   ip=iindex-(kp*(nj+2)*(ni+2))-(jp*(ni+2));
#else
    jp=iindex/(ni+2);
   ip=iindex-(jp*(ni+2));
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];

     #else

     #endif
           for(dim=0; dim<NDIM;dim++)
             {
            
         #ifdef USE_SAC_3D
      if(i<(((p->n[0])+2)) && j<(((p->n[1])+2))  && k<(((p->n[2])+2)))
     #else
       if(i<(((p->n[0])+2)) && j<(((p->n[1])+2)))
     #endif           
	{

 

                  gputompivisc(p,d_wtemp2,d_gmpivisc,iia,dim);

	}

               }

 __syncthreads();
              
               }    
     
     
    __global__ void  mpivisctogpu_parallel(struct params *p,real *d_wtemp2,real *d_gmpivisc)
    {
               
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;
int dim;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  
   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/((nj+2)*(ni+2));
   jp=(iindex-(kp*((nj+2)*(ni+2))))/(ni+2);
   ip=iindex-(kp*(nj+2)*(ni+2))-(jp*(ni+2));
#else
    jp=iindex/(ni+2);
   ip=iindex-(jp*(ni+2));
#endif     


//int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];

     #else

     #endif
           for(dim=0; dim<NDIM;dim++)
             {
            
         #ifdef USE_SAC_3D
      if(i<(((p->n[0])+2)) && j<(((p->n[1])+2))  && k<(((p->n[2])+2)))
     #else
       if(i<(((p->n[0])+2)) && j<(((p->n[1])+2)))
     #endif           
	{

 

                  mpivisctogpu(p,d_wtemp2,d_gmpivisc,iia,dim);

	}

               }

 __syncthreads();
               
               
}



/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, real **w, real **wnew, struct state **state, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct state *ads;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

if(((*p)->rkon)==1)
  hipMalloc((void**)d_wmod, 6*NVAR*dimp*sizeof(real));
else
  hipMalloc((void**)d_wmod, 3*NVAR*dimp*sizeof(real));

  hipMalloc((void**)d_dwn1, NVAR*dimp*sizeof(real));
  hipMalloc((void**)d_wd, NDERV*dimp*sizeof(real));
  hipMalloc((void**)d_wtemp, NTEMP*dimp*sizeof(real));


  #ifdef USE_SAC
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
  #endif
  #ifdef USE_SAC_3D
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)* (((*p)->n[2])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real));
  #endif

  hipMalloc((void**)&adw, NVAR*dimp*sizeof(real));
  hipMalloc((void**)&adwnew, NVAR*dimp*sizeof(real));
  
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->n[1]);

   // *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;

     
printf("allocating %d %d %d %d\n",dimp,(*p)->n[0],(*p)->n[1],(*p)->n[2]);
    hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
    printf("here\n");
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

        // printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->n[0])+((*p)->n[0])*((*p)->n[1])*b1]));



  return 0;



}





#ifdef USE_MPI

//prepare data buffers used to copy data between gpu and cpu
//this will update only the ghost cells transferred between the CPU's


int cuinitmpibuffers(struct params **p,real **w, real **wmod, real **temp2, real **gmpivisc,   real **gmpiw, real **gmpiwmod, struct params **d_p,   real **d_w, real **d_wmod,real **d_wtemp2,    real **d_gmpivisc,   real **d_gmpiw, real **d_gmpiwmod)
{

  int szw,  szvisc;
  #ifdef USE_SAC
  real *dt;
  
  szw=4*(  ((*p)->n[1])  +  ((*p)->n[0])   );
  szvisc=4*(  (((*p)->n[1])+2 )  +  (((*p)->n[0]) +2 )  );
 dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2),sizeof(real));

  #endif
  #ifdef USE_SAC_3D
  
  szw=4*NVAR*(  ((*p)->n[1])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[2])  +  ((*p)->n[0])*((*p)->n[1])  );
  szvisc=4*NVAR*(  (((*p)->n[1])+2)*(((*p)->n[2])+2)  +  (((*p)->n[0])+2)*(((*p)->n[2])+2)  +  (((*p)->n[0])+2)*(((*p)->n[1])+2)  );    
  dt=(real *)calloc( NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2),sizeof(real));
  #endif






  temp2=&dt;
  gmpiwmod=(real **)malloc(szw*sizeof(real));
  gmpiw=(real **)malloc(szw*sizeof(real));
  gmpivisc=(real **)malloc(szvisc*sizeof(real));
  
  
  hipMalloc((void**)d_gmpiwmod, NVAR*szw*sizeof(real));
  hipMalloc((void**)d_gmpiw, NVAR*szw*sizeof(real));
  hipMalloc((void**)d_gmpivisc, szvisc*sizeof(real));
  return 0;
}

//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopywtompiw(struct params **p,real **w, real **wmod,    real **gmpiw, real **gmpiwmod, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw, real **d_gmpiwmod, int order)
{
     int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;

     int szbuf;
     int dimp=(((*p)->n[0]))*(((*p)->n[1]));
     
     
   
     #ifdef USE_SAC_3D  
       dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
     #endif 
     int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif

    // for(var=0; var<NVAR; var++)
    //   for(dim=0;dim<NDIM;dim++)
     gputompiw_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw,*d_gmpiwmod,order);
     hipDeviceSynchronize();
     hipMemcpy(*gmpiwmod, *d_gmpiwmod, NVAR*szbuf*sizeof(real), hipMemcpyDeviceToHost);
     hipMemcpy(*gmpiw, *d_gmpiw, NVAR*szbuf*sizeof(real), hipMemcpyDeviceToHost);
     
     
//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;

                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                                                                     
                      // *(wmod+encode3_i(*p,ii,var))=*(gmpiwmod+encodempiw(*p,i1,i2,i3,var,bound,dim));              
                      // (*w)[encode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
         i3=bound*(bound<2)+( ((*p)->n[2])-(bound-1) )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*wmod)[fencode3_i(*p,ii,var)]=(*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)];              
                       (*w)[fencode3_i(*p,ii,var)]=(*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    

}

//copy mpi recv buffer to gpu memory     
int cucopywfrommpiw(struct params **p,real **w, real **wmod,    real **gmpiw, real **gmpiwmod, struct params **d_p  ,real **d_w, real **d_wmod,   real **d_gmpiw, real **d_gmpiwmod, int order)
{
       int i1,i2,i3;
     int ii[NDIM];
     int var,dim,bound;     
       int szbuf;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif      
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

      //copy data from w and wmod to correct gmpiw and gmpiwmod

//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;                                                                     
                       (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];

                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;  

                       (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];

                  }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      

                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
         i3=bound*(bound<2)+(((*p)->n[2])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                    }                            
                       break;                       
            #endif             
             }
                                     
         }    //encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
     for(var=0; var<NVAR; var++)
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<4;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(bound<2)+(((*p)->n[0])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3;     

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
  
                  }
            #else
         ii[2]=0;
         i1=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(bound<2)+(((*p)->n[1])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i3=0;i3<(((*p)->n[2]));i3++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 

                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                   }

            #else
         ii[2]=0;
         i2=bound*(bound<2)+(   ((*p)->n[1])-(bound-1)   )*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;


                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
         i3=bound*(bound<2)+(((*p)->n[2])-(bound-1))*(bound>1);
         for(i1=0;i1<(((*p)->n[0]));i1++ )
                  for(i2=0;i2<(((*p)->n[1]));i2++ )
                  {
                       ii[0]=i1;
                       ii[1]=i2;
                       ii[2]=i3; 


                      (*gmpiwmod)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*wmod)[fencode3_i(*p,ii,var)];              
                       (*gmpiw)[encodempiw(*p,i1,i2,i3,var,bound,dim)]=(*w)[fencode3_i(*p,ii,var)];      
                   }                            
                       break;                       
            #endif             
             }
                                     
         }    




   	 hipMemcpy(*d_gmpiw, *gmpiw, NVAR*szbuf*sizeof(real), hipMemcpyHostToDevice);     
   	 hipMemcpy(*d_gmpiwmod, *gmpiwmod, NVAR*szbuf*sizeof(real), hipMemcpyHostToDevice);     

     mpiwtogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod,*d_gmpiw,*d_gmpiwmod);
     hipDeviceSynchronize();
}

//copy gpu memory data to mpi send buffer for w and wmod
//just update the edges of w and wmod with values copied from gmpiw, gmpiwmod and gmpivisc
int cucopytompivisc(struct params **p,real **temp2, real **gmpivisc,  struct params **d_p,real **d_wtemp2,    real **d_gmpivisc)
{


     int szbuf;
     int dim,bound,var=0;
     int i1,i2,i3;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
             int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif
     gputompivisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wtemp2,*d_gmpivisc);
     hipDeviceSynchronize();
     hipMemcpy(*gmpivisc, *d_gmpivisc, NVAR*szbuf*sizeof(real), hipMemcpyDeviceToHost);

     //copy data to correct area in temp2
//encodempiw (struct params *dp,int ix, int iy, int iz, int field,int bound,int dim)
     //copy data to correct area in w and wmod
       for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<2;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(((*p)->n[0])+1);
         for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {     
                        
          //i1=(p->n[0])+1;
         
          //temp2[encode3p2_sacmpi (p,i1, i2, i3, tmpnui)]=gmpitgtbufferr[0][i2+i3*((p->n[1])+2)];
          //temp2[encode3p2_sacmpi (p,0, i2, i3, tmpnui)]=gmpitgtbufferl[0][i2+i3*((p->n[1])+2)];
         
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }
            #else
         i3=0;
         i1=bound*(((*p)->n[0])+1);
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(((*p)->n[1])+1);
         for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }

            #else
         i3=0;
         i2=bound*(((*p)->n[1])+1);
                  for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  {
                                                                     
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
                  i3=bound*(((*p)->n[2])+1);
        for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                                                              
                       (*temp2)[encode3p2_i(*p,i1,i2,i3,var)]=(*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    

}

//copy mpi recv buffer to gpu memory     
int cucopyfrommpivisc(struct params **p,real **temp2,real **gmpivisc,  struct params **d_p,real **d_wtemp2,    real **d_gmpivisc)
{
      int dim,bound,var=0;
     int i1,i2,i3;      
       int szbuf;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D  
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

        int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     
     szbuf=2*2*( ((*p)->n[0])+((*p)->n[1]));
     #ifdef USE_SAC3D
     szbuf=2*2*( ((*p)->n[0])*((*p)->n[1])+ ((*p)->n[0])*((*p)->n[2]) + ((*p)->n[1])*((*p)->n[2])        );
     #endif

      //copy data from temp2 to gmpivisc
             for(dim=0;dim<NDIM;dim++) 
         for(bound=0;bound<2;bound++)
         {
            switch(dim)
            {
                       case 0:
            #ifdef USE_SAC3D
         i1=bound*(((*p)->n[0])+1);
         for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {     
                        
          //i1=(p->n[0])+1;
         
          //temp2[encode3p2_sacmpi (p,i1, i2, i3, tmpnui)]=gmpitgtbufferr[0][i2+i3*((p->n[1])+2)];
          //temp2[encode3p2_sacmpi (p,0, i2, i3, tmpnui)]=gmpitgtbufferl[0][i2+i3*((p->n[1])+2)];
         
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }
            #else
         i3=0;
         i1=bound*(((*p)->n[0])+1);
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }            
            
            #endif
                       
                       break;   
                       case 1:
            #ifdef USE_SAC3D
         i2=bound*(((*p)->n[1])+1);
         for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i3=1;i3<(((*p)->n[2])+2);i3++ )
                  {
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }

            #else
         i3=0;
         i2=bound*(((*p)->n[1])+1);
                  for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  {
                                                                     
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }
            
            
            #endif
                       
                       break; 
            #ifdef USE_SAC3D
                       case 2:
                  i3=bound*(((*p)->n[2])+1);
        for(i1=1;i1<(((*p)->n[0])+2);i1++ )
                  for(i2=1;i2<(((*p)->n[1])+2);i2++ )
                  {
                                                              
                       (*gmpivisc)[encodempivisc(*p,i1,i2,i3,bound,dim)]=(*temp2)[encode3p2_i(*p,i1,i2,i3,var)];
                  }                            
                       break;                       
            #endif             
             }
                                     
         }    


   	 hipMemcpy(*d_gmpivisc, *gmpivisc, NVAR*szbuf*sizeof(real), hipMemcpyHostToDevice);     

     mpivisctogpu_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wtemp2,*d_gmpivisc);
     hipDeviceSynchronize();
}


#endif



