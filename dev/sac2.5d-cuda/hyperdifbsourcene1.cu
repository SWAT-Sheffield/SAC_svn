#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdbne1.cuh"



__global__ void hyperdifbsourcene6_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;






   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  if(i<((p->n[0])) && j<((p->n[1])))
	{		               

dwn1[fencode_hdbne1(p,i,j,energy)]=sb*wtemp[fencode_hdbne1(p,i,j,tmp6)];

dwn1[fencode_hdbne1(p,i,j,b1+ii0)]=sb*wtemp[fencode_hdbne1(p,i,j,tmp4)];


   }
}
 __syncthreads();



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         if(i<(ni) && j<(nj))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hdbne1(p,i,j,b1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdbne1(p,i,j,b1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdbne1(p,i,j,b1+ii0)]; 
                             wmod[fencode_hdbne1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdbne1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdbne1(p,i,j,energy)]; 

                         }
              //  }	
}
  __syncthreads();  
  
}


__global__ void hyperdifbsourcene5_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


//int shift=order*NVAR*(p->n[0])*(p->n[1]);



   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));




   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {



       wtemp[fencode_hdbne1(p,i,j,tmp6)]=grad1_hdbne1(wtemp,p,i,j,tmp5,mm);

   }

}
__syncthreads();




}



__global__ void hyperdifbsourcene4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

int shift=order*NVAR*(p->n[0])*(p->n[1]);




   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  if( i<((p->n[0])) && j<((p->n[1])))
  {



wtemp[fencode_hdbne1(p,i,j,tmp5)]=wtemp[fencode_hdbne1(p,i,j,tmp3)]*wmod[(shift)+fencode_hdbne1(p,i,j,b1+jj)];

   }

}
__syncthreads();



  
}



__global__ void hyperdifbsourcene3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;






   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {



 wtemp[fencode_hdbne1(p,i,j,tmp4)]=grad1_hdbne1(wtemp,p,i,j,tmp3,mm);


   }

}
__syncthreads();








   

  
}



__global__ void hyperdifbsourcene2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
int shift=order*NVAR*(p->n[0])*(p->n[1]);
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<((p->n[0])) && j<((p->n[1])))
  {



      wtemp[fencode_hdbne1(p,i,j,tmp3)]=wtemp[fencode_hdbne1(p,i,j,tmp2)]*(wd[fencode_hdbne1(p,i,j,hdnul)]+wd[fencode_hdbne1(p,i,j,hdnur)])/2;

   }

}
__syncthreads();



  
}



__global__ void hyperdifbsourcene1a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;




int shift=order*NVAR*(p->n[0])*(p->n[1]);

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


       wtemp[fencode_hdbne1(p,i,j,tmp2)]=grad1_hdbne1(wtemp,p,i,j,tmp1,dim);



   }

}
__syncthreads();



  
}


__global__ void hyperdifbsourcene1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;




int shift=order*NVAR*(p->n[0])*(p->n[1]);

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode_hdbne1(p,i,j,f)]=0.0;

   dwn1[fencode_hdbne1(p,i,j,energy)]=0.0;
   dwn1[fencode_hdbne1(p,i,j,b1+ii0)]=0.0;

  }
}
 __syncthreads();



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if( i<((p->n[0])) && j<((p->n[1])))
  {

wtemp[fencode_hdbne1(p,i,j,tmp1)]=wmod[(shift)+fencode_hdbne1(p,i,j,b1+field)];



   }

}
__syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


       wtemp[fencode_hdbne1(p,i,j,tmp2)]=grad1_hdbne1(wtemp,p,i,j,tmp1,dim);



   }

}
__syncthreads();*/



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdbne1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifbsourcene1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim, int jj, int ii0,int mm,real sb,real dt)
{



 dim3 dimBlock(dimblock, 1);

    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifbsourcene1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize();
     hyperdifbsourcene5_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize();
     hyperdifbsourcene6_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize(); 


}







