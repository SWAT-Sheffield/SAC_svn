#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_i.cuh"
#include "init_user_i.cuh"



//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp, real *wtemp1, real *wtemp2)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 // int index,k;
int ni=p->n[0];
  int nj=p->n[1];
#ifdef USE_SAC_3D
  int nk=p->n[2];
#endif


// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

   int ord;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;
  int ii[NDIM];
   int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
#endif
   
 #ifdef USE_SAC_3D
   int kp,kpg;
  dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #ifdef USE_SAC
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  

   

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
		//b[i+j*(p->n[0])]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read

	    for(int f=0; f<NVAR; f++)
            { 
		         
                          for(ord=0;ord<(2+3*(p->rkon==1));ord++)
                              wmod[fencode3_i(p,ii,f)+ord*NVAR*(p->n[0])*(p->n[1])]=0;
	    }



//	 __syncthreads();

			}

        }	
	 __syncthreads();



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if((p->readini==0) && ii[0]>1 && ii[1]>1  && ii[2]>1 && ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2 && ii[2]<(p->n[2])-2)
     #else
       if((p->readini==0) && ii[0]>1 && ii[1]>1 && ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)  //this form for OZT test???? 
     // if((p->readini==0) && ii[0]>=0 && ii[1]>=0 && ii[2]>1 && ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)  //this form for OZT test???? 
      //  if((p->readini==0) && ii[0]<(p->n[0]) && ii[1]<(p->n[1]))  //this form for BW test  //still issue here
     #endif
	{


            #ifdef ADIABHYDRO
		    if(i> (((p->n[0])/2)-2) && i<(((p->n[0])/2)+2) && j>(((p->n[1])/2)-2) && j<(((p->n[1])/2)+2) ) 
				w[fencode3_i(p,ii,rho)]=1.3;
            #else
                   // init_alftest (real *w, struct params *p,int i, int j)
                   // init_alftest(w,p,i,j);
                   // init_ozttest (real *w, struct params *p,int i, int j)
                   // init_ozttest(w,p,i,j);
                   // init_bwtest(w,p,i,j);
                   init_user_i(w,p,ii);
           #endif

	

        }
}	
	 __syncthreads();


       




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{
        for(int f=rho; f<NVAR; f++)
        {               
                  wmod[fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];
                  wmod[  (((3*(1+(p->rkon)))-1)*NVAR*dimp)+fencode3_i(p,ii,f)]=w[fencode3_i(p,ii,f)];              
                  dwn1[fencode3_i(p,ii,f)]=0;
                            
        }

        for(int f=tmp1; f<NTEMP; f++)
                 wtemp[fencode3_i(p,ii,f)]=0;


}
}
 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
     
               for(int f=vel1; f<NDERV; f++)
                    wd[fencode3_i(p,ii,f)]=0.0;
     }

 __syncthreads(); 
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, real **w, real **wnew, struct state **state, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct state *ads;

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

if(((*p)->rkon)==1)
  hipMalloc((void**)d_wmod, 6*NVAR*dimp*sizeof(real));
else
  hipMalloc((void**)d_wmod, 3*NVAR*dimp*sizeof(real));

  hipMalloc((void**)d_dwn1, NVAR*dimp*sizeof(real));
  hipMalloc((void**)d_wd, NDERV*dimp*sizeof(real));
  hipMalloc((void**)d_wtemp, NTEMP*dimp*sizeof(real));


  #ifdef USE_SAC
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real));
  #endif
  #ifdef USE_SAC_3D
  hipMalloc((void**)d_wtemp1, NTEMP1*(((*p)->n[0])+1)* (((*p)->n[1])+1)* (((*p)->n[2])+1)*sizeof(real));
  hipMalloc((void**)d_wtemp2, NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real));
  #endif

  hipMalloc((void**)&adw, NVAR*dimp*sizeof(real));
  hipMalloc((void**)&adwnew, NVAR*dimp*sizeof(real));
  
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->n[1]);

   // *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;

     
printf("allocating\n");
    hipMemcpy(*d_w, *w, NVAR*dimp*sizeof(real), hipMemcpyHostToDevice);
   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
    
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp, *d_wtemp1, *d_wtemp2);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

        // printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->n[0])+((*p)->n[0])*((*p)->n[1])*b1]));



  return 0;



}


