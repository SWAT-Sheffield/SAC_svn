#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int encode_i (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_i (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}



__global__ void init_parallel(struct params *p, float *w, float *wnew, float *b, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->ni;
  int nj=p->nj;

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

 int nli = 0.45*(p->ni-1)+1;
  int nui = 0.55*(p->ni-1)+1;
  int nlj = 0.45*(p->nj-1)+1;
  int nuj = 0.55*(p->nj-1)+1; 
  int i,j;
   
   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{
		b[i+j*(p->ni)]=0;

                 //Define b	
       for(int f=0; f<=6; f++)
        { 
                  wd[fencode_i(p,i,j,f)]=0;
        }

        for(int f=rho; f<=b3; f++)
        {               
                  wnew[fencode_i(p,i,j,f)]=w[fencode_i(p,i,j,f)];
                  dwn1[fencode_i(p,i,j,f)]=0;
                  //dwn2[fencode(p,i,j,f)]=0;
                 // dwn3[fencode(p,i,j,f)]=0;
                  //dwn4[fencode(p,i,j,f)]=0;
                 
        }

	 __syncthreads();

			}	
	 __syncthreads();
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, float **w, float **wnew,  float **b, struct state **state, struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd, struct state **d_state)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
  float *adb;
  float *adw, *adwnew;
  struct params *adp;
  struct state *ads;


  hipMalloc((void**)d_wmod, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_dwn1, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_wd, 7*((*p)->ni)* ((*p)->nj)*sizeof(float));

  hipMalloc((void**)&adw, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adwnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adb, 1*(((*p)->ni)* ((*p)->nj))*sizeof(float));
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->nj);

    *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;


    hipMemcpy(*d_w, *w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_wnew, *wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_b, *b, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b, *d_wmod, *d_dwn1,  *d_wd);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);

	//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);




  return 0;



}


