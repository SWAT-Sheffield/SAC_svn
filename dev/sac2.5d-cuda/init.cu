#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int encode_i (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_i (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}



__global__ void init_parallel(struct params *p, float *w, float *wnew, float *b, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->ni;
  int nj=p->nj;

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

 int nli = 0.45*(p->ni-1)+1;
  int nui = 0.55*(p->ni-1)+1;
  int nlj = 0.45*(p->nj-1)+1;
  int nuj = 0.55*(p->nj-1)+1; 
  int i,j;
   
   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{
		b[i+j*(p->ni)]=0;

                 //Define b	
		if((i*(p->dx)) >20001)
		      b[j*(p->ni)+i]=0;
		else if((i*(p->dx)) <20000)
			//b[j*(p->ni)+i]=(5000/20000)*(20000-(i*(p->dx)));
                        b[j*(p->ni)+i]=0;
                        // b[j*(p->ni)+i]=5000*(1.0-(((float)i)/30.0));		



		//initialise the arrays here
               for(k=0;k<1;++k)
      		{
                    index=j*(p->ni)+i+k*(p->ni)*(p->nj);
                    //index=i+j*(p->ni)+(k*(p->nj)*(p->ni));
		    u[index]=0;
		    v[index]=0;
		    h[index]=5;
                    w[index+mom3*(p->ni)*(p->nj)]=0;
                    w[index+energy*(p->ni)*(p->nj)]=0;
                    w[index+b1*(p->ni)*(p->nj)]=0;
                    w[index+b2*(p->ni)*(p->nj)]=0;
                    w[index+b3*(p->ni)*(p->nj)]=0;

//float *wmod, 
//    float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)


      		}
		//h[iindex]=5000;
	
        __syncthreads();
        if(i>=nli && i<=nui && j>=nlj && j<=nuj)
	{
	   //j*(p->ni)+i;
           h[j*(p->ni)+i]=5.030;	
	}

       for(int f=0; f<=5; f++)
        { 
                  wd[fencode_i(p,i,j,f)]=0;
        }

        for(int f=rho; f<=b3; f++)
        {               
                  wnew[fencode_i(p,i,j,f)]=w[fencode_i(p,i,j,f)];
                  dwn1[fencode_i(p,i,j,f)]=0;
                  //dwn2[fencode(p,i,j,f)]=0;
                 // dwn3[fencode(p,i,j,f)]=0;
                  //dwn4[fencode(p,i,j,f)]=0;
                 
        }

	 __syncthreads();

			}	
	 __syncthreads();
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, float **w, float **wnew,  float **b, struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
  float *adb;
  float *adw, *adwnew;
  struct params *adp;

  hipMalloc((void**)d_wmod, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_dwn1, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_wd, 6*((*p)->ni)* ((*p)->nj)*sizeof(float));

  hipMalloc((void**)&adw, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adwnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adb, 1*(((*p)->ni)* ((*p)->nj))*sizeof(float));
  hipMalloc((void**)&adp, sizeof(struct params));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->nj);

    *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;


    hipMemcpy(*d_w, *w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_wnew, *wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_b, *b, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b, *d_wmod, *d_dwn1,  *d_wd);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);




  return 0;



}


