#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int encode_i (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_i (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->ni;
  int nj=p->nj;

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  float *u,  *v,  *h;

  int seg1,seg2,seg3;
  int width=10;
  float m2max=0.001;
  float start=((p->ni)-width)/2;
  seg1=2*(p->ni)/5;
  seg2=3*(p->ni)/5;
  seg3=4*(p->ni)/5;

//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;

   
   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{
		//b[i+j*(p->ni)]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read
	if(p->readini==0)
	{
	    for(int f=0; f<=6; f++)
            { 
		          w[fencode_i(p,i,j,f)]=0;
	    }
	    w[fencode_i(p,i,j,rho)]=1.0;
	    w[fencode_i(p,i,j,b1)]=1.0;
	    w[fencode_i(p,i,j,energy)]=0.0001;

	   if (i > seg1)
	    if (i < seg2)
	      w[fencode_i(p,i,j,mom2)]=m2max;


	   if (i > seg2)
	    if (i < seg3)
	      w[fencode_i(p,i,j,mom2)]=m2max*(i-seg2)/(seg3-seg2);

	   if (i > seg3)
	      w[fencode_i(p,i,j,mom2)]=m2max*((p->ni)-i)/((p->ni)-seg3);

	}


        for(int f=rho; f<=b3; f++)
        {               
                  wnew[fencode_i(p,i,j,f)]=w[fencode_i(p,i,j,f)];
                  dwn1[fencode_i(p,i,j,f)]=0;
                  //dwn2[fencode(p,i,j,f)]=0;
                 // dwn3[fencode(p,i,j,f)]=0;
                  //dwn4[fencode(p,i,j,f)]=0;
                 
        }

//	 __syncthreads();

			}	
	 __syncthreads();


  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, float **w, float **wnew, struct state **state, struct params **d_p, float **d_w, float **d_wnew, float **d_wmod, float **d_dwn1, float **d_wd, struct state **d_state)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // float *adb;
  float *adw, *adwnew;
  struct params *adp;
  struct state *ads;


  hipMalloc((void**)d_wmod, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_dwn1, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_wd, 7*((*p)->ni)* ((*p)->nj)*sizeof(float));

  hipMalloc((void**)&adw, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adwnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->nj);

   // *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;


    hipMemcpy(*d_w, *w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
	hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

         printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->ni)+((*p)->ni)*((*p)->nj)*b1]));



  return 0;



}


