#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_i.cuh"


//*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd

__global__ void init_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, real *wtemp)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->n[0];
  int nj=p->n[1];

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  real *u,  *v,  *h;

  int seg1,seg2,seg3,seg4;
  int width=10;
  real m2max=0.001;
  real start=((p->n[0])-width)/2;
  //seg1=((p->n[0])/3)-1;
  seg1=(p->n[0])/6;
  seg2=((p->n[0])/3);
  seg3=(2*(p->n[0])/3)-1;
  //seg4=(2*(p->n[0])/3);
  seg4=(p->n[0])-seg1;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int i,j;

   
   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->n[0] && j<p->n[1])
	{
		//b[i+j*(p->n[0])]=0;

                 //Define b	

 


	//apply this special condition
	//initiate alfven wave propagtion 
	//if no initial config read
	if(p->readini==0)
	{
	    for(int f=0; f<=NVAR; f++)
            { 
		          w[fencode_i(p,i,j,f)]=0;
	    }
            w[fencode_i(p,i,j,rho)]=1.0;
            #ifdef ADIABHYDRO
		    if(i> (((p->n[0])/2)-2) && i<(((p->n[0])/2)+2) && j>(((p->n[1])/2)-2) && j<(((p->n[1])/2)+2) ) 
				w[fencode_i(p,i,j,rho)]=1.3;
            #else

		    w[fencode_i(p,i,j,rho)]=1.0;
		    w[fencode_i(p,i,j,b1)]=1.0;
		    w[fencode_i(p,i,j,energy)]=0.01;

		    //w[fencode_i(p,i,j,b1)]=15*j;
		    //w[fencode_i(p,i,j,b3)]=150*j;
		    
		   //if (i > seg2)
		    //if (i < seg3)
                   // if (i < seg1)
		   //   w[fencode_i(p,i,j,mom2)]=0.0;

		   if (i > seg1)
		    if (i < seg2)
		      w[fencode_i(p,i,j,mom2)]=m2max*(i-seg1)/(seg2-seg1);

		   if (i > seg2)
		    if (i < seg3)
		      //w[fencode_i(p,i,j,mom2)]=m2max*(i-seg2)/(seg3-seg2);
                      w[fencode_i(p,i,j,mom2)]=m2max;
		   if (i > seg3)
		    if (i < seg4)
		      w[fencode_i(p,i,j,mom2)]=m2max*(seg4-i)/(seg4-seg3);
           #endif

	}


//	 __syncthreads();

			}	
	 __syncthreads();

  if(i<p->n[0] && j<p->n[1])
	{
        for(int f=rho; f<=b3; f++)
        {               
                  wnew[fencode_i(p,i,j,f)]=w[fencode_i(p,i,j,f)];
              for(int ord=0;ord<(1+3*((p->rkon)==1));ord++)
                  dwn1[NVAR*ord*ni*nj+fencode_i(p,i,j,f)]=0;
                  //dwn2[fencode(p,i,j,f)]=0;
                 // dwn3[fencode(p,i,j,f)]=0;
                  //dwn4[fencode(p,i,j,f)]=0;
                 
        }

        for(int f=tmp1; f<=tmprhor; f++)
                 wtemp[fencode_i(p,i,j,f)]=0;


}

 __syncthreads();
        if(i<p->n[0] && j<p->n[1])
               for(int f=current1; f<=hdnul; f++)
                    wd[fencode_i(p,i,j,f)]=0.0;

 __syncthreads(); 
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_i(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, real **w, real **wnew, struct state **state, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors_i("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
 // real *adb;
  real *adw, *adwnew;
  struct params *adp;
  struct state *ads;


  hipMalloc((void**)d_wmod, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)d_dwn1, NVAR*(1+3*((*p)->rkon))*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)d_wd, NDERV*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)d_wtemp, NDERV*((*p)->n[0])* ((*p)->n[1])*sizeof(real));

  hipMalloc((void**)&adw, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  hipMalloc((void**)&adwnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real));
  
  hipMalloc((void**)&adp, sizeof(struct params));
  hipMalloc((void**)&ads, sizeof(struct state));
  checkErrors_i("memory allocation");

printf("ni is %d\n",(*p)->n[1]);

   // *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;
    *d_state=ads;


    hipMemcpy(*d_w, *w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
   // hipMemcpy(*d_wnew, *wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyHostToDevice);
    
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    hipMemcpy(*d_state, *state, sizeof(struct state), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_wmod, *d_dwn1,  *d_wd, *d_wtemp);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);

	hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
        hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

        // printf("mod times step %f %f\n",(*p)->dt, ((*wnew)[10+16*((*p)->n[0])+((*p)->n[0])*((*p)->n[1])*b1]));



  return 0;



}


