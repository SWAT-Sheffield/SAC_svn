#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdmne.cuh"



__global__ void hyperdifmomsourcene_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, int field, int dim, int ii, int ii0)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmp2,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
    for(int f=tmprhol; f<=tmprhor; f++)	
        wtemp[fencode_hdmne(p,i,j,f)]=0.0;

 __syncthreads();

  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  {

#ifdef USE_SAC
       wtemp[fencode_hdmne(p,i,j,tmprhor)]=(wmod[fencode_hdmne(p,i,j,rho)]+wmod[fencode_hdmne(p,i,j,rhob)]+wmod[fencode_hdmne(p,i+(dim==0),j+(dim==1),rho)]+wmod[fencode_hdmne(p,i+(dim==0),j+(dim==1),rhob)])/2;
       wtemp[fencode_hdmne(p,i,j,tmprhol)]=(wmod[fencode_hdmne(p,i,j,rho)]+wmod[fencode_hdmne(p,i,j,rhob)]+wmod[fencode_hdmne(p,i-(dim==0),j+(dim==1),rho)]+wmod[fencode_hdmne(p,i-(dim==0),j+(dim==1),rhob)])/2;
#else
       wtemp[fencode_hdmne(p,i,j,tmprhor)]=(wmod[fencode_hdmne(p,i,j,rho)]+wmod[fencode_hdmne(p,i+(dim==0),j+(dim==1),rho)])/2;
       wtemp[fencode_hdmne(p,i,j,tmprhol)]=(wmod[fencode_hdmne(p,i,j,rho)]+wmod[fencode_hdmne(p,i-(dim==0),j+(dim==1),rho)])/2;
#endif

   }
__syncthreads();


  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  {


#ifdef USE_SAC
     wtemp[fencode_hdmne(p,i,j,tmp1)]=wmod[fencode_hdmne(p,i,j,mom1+field)]/(wmod[fencode_hdmne(p,i,j,rho)]+wmod[fencode_hdmne(p,i,j,rhob)]);
#else
     wtemp[fencode_hdmne(p,i,j,tmp1)]=wmod[fencode_hdmne(p,i,j,mom1+field)]/wmod[fencode_hdmne(p,i,j,rho)];
#endif



  }

__syncthreads();



  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
             //ii1=0
             //case i=k, ii0=l

    

 /* for(ii1=0;ii1<=1;ii1++)
  {
        if (ii1 == 0)
        {
                           ii=field-mom1;
                           ii0=dim;
        }
         else
         {
                           ii=dim;
                           ii0=field-mom1;
        }*/



              if(field==dim)
    {
                        //ii0=field;
                        //ii=dim;

     wtemp[fencode_hdmne(p,i,j,tmp2)]=grad1l_hdmne(wtemp,p,i,j,tmp1,dim);
     wtemp[fencode_hdmne(p,i,j,tmp3)]=grad1r_hdmne(wtemp,p,i,j,tmp1,dim);


wtemp[fencode_hdmne(p,i,j,tmp4)]=wtemp[fencode_hdmne(p,i,j,tmprhor)]*wd[fencode_hdmne(p,i,j,hdnur)]*wtemp[fencode_hdmne(p,i,j,tmp3)];
wtemp[fencode_hdmne(p,i,j,tmp5)]=wtemp[fencode_hdmne(p,i,j,tmprhol)]*wd[fencode_hdmne(p,i,j,hdnul)]*wtemp[fencode_hdmne(p,i,j,tmp2)];


//grad1l_hdmne()

dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdmne(p,i,j,energy)]=(wtemp[fencode_hdmne(p,i,j,tmp4)]-wtemp[fencode_hdmne(p,i,j,tmp5)])/(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1))/2;

dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdmne(p,i,j,mom1+ii0)]=(wtemp[fencode_hdmne(p,i,j,tmp4)]-wtemp[fencode_hdmne(p,i,j,tmp5)])/(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1))/2;

         }
    else
    {




     



			//ii=field;
                        //ii0=dim;
     wtemp[fencode_hdmne(p,i,j,tmp2)]=grad1_hdmne(wtemp,p,i,j,tmp1,dim)*(wd[fencode_hdmne(p,i,j,hdnur)]+wd[fencode_hdmne(p,i,j,hdnul)])/4.0;

#ifdef USE_SAC
     wtemp[fencode_hdmne(p,i,j,tmp3)]=wtemp[fencode_hdmne(p,i,j,tmp2)]*(wmod[fencode_hdmne(p,i,j,rho)]+wmod[fencode_hdmne(p,i,j,rhob)]);
#else
     wtemp[fencode_hdmne(p,i,j,tmp3)]=wtemp[fencode_hdmne(p,i,j,tmp2)]*(wmod[fencode_hdmne(p,i,j,rho)]);
#endif
 wtemp[fencode_hdmne(p,i,j,tmp4)]=grad1_hdmne(wtemp,p,i,j,tmp3,ii);

dwn1[fencode_hdmne(p,i,j,mom1+ii0)]=wtemp[fencode_hdmne(p,i,j,tmp4)];

wtemp[fencode_hdmne(p,i,j,tmp5)]=wmod[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdmne(p,i,j,mom1+ii0)]*(wmod[fencode_hdmne(p,i,j,rho)]+wtemp[fencode_hdmne(p,i,j,tmp3)]);
wtemp[fencode_hdmne(p,i,j,tmp4)]=grad1_hdmne(wtemp,p,i,j,tmp5,ii);

dwn1[fencode_hdmne(p,i,j,energy)]=wtemp[fencode_hdmne(p,i,j,tmp4)];


 //}
               
 


	}
   }
 __syncthreads();


			 if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hdmne(p,i,j,field)+(order*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdmne(p,i,j,field)+(order*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdmne(p,i,j,field)]; 
//wmod[fencode_hdmne(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=dwn1[fencode_hdmne(p,i,j,f2)];
                              //dwn1[fencode_hdmne(p,i,j,f)]=0;
                         }
              //  }	

  __syncthreads();


  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdmne(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsourcene(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, int field, int dim, int ii, int ii0)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifmomsourcene_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order,*d_wtemp, field, dim,ii,ii0);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







