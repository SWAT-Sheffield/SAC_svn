#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void init_parallel(struct params *p, float *b, float *u, float *v, float *h)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
 // int i = blockIdx.x * blockDim.x + threadIdx.x;
 // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->ni;
  int nj=p->nj;

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    


 int nli = 0.45*(p->ni-1)+1;
  int nui = 0.55*(p->ni-1)+1;
  int nlj = 0.45*(p->nj-1)+1;
  int nuj = 0.55*(p->nj-1)+1; 
  int i,j;
   
   j=iindex/ni;
   i=iindex-j*ni;
  if(i<p->ni && j<p->nj)
	{
		b[j+i*(p->nj)]=0;
		//initialise the arrays here
               for(k=0;k<2;k++)
      		{
                    index=i+j*(p->ni)+(k*(p->nj)*(p->ni));
		    u[index]=0;
		    v[index]=0;
		    h[index]=0;
      		}
		h[i+j*(p->ni)]=5000;
	}
        __syncthreads();
        if(i>=nli && i<=nui && j>=nlj && j<=nuj)
	{
	   
           h[iindex]=5030;	
	}
	 __syncthreads();
//Define b	
  if(i<p->ni && j<p->nj)
	{
		
		if(i*p->dx >20001)
		      b[iindex]=0;
		else if(i*p->dx <20000)
			b[iindex]=(5000/20000)*(20000-i*p->dx);		
	}	
	 __syncthreads();
  
}



__global__ void prop_parallel(struct params *p, float *b, float *u, float *v, float *h)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;

  if(i>0 && j >0 && i<((p->ni)-1) && j<((p->nj)-1))
	{
		//update the arrays here
               u[i+j*ni+ni*nj] = ((u[i+1+j*ni] + u[i-1+j*ni] + u[i+(j+1)*ni] + u[i+(j-1)*ni])/4)- 0.5*(dt/dx)*(((u[(i+1)+ni*j])*u[(i+1)+ni*j])/2) - ((u[i-1+j*ni]*u[i-1+j*ni])/2) - 0.5*(dt/dy)*(v[i+j*ni])*(u[i+(j+1)*ni] - u[i+(j-1)*ni]) - 0.5*g*(dt/dx)*(h[i+1+j*ni]-h[i-1+j*ni]);

v[i+j*ni+ni*nj] = ((v[i+1+j*ni] + v[i-1+j*ni] + v[i+(j+1)*ni] + v[i+(j-1)*ni])/4)- 0.5*(dt/dy)*((v[i+ni*(j+1)]*v[(i)+ni*(j+1)])/2 - (v[i+(j-1)*ni]*v[i+(j-1)*ni])/2) - 0.5*(dt/dx)*(u[i+j*ni])*(v[i+1+j*ni] - v[i-1+j*ni]) - 0.5*g*(dt/dy)*(h[i+(j+1)*ni]-h[i+(j-1)*ni]);

h[i+j*ni+ni*nj] = ((h[i+1+j*ni] + h[i-1+j*ni] + h[i+(j+1)*ni] + h[i+(j-1)*ni])/4)- 0.5*(dt/dx)*(u[i+j*ni])*((h[i+1+j*ni]-b[i+1+j*ni]) - (h[i-1+j*ni]-b[i-1+j*ni])) - 0.5*(dt/dy)*(v[i+j*ni])*((h[i+(j+1)*ni]-b[i+(j+1)*ni]) - (h[i+(j-1)*ni]-b[i+(j-1)*ni])) - 0.5*(dt/dx)*(h[i+j*ni]-b[i+j*ni])*(u[i+1+j*ni]- u[i-1+j*ni])- 0.5*(dt/dy)*(h[i+j*ni]-b[i+j*ni])*(v[i+(j+1)*ni] - v[i+(j-1)*ni]);

	}
  
}

__global__ void boundary_parallel(struct params *p, float *b, float *u, float *v, float *h)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;

  if(i<p->ni && j<p->nj)
	{

		if(i=0)
		{
			u[1+j*ni+ni*nj] = 2.5*u[2+j*ni+ni*nj] - 2*u[3+j*ni+ni*nj] + 0.5*u[4+j*ni+ni*nj];
			u[ni-1+j*ni+ni*nj] = 2.5*u[ni-1+j*ni+ni*nj] - 2*u[ni-2+ni*j+ni*nj] + 0.5*u[ni-3+j*ni+ni*nj];
			v[1+j*ni+ni*nj] = 2.5*v[2+j*ni+ni*nj] - 2*v[3+j*ni+ni*nj] + 0.5*v[4+j*ni+ni*nj];
		 	v[ni-1+j*ni+ni*nj] = 2.5*v[ni-1+j*ni+ni*nj] - 2*v[ni-2+ni*j+ni*nj] + 0.5*v[ni-3+j*ni+ni*nj];
		 	h[1+j*ni+ni*nj] = 2.5*h[2+j*ni+ni*nj] - 2*h[3+j*ni+ni*nj] + 0.5*h[4+j*ni+ni*nj];
			h[ni-1+j*ni+ni*nj] = 2.5*h[ni-1+j*ni+ni*nj] - 2*h[ni-2+ni*j+ni*nj] + 0.5*h[ni-3+j*ni+ni*nj];
		}

		if(j=0)
		{
			u[i+ni+ni*nj] = 2.5*u[i+2*ni+ni*nj] - 2*u[i+3*ni+ni*nj] + 0.5*u[i+4*ni+ni*nj];
			u[i+(nj-1)*ni+ni*nj] = 2.5*u[i+(nj-2)*ni+ni*nj] - 2*u[i+(nj-3)*ni+ni*nj] + 0.5*u[i+(nj-4)*ni+ni*nj];
			v[i+ni+ni*nj] = 2.5*v[i+2*ni+ni*nj] - 2*v[i+3*ni+ni*nj] + 0.5*v[i+4*ni+ni*nj];
			v[i+(nj-1)*ni+ni*nj] = 2.5*v[i+(nj-2)*ni+ni*nj] - 2*v[i+(nj-3)*ni+ni*nj] + 0.5*v[i+(nj-4)*ni+ni*nj];
			h[i+ni+ni*nj] = 2.5*h[i+2*ni+ni*nj] - 2*h[i+3*ni+ni*nj] + 0.5*h[i+4*ni+ni*nj];
			h[i+(nj-1)*ni+ni*nj] = 2.5*h[i+(nj-2)*ni+ni*nj] - 2*h[i+(nj-3)*ni+ni*nj] + 0.5*h[i+(nj-4)*ni+ni*nj];
		}
	}
  
}

__global__ void update_parallel(struct params *p, float *b, float *u, float *v, float *h)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;


  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;



  if(i<p->ni && j<p->nj)
	{
            u[i+(nj-1)*ni]=u[i+(nj-1)*ni+ni*nj];
            v[i+(nj-1)*ni]=v[i+(nj-1)*ni+ni*nj];
	    h[i+(nj-1)*ni]=h[i+(nj-1)*ni+ni*nj];
	}
  
}
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void saxpy_parallel(int n, float alpha, float *x, float *y)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // except for special cases, the total number of threads in all blocks
  // adds up to more than the vector length n, so this conditional is
  // EXTREMELY important to avoid writing past the allocated memory for
  // the vector y.
  if (i<n)
    y[i] = alpha*x[i] + y[i];
}

/////////////////////////////////////
// kernel function (CPU)
/////////////////////////////////////
void saxpy_serial(int n, float alpha, float *x, float *y)
{
  int i;
  for (i=0; i<n; i++)
    y[i] = alpha*x[i] + y[i];
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int stepfunc()
{
 /////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
  {
    fprintf(stderr, "Sorry, no CUDA device fount");
    return 1;
  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  hipSetDevice(selectedDevice);
  checkErrors("initialisations");
  

  
  /////////////////////////////////////
  // (2) allocate memory on host (main CPU memory) and device,
  //     h_ denotes data residing on the host, d_ on device
  /////////////////////////////////////
  float *h_x = (float*)malloc(N*sizeof(float));
  float *h_y = (float*)malloc(N*sizeof(float));
  float *d_x;
  hipMalloc((void**)&d_x, N*sizeof(float));
  float *d_y;
  hipMalloc((void**)&d_y, N*sizeof(float));
  checkErrors("memory allocation");



  /////////////////////////////////////
  // (3) initialise data on the CPU
  /////////////////////////////////////
  int i;
  for (i=0; i<N; i++)
  {
    h_x[i] = 1.0f + i;
    h_y[i] = (float)(N-i+1);
  }



  /////////////////////////////////////
  // (4) copy data to device
  /////////////////////////////////////
  hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);
  checkErrors("copy data to device");



  /////////////////////////////////////
  // (5) perform computation on host (to enable result comparison later)
  /////////////////////////////////////
  saxpy_serial(N, 2.0f, h_x, h_y);



  /////////////////////////////////////
  // (6) perform computation on device
  //     - we use numThreadsPerBlock threads per block
  //     - the total number of blocks is obtained by rounding the
  //       vector length N up to the next multiple of numThreadsPerBlock
  /////////////////////////////////////
  int numBlocks = (N+numThreadsPerBlock-1) / numThreadsPerBlock;
  saxpy_parallel<<<numBlocks, numThreadsPerBlock>>>(N, 2.0, d_x, d_y);
  checkErrors("compute on device");



  /////////////////////////////////////
  // (7) read back result from device into temp vector
  /////////////////////////////////////
  float *h_z = (float*)malloc(N*sizeof(float));
  hipMemcpy(h_z, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");

  
  /////////////////////////////////////
  // (8) perform result comparison
  /////////////////////////////////////
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_y[i]-h_z[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");



  /////////////////////////////////////
  // (9) clean up, free memory
  /////////////////////////////////////
  free(h_x);
  free(h_y);
  free(h_z);
  hipFree(d_x);
  hipFree(d_y);
  return 0;

}

int cuinit(struct params **p, float **u, float **v, float **b, float **h,struct params **d_p, float **d_u, float **d_v, float **d_b, float **d_h)
{
/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
  {
    fprintf(stderr, "Sorry, no CUDA device fount");
    return 1;
  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  checkErrors("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
  float *adu,*adv,*adh,*adb;
  struct params *adp;

  hipMalloc((void**)&adu, ((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adv, ((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adh, ((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adb, ((*p)->ni)*sizeof(float));
  hipMalloc((void**)&adp, sizeof(struct params));
  checkErrors("memory allocation");

printf("ni is %d\n",(*p)->nj);

    *d_u=adu;
    *d_v=adv;
    *d_h=adh;
    *d_b=adb;
    *d_p=adp;


    hipMemcpy(*d_u, *u, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_v, *v, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_h, *h, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_b, *b, ((*p)->ni)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
     hipDeviceSynchronize();
	    printf("called initialiser\n");

	hipMemcpy(*u, *d_u, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(*v, *d_v, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(*b, *d_b, ((*p)->ni)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(*h, *d_h, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);




  return 0;



}


int cuprop(struct params **p, float **u, float **v, float **b, float **h,struct params **d_p, float **d_u, float **d_v, float **d_b, float **d_h)
{


printf("calling propagate solution\n");

    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);


     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    printf("called prop\n"); 
     hipDeviceSynchronize();
     boundary_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    printf("called boundary\n");  
     hipDeviceSynchronize();
     update_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    printf("called update\n"); 
    hipDeviceSynchronize();
 hipMemcpy(*u, *d_u, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*v, *d_v, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*b, *d_b, ((*p)->ni)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*h, *d_h, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);

  checkErrors("copy data from device");


 


}

int cufinish(struct params **p, float **u, float **v, float **b, float **h,struct params **d_p, float **d_u, float **d_v, float **d_b, float **d_h)
{
  

 hipMemcpy(*u, *d_u, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*v, *d_v, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*b, *d_b, ((*p)->ni)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*h, *d_h, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);

  checkErrors("copy data from device");


  hipFree(*d_p);

  hipFree(*d_u);
  hipFree(*d_v);
  hipFree(*d_h);
  hipFree(*d_b);


}
