#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv1a.cuh"

__device__ __host__
void bc_periodic1_temp2_hdv1a(real *wt, struct params *p,int i, int j, int f) {

                if(i==1 )                
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,6,j,f)];
                else if((i==((p->n[0]))) )                
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i-4,j,f)];
                else if(j==1  )                
                  wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,6,f)];
                else if((j==((p->n[1]))) )                
                  wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,j-4,f)];
}

__device__ __host__
void bc_periodic2_temp2_hdv1a(real *wt, struct params *p,int i, int j, int f) {


               if(i<1 && j<1)
                {
                  if(i==j)
                    //wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,6,f)];
                  else                  
                    //wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,6,j,f)];                                    
                }
                else if(i<1 && j>((p->n[1])-1))
                {
                  if(i==(j-(p->n[1])-1))                  
                    //wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,6,j,f)];                                     
                  else                  
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,j-6,f)];                                     
                }
                else if(i>((p->n[0])-1) && j<1)
                {
                  if((i-(p->n[0])+1)==j)                  
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i-5,j,f)];                                    
                  else                  
                   wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,4,f)];                                    
                }
                else if(i>((p->n[0])-1) && j>((p->n[1])-1))
                {
                  if(i==j)                  
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i,j-5,f)];                                    
                  else                  
                    wt[fencode_hdv1a(p,i,j,f)]=wt[fencode_hdv1a(p,i-5,j,f)];                                    
                }                       
                 
                




}



__global__ void hyperdifvisc1a_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
//int numBlocks = (ni*nj+numThreadsPerBlock-1) / numThreadsPerBlock;
  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);
  __shared__ real wts[512];
  __shared__ real wms[512];



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if( i<((p->n[0])) && j<((p->n[1])))
   {
     //tmp6 is tmp_nuI
     wtemp2[fencode_hdv1a(p,i+1,j+1,tmpnui)]=wtemp[fencode_hdv1a(p,i,j,tmp6)];

   }

   }
   __syncthreads();








 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv1a(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc1a(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim,int hand)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifvisc1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_hdv1a,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv1a,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv1a,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_hdv1a, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







