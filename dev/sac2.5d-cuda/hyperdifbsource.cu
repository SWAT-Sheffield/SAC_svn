#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdb.cuh"














__global__ void hyperdifbsource_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii,jj,ii1,ii0;
  real fip,fim1,tmp2,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real sb;
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
    for(int f=tmp1; f<=tmprhor; f++)	
        wtemp[fencode_hdb(p,i,j,f)]=0.0;

 __syncthreads();

  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  {
       wtemp[fencode_hdb(p,i,j,tmprhor)]=(wmod[fencode_hdb(p,i,j,rho)]+wmod[fencode_hdb(p,i+(field==0),j+(field==1),rho)])/2;
       wtemp[fencode_hdb(p,i,j,tmprhol)]=(wmod[fencode_hdb(p,i,j,rho)]+wmod[fencode_hdb(p,i-(field==0),j+(field==1),rho)])/2;

   }
__syncthreads();


  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  {
     wtemp[fencode_hdb(p,i,j,tmp1)]=wmod[fencode_hdb(p,i,j,mom1+field)]/wmod[fencode_hdb(p,i,j,rho)];
     wtemp[fencode_hdb(p,i,j,tmp2)]=grad1_hdb(wtemp,p,i,j,tmp1,0);
     wtemp[fencode_hdb(p,i,j,tmp3)]=grad1_hdb(wtemp,p,i,j,tmp1,1);
  }

__syncthreads();


 



  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
             //ii1=0
             //case i=k, ii0=l
                      //   ii0=k
           //   m=l
           //   sB=-1.d0
           //   j=k

             ii1=0;
             ii0=dim;
             m=field;
             jj=dim;
             sb=-1.0;

             

                     fip=wmod[fencode_hdb(p,i+(field==0),j+(field==1),rho)]*((field==0)*wtemp[fencode_hdb(p,i+(field==0),j+(field==1),tmp2)] + (field==1)*wtemp[fencode_hdb(p,i+(field==0),j+(field==1),tmp3)])*(wtemp[fencode_hdb(p,i+(field==0),j+(field==1),hdnur)]+wtemp[fencode_hdb(p,i+(field==0),j+(field==1),hdnul)])/4.0;




                     fim1=wmod[fencode_hdb(p,i-(field==0),j-(field==1),rho)]*((field==0)*wtemp[fencode_hdb(p,i-(field==0),j-(field==1),tmp2)] + (field==1)*wtemp[fencode_hdb(p,i-(field==0),j-(field==1),tmp3)])*(wtemp[fencode_hdb(p,i-(field==0),j-(field==1),hdnur)]+wtemp[fencode_hdb(p,i-(field==0),j-(field==1),hdnul)]);
                     
		     //dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]+(evalgrad1_hdb(fip, fim1, p,field))/(((p->dx[0])*(field==0))+(p->dx[1])*(field==1));
                      //dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+field)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]-(p->chyp)*grad2_hdb(wmod,p,i,i,mom1+field,dim);

//dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1)]-(p->chyp)*grad2_hdb(wmod,p,i,i,mom1+field,dim);

             ii1=1;
             ii0=field;
             m=dim;
             jj=field;
             sb=1.0;

		     //dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]+( wtemp[fencode_hdb(p,i,j,tmprhor)]*(wd[fencode_hdb(p,i,j,hdnur)]*grad1r_hdb(wtemp,p,i,j,tmp1,field))-wtemp[fencode_hdb(p,i,j,tmprhol)]*(wd[fencode_hdb(p,i,j,hdnul)]*grad1l_hdb(wtemp,p,i,j,tmp1,field)) )/(((p->dx[0])*(field==0))+(p->dx[1])*(field==1));

dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,field)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,field)]+(evalgrad1_hdb(fip, fim1, p,field))/(((p->dx[0])*(field==0))+(p->dx[1])*(field==1));



         /*    for(ii1=0; ii1<2; i++)
             {
		     if(ii1==0)
                     {
                        ii0=field;
                        ii=dim;
                     }
                     else
                     {
			ii=field;
                        ii0=dim;
                     }

		     if(ii==field)
		     {
		     ;//dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]+( wtemp[fencode_hdb(p,i,j,tmprhor)]*(wd[fencode_hdb(p,i,j,hdnur)]*grad1r_hdb(wtemp,p,i,j,tmp1,field))-wtemp[fencode_hdb(p,i,j,tmprhol)]*(wd[fencode_hdb(p,i,j,hdnul)]*grad1l_hdb(wtemp,p,i,j,tmp1,field)) )/(((p->dx[0])*(field==0))+(p->dx[1])*(field==1));
		     }
		     else
		     {

                     fip=wmod[fencode_hdb(p,i+(field==0),j+(field==1),rho)]*((field==0)*wtemp[fencode_hdb(p,i+(field==0),j+(field==1),tmp2)] + (field==1)*wtemp[fencode_hdb(p,i+(field==0),j+(field==1),tmp3)])*(wtemp[fencode_hdb(p,i+(field==0),j+(field==1),hdnur)]+wtemp[fencode_hdb(p,i+(field==0),j+(field==1),hdnul)])/4.0;




                     fim1=wmod[fencode_hdb(p,i-(field==0),j-(field==1),rho)]*((field==0)*wtemp[fencode_hdb(p,i-(field==0),j-(field==1),tmp2)] + (field==1)*wtemp[fencode_hdb(p,i-(field==0),j-(field==1),tmp3)])*(wtemp[fencode_hdb(p,i-(field==0),j-(field==1),hdnur)]+wtemp[fencode_hdb(p,i-(field==0),j-(field==1),hdnul)]);
                     
		     ;//dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]=dwn1[(NVAR*(p->n[0])*(p->n[1])*order)+fencode_hdb(p,i,j,mom1+ii0)]+(evalgrad1_hdb(fip, fim1, p,field))/(((p->dx[0])*(field==0))+(p->dx[1])*(field==1));
		     
}
             }*/


 
               
 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdb(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifbsource(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, int field, int dim)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifbsource_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order,*d_wtemp, field, dim);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







