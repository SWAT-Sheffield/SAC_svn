#include "hip/hip_runtime.h"


__device__ __host__
void computej_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad_MODID(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_MODID(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_MODID(wmod,p,i,j,b2,0);
 // dbxdy=grad_MODID(wmod,p,i,j,b1,1);

  wd[fencode_MODID(p,i,j,0)]=(grad_MODID(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_MODID(p,i,j,1)]=(grad_MODID(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_MODID(p,i,j,2)]=(grad_MODID(wmod,p,i,j,b2,0)-grad_MODID(wmod,p,i,j,b1,1))/(p->mu);
  
          #ifdef USE_SAC
	  wd[fencode_MODID(p,i,j,0)]+=(grad_MODID(wmod,p,i,j,b3b,1))/(p->mu);
	  wd[fencode_MODID(p,i,j,1)]+=(grad_MODID(wmod,p,i,j,b3b,0))/(p->mu);
	  wd[fencode_MODID(p,i,j,2)]+=(grad_MODID(wmod,p,i,j,b2b,0)-grad_MODID(wmod,p,i,j,b1b,1))/(p->mu);


         #endif

 
  //return ( status);
}

__device__ __host__
void computebdotv_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)];
//  wd[fencode_MODID(p,i,j,4)]=  wd[fencode_MODID(p,i,j,3)]+0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]);
        #ifdef USE_SAC

wd[fencode_MODID(p,i,j,bdotv)]=((wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b1b)])*wmod[fencode_MODID(p,i,j,mom1)]+(wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b2b)])*wmod[fencode_MODID(p,i,j,mom2)]+(wmod[fencode_MODID(p,i,j,b3)]+wmod[fencode_MODID(p,i,j,b3b)])*wmod[fencode_MODID(p,i,j,mom3)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
         #else
wd[fencode_MODID(p,i,j,bdotv)]=(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,mom3)])/wmod[fencode_MODID(p,i,j,rho)];
         #endif
 // return ( status);
}

__device__ __host__
void computedivb_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)];
//  wd[fencode_MODID(p,i,j,4)]=  wd[fencode_MODID(p,i,j,3)]+0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]);

wd[fencode_MODID(p,i,j,divb)]=grad_MODID(wmod,p,i,j,b1,0)+grad_MODID(wmod,p,i,j,b2,1);
        #ifdef USE_SAC
		wd[fencode_MODID(p,i,j,divb)]+=grad_MODID(wmod,p,i,j,b1b,0)+grad_MODID(wmod,p,i,j,b2b,1);
         #endif
 // return ( status);
}


__device__ __host__
void computept_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode_MODID(p,i,j,pressuret)]=(p->adiab)*pow(wmod[fencode_MODID(p,i,j,rho)],p->gamma);
#elif defined(USE_SAC)
  wd[fencode_MODID(p,i,j,pressuret)]=  wd[fencode_MODID(p,i,j,pressurek)]+0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)])+(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3)]);

  wd[fencode_MODID(p,i,j,ptb)]=  wd[fencode_MODID(p,i,j,pkb)]+0.5*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3b)]);


#else

 //real bsq=wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)];
  wd[fencode_MODID(p,i,j,pressuret)]=  wd[fencode_MODID(p,i,j,pressurek)]+0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]);

#endif



  if(wd[fencode_MODID(p,i,j,pressuret)]<0)
              wd[fencode_MODID(p,i,j,pressuret)]=0.001;


 // return ( status);
}
__device__ __host__
void computepk_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_MODID(p,i,j,pressurek)]=(p->adiab)*pow(wmod[fencode_MODID(p,i,j,rho)],p->gamma);
wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]);
wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]);
wd[fencode_MODID(p,i,j,vel3)]=wmod[fencode_MODID(p,i,j,mom3)]/(wmod[fencode_MODID(p,i,j,rho)]);
#elif defined(USE_SAC)

wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);
wd[fencode_MODID(p,i,j,vel3)]=wmod[fencode_MODID(p,i,j,mom3)]/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]);

 wd[fencode_MODID(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]- 0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,mom3)]*wmod[fencode_MODID(p,i,j,mom3)])-0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]) -(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3)]) );


wd[fencode_MODID(p,i,j,pkb)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energyb)]- 0.5*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3b)]) );

#else
wd[fencode_MODID(p,i,j,vel1)]=wmod[fencode_MODID(p,i,j,mom1)]/(wmod[fencode_MODID(p,i,j,rho)]);
wd[fencode_MODID(p,i,j,vel2)]=wmod[fencode_MODID(p,i,j,mom2)]/(wmod[fencode_MODID(p,i,j,rho)]);
wd[fencode_MODID(p,i,j,vel3)]=wmod[fencode_MODID(p,i,j,mom3)]/(wmod[fencode_MODID(p,i,j,rho)]);
  //real momsq=wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,mom3)]*wmod[fencode_MODID(p,i,j,mom3)];
  //real bsq=wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)];
  wd[fencode_MODID(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_MODID(p,i,j,energy)]- 0.5*(wmod[fencode_MODID(p,i,j,mom1)]*wmod[fencode_MODID(p,i,j,mom1)]+wmod[fencode_MODID(p,i,j,mom2)]*wmod[fencode_MODID(p,i,j,mom2)]+wmod[fencode_MODID(p,i,j,mom3)]*wmod[fencode_MODID(p,i,j,mom3)])/wmod[fencode_MODID(p,i,j,rho)]-0.5*(wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]) );


#endif






  if(wd[fencode_MODID(p,i,j,pressurek)]<0)
              wd[fencode_MODID(p,i,j,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{

  
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode_MODID(p,i,j,soundspeed)]=sqrt((p->adiab)/wmod[fencode_MODID(p,i,j,rho)]);
#elif defined(USE_SAC)
wd[fencode_MODID(p,i,j,soundspeed)]=sqrt((  (p->gamma))*(wd[fencode_MODID(p,i,j,pressuret)]+wd[fencode_MODID(p,i,j,ptb)])/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]   ));
wd[fencode_MODID(p,i,j,cfast)]=sqrt((   ( (wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)]) + (wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1b)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2b)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3b)]) +2.0*(wmod[fencode_MODID(p,i,j,b1b)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2b)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3b)]*wmod[fencode_MODID(p,i,j,b3)])    )/(wmod[fencode_MODID(p,i,j,rho)]+wmod[fencode_MODID(p,i,j,rhob)]))+(wd[fencode_MODID(p,i,j,soundspeed)]*wd[fencode_MODID(p,i,j,soundspeed)]));
#else
wd[fencode_MODID(p,i,j,soundspeed)]=sqrt(((p->gamma))*wd[fencode_MODID(p,i,j,pressuret)]/wmod[fencode_MODID(p,i,j,rho)]);
wd[fencode_MODID(p,i,j,cfast)]=sqrt(((wmod[fencode_MODID(p,i,j,b1)]*wmod[fencode_MODID(p,i,j,b1)]+wmod[fencode_MODID(p,i,j,b2)]*wmod[fencode_MODID(p,i,j,b2)]+wmod[fencode_MODID(p,i,j,b3)]*wmod[fencode_MODID(p,i,j,b3)])/wmod[fencode_MODID(p,i,j,rho)])+(wd[fencode_MODID(p,i,j,soundspeed)]*wd[fencode_MODID(p,i,j,soundspeed)]));
#endif



  
}

__device__ __host__
void computecmax_MODID(real *wmod,real *wd,struct params *p,int i,int j)
{
#ifdef ADIABHYDRO
       if(wd[fencode_MODID(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_MODID(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_MODID(p,i,j,soundspeed)]);
#else
       if(wd[fencode_MODID(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_MODID(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_MODID(p,i,j,soundspeed)]);
       if(wd[fencode_MODID(p,i,j,cfast)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_MODID(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_MODID(p,i,j,cfast)]);
#endif

}


