#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdm1.cuh"



__global__ void hyperdifmomsource3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
int shift=order*NVAR*(p->n[0])*(p->n[1]);
  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
  if(i<((p->n[0])) && j<((p->n[1])))
	{		               

dwn1[fencode_hdm1(p,i,j,energy)]=wtemp[fencode_hdm1(p,i,j,tmp6)]*wd[fencode_hdm1(p,i,j,hdnur)]*wtemp[fencode_hdm1(p,i,j,tmp8)]-wtemp[fencode_hdm1(p,i,j,tmp5)]*wd[fencode_hdm1(p,i,j,hdnul)]*wtemp[fencode_hdm1(p,i,j,tmp7)]/(rdx)/2;

dwn1[fencode_hdm1(p,i,j,mom1+ii0)]=(wtemp[fencode_hdm1(p,i,j,tmp3)]*wd[fencode_hdm1(p,i,j,hdnur)]*wtemp[fencode_hdm1(p,i,j,tmp8)]-wtemp[fencode_hdm1(p,i,j,tmp2)]*wd[fencode_hdm1(p,i,j,hdnul)]*wtemp[fencode_hdm1(p,i,j,tmp7)])/(rdx)/2;

                              wmod[fencode_hdm1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdm1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdm1(p,i,j,mom1+ii0)]; 
                             wmod[fencode_hdm1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdm1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdm1(p,i,j,energy)]; 


/*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,energy)]=(

(  wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm1(p,i,j,hdnur)]*grad1r_hdm1(wtemp,p,i,j,tmp1,dim)

-(wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm1(p,i,j,hdnul)]*grad1l_hdm1(wtemp,p,i,j,tmp1,dim))/(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1))/2;

dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+ii0)]=(wtemp[fencode_hdm1(p,i,j,tmp2)]*wd[fencode_hdm1(p,i,j,hdnur)]*grad1r_hdm1(wtemp,p,i,j,tmp1,dim)-wtemp[fencode_hdm1(p,i,j,tmp3)]*wd[fencode_hdm1(p,i,j,hdnul)]*grad1l_hdm1(wtemp,p,i,j,tmp1,dim))/(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1))/2;*/


/*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,energy)]=(

(  wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm1(p,i,j,hdnur)]*grad1r_hdm1(wtemp,p,i,j,tmp1,dim)

-(wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j+(dim==1),mom1+field)])*wd[fencode_hdm1(p,i,j,hdnul)]*grad1l_hdm1(wtemp,p,i,j,tmp1,dim))/2;

dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+ii0)]=(wtemp[fencode_hdm1(p,i,j,tmp2)]*wd[fencode_hdm1(p,i,j,hdnur)]*grad1r_hdm1(wtemp,p,i,j,tmp1,dim)-wtemp[fencode_hdm1(p,i,j,tmp3)]*wd[fencode_hdm1(p,i,j,hdnul)]*grad1l_hdm1(wtemp,p,i,j,tmp1,dim))/2;*/


   }
}
 __syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
          {

                  bc_periodic1_hdm1(dwn1,p,i,j,mom1+ii0);
                  bc_periodic1_hdm1(dwn1,p,i,j,energy);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
          {

                  bc_periodic2_hdm1(dwn1,p,i,j,mom1+ii0);
                  bc_periodic2_hdm1(dwn1,p,i,j,energy);
             }
}
                __syncthreads();*/



   
/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
			// if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                        if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hdm1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdm1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdm1(p,i,j,mom1+ii0)]; 
                             wmod[fencode_hdm1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdm1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdm1(p,i,j,energy)]; 

                         }
              //  }	
}
  __syncthreads();*/


  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
/*void checkErrors_hdm1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}
*/



__global__ void hyperdifmomsource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

     wtemp[fencode_hdm1(p,i,j,tmp8)]=grad1r_hdm1(wtemp,p,i,j,tmp4,dim);
     wtemp[fencode_hdm1(p,i,j,tmp7)]=grad1l_hdm1(wtemp,p,i,j,tmp4,dim);

   }

}
__syncthreads();  //can remove?



  
}


__global__ void hyperdifmomsource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
int shift=order*NVAR*(p->n[0])*(p->n[1]);
  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode_hdm1(p,i,j,f)]=0.0;

//dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,energy)]=0.0;
//dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdm1(p,i,j,mom1+ii0)]=0.0;
dwn1[fencode_hdm1(p,i,j,energy)]=0.0;
dwn1[fencode_hdm1(p,i,j,mom1+ii0)]=0.0;
   }


}
 __syncthreads();

//tmp2  rhor
//tmp3  rhol
//tmp1  mom+field/rho

//tmp4  rhoc

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if( i<((p->n[0])) && j<((p->n[1])))
  {

#ifdef USE_SAC


     wtemp[fencode_hdm1(p,i,j,tmp4)]=wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]/(wmod[(shift)+fencode_hdm1(p,i,j,rho)]+wmod[(shift)+fencode_hdm1(p,i,j,rhob)]);

#else
 

     wtemp[fencode_hdm1(p,i,j,tmp4)]=wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]/wmod[(shift)+fencode_hdm1(p,i,j,rho)];


#endif
  /*  wtemp[fencode_hdm1(p,i,j,tmp2)]=wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+field)];
    wtemp[fencode_hdm1(p,i,j,tmp3)]=wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j+(dim==1),mom1+field)];*/
   }

}
__syncthreads();




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

#ifdef USE_SAC
       wtemp[fencode_hdm1(p,i,j,tmp2)]=(wmod[(shift)+fencode_hdm1(p,i,j,rho)]+wmod[(shift)+fencode_hdm1(p,i,j,rhob)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),rho)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),rhob)])/2;
       wtemp[fencode_hdm1(p,i,j,tmp3)]=(wmod[(shift)+fencode_hdm1(p,i,j,rho)]+wmod[(shift)+fencode_hdm1(p,i,j,rhob)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),rho)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),rhob)])/2;



#else
       wtemp[fencode_hdm1(p,i,j,tmp2)]=(wmod[(shift)+fencode_hdm1(p,i,j,rho)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),rho)])/2;
       wtemp[fencode_hdm1(p,i,j,tmp3)]=(wmod[(shift)+fencode_hdm1(p,i,j,rho)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),rho)])/2;




#endif
  /*  wtemp[fencode_hdm1(p,i,j,tmp2)]=wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+field)];
    wtemp[fencode_hdm1(p,i,j,tmp3)]=wmod[(shift)+fencode_hdm1(p,i,j,mom1+field)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j+(dim==1),mom1+field)];*/
   }

}
__syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//if(i<((p->n[0])) && j<((p->n[1])))
if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {
     wtemp[fencode_hdm1(p,i,j,tmp5)]=(wmod[(shift)+fencode_hdm1(p,i,j,mom1+ii0)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),mom1+ii0)])/2;
     wtemp[fencode_hdm1(p,i,j,tmp6)]=(wmod[(shift)+fencode_hdm1(p,i,j,mom1+ii0)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+ii0)])/2;
   }

}
__syncthreads();



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdm1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsource1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{


 dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifmomsource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();

}







