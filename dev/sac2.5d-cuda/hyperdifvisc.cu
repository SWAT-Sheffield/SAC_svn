#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv.cuh"

__global__ void hyperdifvisc_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
int bfac2= (field==rho || field>mom2);
int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {
        for(int f=tmp1; f<=tmp9; f++)
                 wtemp[fencode_hdv(p,i,j,f)]=0;


        //temp value for viscosity


#ifdef USE_SAC
     if(field !=energy)
        wtemp[fencode_hdv(p,i,j,tmp1)]=wmod[fencode_hdv(p,i,j,field)+shift]/(bfac2+bfac3*(wmod[fencode_hdv(p,i,j,rho)+shift] ));

     else
        wtemp[fencode_hdv(p,i,j,tmp1)]=wmod[fencode_hdv(p,i,j,energy)+shift]-0.5*(wmod[fencode_hdv(p,i,j,b1)+shift]*wmod[fencode_hdv(p,i,j,b1)+shift]+wmod[fencode_hdv(p,i,j,b2)+shift]*wmod[fencode_hdv(p,i,j,b2)+shift])+(wmod[fencode_hdv(p,i,j,mom1)+shift]*wmod[fencode_hdv(p,i,j,mom1)+shift]+wmod[fencode_hdv(p,i,j,mom2)+shift]*wmod[fencode_hdv(p,i,j,mom2)+shift])/(wmod[fencode_hdv(p,i,j,rho)+shift] );

#else
     if(field !=energy)
        wtemp[fencode_hdv(p,i,j,tmp1)]=wmod[fencode_hdv(p,i,j,field)+shift]/(bfac2+bfac3*(wmod[fencode_hdv(p,i,j,rho)+shift] ));

     else
        wtemp[fencode_hdv(p,i,j,tmp1)]=wmod[fencode_hdv(p,i,j,energy)+shift]-0.5*(wmod[fencode_hdv(p,i,j,b1)+shift]*wmod[fencode_hdv(p,i,j,b1)+shift]+wmod[fencode_hdv(p,i,j,b2)+shift]*wmod[fencode_hdv(p,i,j,b2)+shift])+(wmod[fencode_hdv(p,i,j,mom1)+shift]*wmod[fencode_hdv(p,i,j,mom1)+shift]+wmod[fencode_hdv(p,i,j,mom2)+shift]*wmod[fencode_hdv(p,i,j,mom2)+shift])/(wmod[fencode_hdv(p,i,j,rho)+shift] );
        wd[fencode_hdv(p,i,j,hdnur)]=0;
        wd[fencode_hdv(p,i,j,hdnul)]=0;
#endif
   }

}
   __syncthreads();

/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))

                  bc_periodic1_hdv(wtemp,p,i,j,tmp1);

}
                __syncthreads();*/

/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
                  //bc_cont_cd1(dwn1,p,i,j,f1+fid);
                  bc_periodic2_hdv(wtemp,p,i,j,tmp1);
}
                __syncthreads();*/

   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l




   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
 
   //if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))
   if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))            
   { 
     if(hand==0)
     {
           wtemp[fencode_hdv(p,i,j,tmp2)]=fabs(3.0*(wtemp[fencode_hdv(p,i+(dim==0),j+(dim==1),tmp1)] - wtemp[fencode_hdv(p,i,j,tmp1)] ) - (wtemp[fencode_hdv(p,i+2*(dim==0),j+2*(dim==1),tmp1)] - wtemp[fencode_hdv(p,i-(dim==0),j-(dim==1),tmp1)]    ));

           wtemp[fencode_hdv(p,i,j,tmp3)]=fabs((wtemp[fencode_hdv(p,i+(dim==0),j+(dim==1),tmp1)] - wtemp[fencode_hdv(p,i,j,tmp1)] ));
     }
     else
     {
           wtemp[fencode_hdv(p,i,j,tmp2)]=fabs(3.0*(wtemp[fencode_hdv(p,i,j,tmp1)] - wtemp[fencode_hdv(p,i-(dim==0),j-(dim==0),tmp1)]) - (wtemp[fencode_hdv(p,i+(dim==0),j+(dim==1),tmp1)] - wtemp[fencode_hdv(p,i-2*(dim==0),j-2*(dim==1),tmp1)]    ));
           wtemp[fencode_hdv(p,i,j,tmp3)]=fabs((wtemp[fencode_hdv(p,i,j,tmp1)] - wtemp[fencode_hdv(p,i-(dim==0),j-(dim==1),tmp1)] ));
     }
   }
}
   __syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv(wtemp,p,i,j,tmp2);
                  bc_periodic1_hdv(wtemp,p,i,j,tmp3);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv(wtemp,p,i,j,tmp2);
                  bc_periodic2_hdv(wtemp,p,i,j,tmp3);
             }
}
                __syncthreads();*/



  //compute md3r and md1r
//tmp4    md3r
//tmp5    md1r
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))            
   {
         maxt=0;
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp[fencode_hdv(p,i+is,j+js,tmp2)]>maxt)
                         maxt=wtemp[fencode_hdv(p,i+is,j+js,tmp2)];

                }
          wtemp[fencode_hdv(p,i,j,tmp4)]=maxt;

         maxt=0;
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp[fencode_hdv(p,i+is,j+js,tmp3)]>maxt)
                        maxt=wtemp[fencode_hdv(p,i+is,j+js,tmp3)];

                }
          wtemp[fencode_hdv(p,i,j,tmp5)]=maxt;
   }
}
   __syncthreads();





 /*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv(wtemp,p,i,j,tmp4);
                  bc_periodic1_hdv(wtemp,p,i,j,tmp5);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv(wtemp,p,i,j,tmp4);
                  bc_periodic2_hdv(wtemp,p,i,j,tmp5);
             }
}
                __syncthreads();*/



   p->maxviscoef=0;


    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


   if(i<((p->n[0])) && j<((p->n[1])))
   {
     if(wtemp[fencode_hdv(p,i,j,tmp5)]>0)
	wd[fencode_hdv(p,i,j,hdnur+hand)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp[field])*wtemp[fencode_hdv(p,i,j,tmp4)]/wtemp[fencode_hdv(p,i,j,tmp5)];


     else
        wd[fencode_hdv(p,i,j,hdnur+hand)]=0;


   }
}
 __syncthreads();



 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, int field, int dim,int hand)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifvisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order, *d_wtemp, field, dim,hand);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_hdv,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_hdv, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







