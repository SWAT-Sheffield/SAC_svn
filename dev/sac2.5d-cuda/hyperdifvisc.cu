#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv.cuh"

__global__ void hyperdifvisc_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);


    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {
        for(int f=tmp1; f<=tmp9; f++)
                 wtemp[fencode_hdv(p,i,j,f)]=0;


        //temp value for viscosity

#ifdef USE_SAC
        wtemp[fencode_hdv(p,i,j,tmp1)]=wmod[fencode_hdv(p,i,j,field)+order*NVAR*(p->n[0])*(p->n[1])]/((field==rho || field>mom3)+(field>rho && field<energy)*(wmod[fencode_hdv(p,i,j,rho)+order*NVAR*(p->n[0])*(p->n[1])]+wmod[fencode_hdv(p,i,j,rhob)]+order*NVAR*(p->n[0])*(p->n[1])));
        if(field=rho)
           wtemp[fencode_hdv(p,i,j,tmp1)]+=wmod[fencode_hdv(p,i,j,rhob)+order*NVAR*(p->n[0])*(p->n[1])];

       if(field=b1 || field==b2)
           wtemp[fencode_hdv(p,i,j,tmp1)]+=wmod[fencode_hdv(p,i,j,field+5)+order*NVAR*(p->n[0])*(p->n[1])];
#else
        wtemp[fencode_hdv(p,i,j,tmp1)]=wmod[fencode_hdv(p,i,j,field)+order*NVAR*(p->n[0])*(p->n[1])]/( (field==rho || field>mom3)+(field>rho && field<energy)*wmod[fencode_hdv(p,i,j,rho)+order*NVAR*(p->n[0])*(p->n[1])] );
#endif
        wd[fencode_hdv(p,i,j,hdnur)]=0;
        wd[fencode_hdv(p,i,j,hdnul)]=0;
   }

   __syncthreads();


   //boundaries
     if(i<((p->n[0])) && j<((p->n[1])))            
               {                                                      
                if(dim==0)
                {
		        if(i==0 )
		          wtemp[fencode_hdv(p,i,j,tmp1)]=wtemp[fencode_hdv(p,4,j,tmp1)];
		        if( (i==((p->n[0])-1)) )
		          wtemp[fencode_hdv(p,i,j,tmp1)]=wtemp[fencode_hdv(p,((p->n[0])-4),j,tmp1)];
                }

                if(dim==1)
                {
		        if(j==0 )
		          wtemp[fencode_hdv(p,i,j,tmp1)]=wtemp[fencode_hdv(p,i,4,tmp1)];
		        if( (j==((p->n[1])-1)) )
		          wtemp[fencode_hdv(p,i,j,tmp1)]=wtemp[fencode_hdv(p,i,((p->n[1])-4),tmp1)];                                  
                }
               }
               
   __syncthreads();

   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l


   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r
 
   if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))            
   { 
           wtemp[fencode_hdv(p,i,j,tmp2)]=fabs(3.0*(wtemp[fencode_hdv(p,i+(dim==0),j+(dim==1),tmp1)] - wtemp[fencode_hdv(p,i,j,tmp1)] ) - (wtemp[fencode_hdv(p,i+2*(dim==0),j+2*(dim==1),tmp1)] - wtemp[fencode_hdv(p,i-(dim==0),j-(dim==1),tmp1)]    ));



           wtemp[fencode_hdv(p,i,j,tmp3)]=fabs((wtemp[fencode_hdv(p,i+(dim==0),j+(dim==1),tmp1)] - wtemp[fencode_hdv(p,i,j,tmp1)] ));
   }
   __syncthreads();



  //compute md3r and md1r
//tmp4    md3r
//tmp5    md1r
   if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))            
   {
         maxt=0;
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp[fencode_hdv(p,i+is,j+js,tmp2)]>maxt)
                         wtemp[fencode_hdv(p,i+is,j+js,tmp2)]=maxt;

                }
          wtemp[fencode_hdv(p,i,j,tmp4)]=maxt;

         maxt=0;
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp[fencode_hdv(p,i+is,j+js,tmp3)]>maxt)
                        maxt=wtemp[fencode_hdv(p,i+is,j+js,tmp3)];

                }
          wtemp[fencode_hdv(p,i,j,tmp5)]=maxt;
   }



  //compute d3l and d1l
//tmp6    d3l
//tmp7    d1l






   if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))            
   { 
           wtemp[fencode_hdv(p,i,j,tmp6)]=fabs(3.0*(wtemp[fencode_hdv(p,i,j,tmp1)] - wtemp[fencode_hdv(p,i-(dim==0),j-(dim==0),tmp1)] - wtemp[fencode_hdv(p,i+(dim==0),j+(dim==1),tmp1)] - wtemp[fencode_hdv(p,i-2*(dim==0),j-2*(dim==1),tmp1)]    ));
           wtemp[fencode_hdv(p,i,j,tmp7)]=fabs((wtemp[fencode_hdv(p,i,j,tmp1)] - wtemp[fencode_hdv(p,i-(dim==0),j-(dim==1),tmp1)] ));
   }
   __syncthreads();



  //compute md3l and md1l
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l

   if(i>1 && j>1 && i<((p->n[0])-1) && j<((p->n[1])-1))            
   {
         maxt=0;
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp[fencode_hdv(p,i+is,j+js,tmp6)]>maxt)
                         maxt=wtemp[fencode_hdv(p,i+is,j+js,tmp6)];

                }
          wtemp[fencode_hdv(p,i,j,tmp8)]=maxt;

         maxt=0;
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp[fencode_hdv(p,i+is,j+js,tmp7)]>maxt)
                        maxt=wtemp[fencode_hdv(p,i+is,j+js,tmp7)];

                }
          wtemp[fencode_hdv(p,i,j,tmp9)]=maxt;
   }
 __syncthreads();

   p->maxviscoef=0;
    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r

   if(i<((p->n[0])) && j<((p->n[1])))
   {
     if(wtemp[fencode_hdv(p,i,j,tmp5)]>0)
	wd[fencode_hdv(p,i,j,hdnur)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp)*wtemp[fencode_hdv(p,i,j,tmp4)]/wtemp[fencode_hdv(p,i,j,tmp5)];
     else
        wd[fencode_hdv(p,i,j,hdnur)]=0;
   }
 __syncthreads();
   if(i<((p->n[0])) && j<((p->n[1])))
   {
       if(wd[fencode_hdv(p,i,j,hdnur)]>(p->maxviscoef))
          p->maxviscoef=wd[fencode_hdv(p,i,j,hdnur)];
   }

//tmp8    md3l
//tmp9    md1l
   if(i<((p->n[0])) && j<((p->n[1])))
   {
     if(wtemp[fencode_hdv(p,i,j,tmp9)]>0)
	wd[fencode_hdv(p,i,j,hdnul)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp)*wtemp[fencode_hdv(p,i,j,tmp8)]/wtemp[fencode_hdv(p,i,j,tmp9)];
     else
        wd[fencode_hdv(p,i,j,hdnul)]=0;
   }
 __syncthreads();
   if(i<((p->n[0])) && j<((p->n[1])))
   {
       if(wd[fencode_hdv(p,i,j,hdnul)]>(p->maxviscoef))
          p->maxviscoef=wd[fencode_hdv(p,i,j,hdnul)];
   }
  __syncthreads();
 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order, real **d_wtemp, int field, int dim)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     hyperdifvisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order, *d_wtemp, field, dim);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_hdv,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_hdv,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_hdv, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







