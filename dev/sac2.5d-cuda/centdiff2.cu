#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd2.cuh"
#include "dervfields_cd2.cuh"

__device__ __host__
real fluxe2(real *dw, real *wd, real *w, struct params *p,int *ii, int dir) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


        #if defined USE_SAC  || defined USE_SAC_3D
computept3_cd2(w,wd,p,ii);



      		flux= -wd[fencode3_cd2(p,ii,ptb)]*grad3d_cd2(wd,p,ii,vel1+dir,dir);
               flux += +w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel1,0)+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel1+1,1);
         #endif


        #if defined USE_SAC_3D
               flux += +w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel3,0);
        #endif

  return flux;


  //return ( ddc1-ddc2);
}



__device__ __host__
int divflux_cd2(real *dw, real *wd, real *w, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  dw[fencode3_cd2(p,ii,field)]= grad3d_cd2(wd,p,ii,flux,dir);//+grad_cd2(wd,p,ii,f2,1); 


 #ifdef USE_SAC

  //commented out to test against vac
 /* if(field==energy)
  {    
     dw[fencode3_cd2(p,ii,field)]+=fluxe2(dw, wd, w, p,ix, iy,dir)+w[fencode3_cd2(p,ii,rho)]*((p->g[dir])*w[fencode3_cd2(p,ii,mom1+dir)]    )/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
   }*/


 #endif
  return ( status);
}


__device__ __host__
int addenergyterms_cd2(real *dw, real *wd, real *w, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  //dw[fencode3_cd2(p,ii,field)]= grad_cd2(wd,p,ii,flux,dir);//+grad_cd2(wd,p,ii,f2,1); 


 #if defined USE_SAC  ||  defined USE_SAC_3D

  
  if(field==energy)
  {    
     computept3_cd2(w,wd,p,ii);
     dw[fencode3_cd2(p,ii,field)]=fluxe2(dw, wd, w, p,ii,dir)+w[fencode3_cd2(p,ii,rho)]*((p->g[dir])*w[fencode3_cd2(p,ii,mom1+dir)]    )/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
   }


 #endif
  return ( status);
}



__device__ __host__
real transportflux_cd2 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
   //use versions with velocity less ops may improve performance
    switch(direction)
  {
     case 0:
        #if defined USE_SAC  || defined USE_SAC_3D
     flux= w[fencode3_cd2(p,ii,mom1)]*w[fencode3_cd2(p,ii,field)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
     //flux= w[fencode3_cd2(p,ii,mom1)]*w[fencode3_cd2(p,ii,field)]/w[fencode3_cd2(p,ii,rho)];

        #else
     flux= w[fencode3_cd2(p,ii,mom1)]*w[fencode3_cd2(p,ii,field)]/w[fencode3_cd2(p,ii,rho)];

        #endif
     break;
     case 1:
        #if defined USE_SAC  || defined USE_SAC_3D
      flux= w[fencode3_cd2(p,ii,mom2)]*w[fencode3_cd2(p,ii,field)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
    // flux= w[fencode3_cd2(p,ii,mom2)]*w[fencode3_cd2(p,ii,field)]/w[fencode3_cd2(p,ii,rho)];

        #else
     //flux= wd[fencode3_cd2(p,ii,vel2)]*w[fencode3_cd2(p,ii,field)];
     flux= w[fencode3_cd2(p,ii,mom2)]*w[fencode3_cd2(p,ii,field)]/w[fencode3_cd2(p,ii,rho)];

        #endif
     break;
        #if  defined USE_SAC_3D
     case 2:
     flux= w[fencode3_cd2(p,ii,mom3)]*w[fencode3_cd2(p,ii,field)]/w[fencode3_cd2(p,ii,rho)];
     //flux= wd[fencode3_cd2(p,ii,vel3)]*w[fencode3_cd2(p,ii,field)];
     break;
     #endif
   }
  return flux;


  //return ( ddc1-ddc2);
}




__device__ __host__
real fluxb1(real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
  real ddcx=0,ddcy=0;

   real flux=0;

    switch(field)
    {
      case b1:
      //if(direction !=0)
        #if defined USE_SAC  || defined USE_SAC_3D

  flux= -(w[fencode3_cd2(p,ii,field+direction)]+w[fencode3_cd2(p,ii,field+(NDIM+2)+direction)])*w[fencode3_cd2(p,ii,mom1)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

flux+= (w[fencode3_cd2(p,ii,field+(NDIM+2))])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
         #endif

       break;

      case b2:
      //if(direction !=1)
        #if defined USE_SAC  || defined USE_SAC_3D
		flux= -(w[fencode3_cd2(p,ii,b1+direction)]+w[fencode3_cd2(p,ii,b1+(NDIM+2)+direction)])*w[fencode3_cd2(p,ii,mom2)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

               flux+= (w[fencode3_cd2(p,ii,field+(NDIM+2))])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
         #endif 
       break;
        #if defined USE_SAC_3D
      case b3:
      //if(direction !=2)

		flux= -(w[fencode3_cd2(p,ii,b1+direction)]+w[fencode3_cd2(p,ii,b1+(NDIM+2)+direction)])*w[fencode3_cd2(p,ii,mom2)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

               flux+= (w[fencode3_cd2(p,ii,field+(NDIM+2))])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

       break;
         #endif
     }


  return flux;
}



__device__ __host__
real fluxe1(real *dw, real *wd, real *w, struct params *p,int *ii, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
  real ddcx=0,ddcy=0;

   real flux=0;

computept3_cd2(w,wd,p,ii);

        #if defined USE_SAC || defined USE_SAC_3D

flux= -w[fencode3_cd2(p,ii,b1+direction)]*wd[fencode3_cd2(p,ii,bdotv)]+(w[fencode3_cd2(p,ii,mom1+direction)]*wd[fencode3_cd2(p,ii,energyb)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]));

flux += w[fencode3_cd2(p,ii,mom1+direction)]*(wd[fencode3_cd2(p,ii,pressuret)]+wd[fencode3_cd2(p,ii,ptb)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

flux -= w[fencode3_cd2(p,ii,b1b+direction)]*(w[fencode3_cd2(p,ii,b1)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2)]*w[fencode3_cd2(p,ii,mom2)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)])
            - w[fencode3_cd2(p,ii,b1+direction)]*(w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,mom2)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

         #endif

#ifdef USE_SAC_3D

flux -= w[fencode3_cd2(p,ii,b1b+direction)]*(w[fencode3_cd2(p,ii,b3)]*w[fencode3_cd2(p,ii,mom3)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)])
            - w[fencode3_cd2(p,ii,b1+direction)]*(w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,mom3)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

#endif

  return flux;

}








__device__ __host__
int computefluxe(real *dw, real *wd, real *w, struct params *p,int *ii,int direction) {

  int field;//, direction;
  int status=0;

         #if defined USE_SAC  || defined USE_SAC_3D
	     wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,energy,direction)+fluxe1(dw,wd,w,p,ii,direction);
         #endif

        
  return ( status);
}

__device__ __host__
int computefluxb (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {


  int status=0;


     switch(field)
     {
       case b1 :
         #if defined USE_SAC  || defined USE_SAC_3D
      if(direction==0)
wd[fencode3_cd2(p,ii,flux)]= 0.0;
      else
wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,field,direction)+fluxb1(dw,wd,w,p,ii,field,direction);
         #endif

       break;

       case b2 :
         #if defined USE_SAC  || defined USE_SAC_3D
      if(direction==1)
wd[fencode3_cd2(p,ii,flux)]= 0.0;
else
wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,field,direction)+fluxb1(dw,wd,w,p,ii,field,direction);
         #endif

       break;

 #ifdef USE_SAC_3D
       case b3 :

      if(direction==2)
wd[fencode3_cd2(p,ii,flux)]= 0.0;
else
wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,field,direction)+fluxb1(dw,wd,w,p,ii,field,direction);


       break;
  #endif

    }
   
    
  return ( status);
}






//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux_cd2 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case energy:
      computefluxe(dw,wd,w,p,ii,dir);
      
      // add the following terms for SAC
      // del((b bb+ bb b).v)+ptb del v - bb bb del v
     break;
     case b1:
      computefluxb(dw,wd,w,p,ii,field,dir);
     break;
     case b2:
       computefluxb(dw,wd,w,p,ii,field,dir);
     break;
#ifdef USE_SAC_3D
     case b3:
      computefluxb(dw,wd,w,p,ii,field,dir);
     break;
#endif
  }
  //return ( status);
}



__global__ void centdiff2a_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  

   fid=0;





   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

			//if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                          #ifdef USE_SAC_3D
       				if(ii[0]<((p->n[0])-2) && ii[0]>1 && ii[1]>1 && ii[1]<((p->n[1])-2) && ii[2]>1 && ii[2]<((p->n[2])-2))
     			  #else
       				if(ii[0]<((p->n[0]))-2 && ii[0]>1  && ii[1]>1 && ii[1]<((p->n[1])-1))
     			  #endif
                                divflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 


}
__syncthreads();
                        






   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

                        switch(dir)
                        {
                         case 0:

                         //if(i<(ni)  && j >1 &&  j<(nj-2))
                         #ifdef USE_SAC_3D
       				if(ii[0]<((p->n[0]))  && ii[1]>1 && ii[1]<((p->n[1])-2) && ii[2]>1 && ii[2]<((p->n[2])-2))
     			  #else
       				if(ii[0]<((p->n[0]))   && ii[1]>1 && ii[1]<((p->n[1])-2))
     			  #endif
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)]; 
                         break;
                         case 1:
                         #ifdef USE_SAC_3D
       				if(ii[0]>1 && ii[0]<((p->n[0])-2)  &&  ii[1]<((p->n[1])) && ii[2]>1 && ii[2]<((p->n[2])-2))
     			  #else
       				if(ii[0]>1 && ii[0]<((p->n[0])-2)   && ii[1]<((p->n[1])) )
     			  #endif
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)];
                         break;
                         #ifdef USE_SAC_3D
                         case 2:

                         //if(i>1 &&  i<(ni-2) && j<(nj))
      			if(ii[0]>1 && ii[0]<((p->n[0])-2)  && ii[1]>1 && ii[1]<((p->n[1])-2)  && ii[2]<((p->n[2])))
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)];
                         break;
                         #endif
                        }


}
__syncthreads(); 


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif


     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                            dwn1[fencode3_cd2(p,ii,f)]=0.0;


			//if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
     #ifdef USE_SAC_3D
       if(ii[0]<((p->n[0])-2) && ii[1]<((p->n[1])-2) && ii[2]<((p->n[2])-2)     && ii[0]>1    &&  ii[1]>1   && ii[2]>1   )
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                                addenergyterms_cd2(dwn1,wd,wmod+ordero*NVAR*dimp,p,ii,f,dir); 

}
__syncthreads();




   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

                        switch(dir)
                        {
                         case 0:

                         //if(i<(ni)  && j >1 &&  j<(nj-2))
			     #ifdef USE_SAC
				   if(ii[0]<ni && ii[1] >1 && ii[1]<(nj-2))
			     #endif
			     #ifdef USE_SAC_3D
				  if(ii[0]<ni && ii[1] >1 && ii[2] >1  && ii[1]<(nj-2) && ii[2]<(nk-2))
			     #endif   
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)]; 
                         break;
                         case 1:

                         //if(i>1 &&  i<(ni-2) && j<(nj))
			     #ifdef USE_SAC
				   if(ii[1]<nj && ii[0] >1 && ii[0]<(ni-2))
			     #endif
			     #ifdef USE_SAC_3D
				  if(ii[1]<nj && ii[0] >1 && ii[2] >1  && ii[0]<(ni-2) && ii[2]<(nk-2))
			     #endif
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)];
                         break;
                   #ifdef USE_SAC_3D
                         case 2:
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                           if(ii[2]<nk && ii[0] >1 && ii[1] >1  && ii[1]<(nj-2) && ii[0]<(ni-2))
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)];
                         break;
                   #endif
                        }


}
__syncthreads(); 

                         
}





__global__ void centdiff2_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  

   fid=0;
   



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd2(p,ii,f)]=0.0;

                               wd[fencode3_cd2(p,ii,flux)]=0.0;

                        }

}
  __syncthreads();   


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif


//1. 11/1/11 could swap cases below
                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-1) && ii[2]>1 && ii[2]<(p->n[2]-1))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-1))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-1) && ii[2]>1 && ii[2]<(p->n[2]-1))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-1))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-1) && ii[1]>1 && ii[1]<(p->n[1]-1))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 
                         break;
                         #endif
                        }
              //  }
                        //might need to set boundaries correctly
 
}
__syncthreads();                        






                         
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff2(struct params **p, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt, int field,int dir)
{

    dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
   //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
   // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);


     centdiff2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();

     centdiff2a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();

     // hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

     //checkErrors("copy data from device");

}


