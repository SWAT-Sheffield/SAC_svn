#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd2.cuh"

__device__ __host__
real transportflux_cd2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
      flux= wd[fencode_cd2(p,ix,iy,vel1)]*w[fencode_cd2(p,ix,iy,field)];
     break;
     case 1:
      flux= wd[fencode_cd2(p,ix,iy,vel2)]*w[fencode_cd2(p,ix,iy,field)];
     break;
     /*case 2:
      flux= wd[fencode_cd2(p,ix,iy,vel3)]*w[fencode_cd2(p,ix,iy,field)];
     break;*/
   }
  return flux;


  //return ( ddc1-ddc2);
}




__device__ __host__
real fluxb1(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

    switch(field)
    {
      case b1:


      //if(direction !=0)
        #ifdef USE_SAC
      		//flux= -(w[fencode_cd2(p,ix,iy,field)]+w[fencode_cd2(p,ix,iy,field+5)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
               flux= -(w[fencode_cd2(p,ix,iy,field)]+w[fencode_cd2(p,ix,iy,field+4)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
         #endif
        #ifdef USE_VAC
      		//flux= -w[fencode_cd2(p,ix,iy,field)]*w[fencode_cd2(p,ix,iy,mom1+direction)]/w[fencode_cd2(p,ix,iy,rho)];
                flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*w[fencode_cd2(p,ix,iy,mom1)]/w[fencode_cd2(p,ix,iy,rho)];
         #endif
       break;

      case b2:
      //if(direction !=1)
        #ifdef USE_SAC
      		//flux= -(w[fencode_cd2(p,ix,iy,field)]+w[fencode_cd2(p,ix,iy,field+5)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
		flux= -(w[fencode_cd2(p,ix,iy,field)]+w[fencode_cd2(p,ix,iy,field+4)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
         #endif
        #ifdef USE_VAC
      		//flux= -w[fencode_cd2(p,ix,iy,field)]*w[fencode_cd2(p,ix,iy,mom1+direction)]/w[fencode_cd2(p,ix,iy,rho)];
                flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*w[fencode_cd2(p,ix,iy,mom2)]/w[fencode_cd2(p,ix,iy,rho)];

         #endif
       break;

     /* case b3:
      if(direction !=2)
        #ifdef USE_SAC
      		flux= -(w[fencode_cd2(p,ix,iy,field)]+w[fencode_cd2(p,ix,iy,field+5)])*w[fencode_cd2(p,ix,iy,mom1+direction)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);
         #endif
        #ifdef USE_VAC
      		flux= -w[fencode_cd2(p,ix,iy,field)]*w[fencode_cd2(p,ix,iy,mom1+direction)]/w[fencode_cd2(p,ix,iy,rho)];
         #endif
       break;*/

     }


  return flux;
}



__device__ __host__
real fluxe1(real *dw, real *wd, real *w, struct params *p,int ix, int iy, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


        #ifdef USE_SAC
      		flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*wd[fencode_cd2(p,ix,iy,bdotv)]+(w[fencode_cd2(p,ix,iy,mom1+direction)]*wd[fencode_cd2(p,ix,iy,pressuret)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]))+(w[fencode_cd2(p,ix,iy,mom1+direction)]*wd[fencode_cd2(p,ix,iy,energyb)]/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]));

flux -= w[fencode_cd2(p,ix,iy,b1b+direction)]*(w[fencode_cd2(p,ix,iy,b1)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,b2)]*w[fencode_cd2(p,ix,iy,mom2)])/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)])
            - w[fencode_cd2(p,ix,iy,b1+direction)]*(w[fencode_cd2(p,ix,iy,b1b)]*w[fencode_cd2(p,ix,iy,mom1)]+w[fencode_cd2(p,ix,iy,b2b)]*w[fencode_cd2(p,ix,iy,mom2)])/(w[fencode_cd2(p,ix,iy,rho)]+w[fencode_cd2(p,ix,iy,rhob)]);

        


         #endif
        #ifdef USE_VAC
      		flux= -w[fencode_cd2(p,ix,iy,b1+direction)]*wd[fencode_cd2(p,ix,iy,bdotv)]+(w[fencode_cd2(p,ix,iy,mom1+direction)]*wd[fencode_cd2(p,ix,iy,pressuret)]/w[fencode_cd2(p,ix,iy,rho)]);
               
         #endif

  return flux;


  //return ( ddc1-ddc2);
}

__device__ __host__
real fluxe2(real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


        #ifdef USE_SAC
      		flux= wd[fencode_cd2(p,ix,iy,ptb)]*(grad_cd2(wd,p,ix,iy,vel1,0)+grad_cd2(wd,p,ix,iy,vel2,1));
                flux      +=w[fencode_cd2(p,ix,iy,b1b)]*([fencode_cd2(p,ix,iy,b1b)]*grad_cd2(wd,p,ix,iy,vel1,0)+[fencode_cd2(p,ix,iy,b2b)]*grad_cd2(wd,p,ix,iy,vel2,1)) +w[fencode_cd2(p,ix,iy,b2b)]*([fencode_cd2(p,ix,iy,b1b)]*grad_cd2(wd,p,ix,iy,vel1,0)+[fencode_cd2(p,ix,iy,b2b)]*grad_cd2(wd,p,ix,iy,vel2,1));  
         #endif

  return flux;


  //return ( ddc1-ddc2);
}








__device__ __host__
int computefluxe(real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int field, direction;
  int status=0;
  for(direction=0;direction<2;direction++)
         #ifdef USE_SAC
	     wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,energy,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,energyb,direction)+fluxe1(dw,wd,w,p,ix,iy,direction);
         #endif
         #ifdef USE_VAC
             wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,energy,direction)+fluxe1(dw,wd,w,p,ix,iy,direction);
         #endif
        
  return ( status);
}

__device__ __host__
int computefluxb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  int direction;
  int status=0;
  for(direction=0;direction<2;direction++)
  {

     switch(field)
     {
       case b1 :
         #ifdef USE_SAC
      if(direction==0)
     //wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,field+5,direction);
wd[fencode_cd2(p,ix,iy,f1+direction)]= 0.0;
      else
wd[fencode_cd2(p,ix,iy,f1+direction)]= /*transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+*/fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
         #ifdef USE_VAC
      if(direction==0)
    // wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction);
 wd[fencode_cd2(p,ix,iy,f1+direction)]= 0.0;
      else
wd[fencode_cd2(p,ix,iy,f1+direction)]= /*transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+*/fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
       break;

       case b2 :
         #ifdef USE_SAC
      if(direction==1)
     //wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,field+5,direction);
wd[fencode_cd2(p,ix,iy,f1+direction)]= 0.0;
else
wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
         #ifdef USE_VAC
      if(direction==1)
    // wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction);
 wd[fencode_cd2(p,ix,iy,f1+direction)]= 0.0;
      else
wd[fencode_cd2(p,ix,iy,f1+direction)]= /*transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+*/fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
       break;

   /*    case b3 :
         #ifdef USE_SAC
      if(direction==2)
         //wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+transportflux_cd2(dw,wd,w,p,ix,iy,field+5,direction);
wd[fencode_cd2(p,ix,iy,f1+direction)]= 0;
      else
         wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
         #ifdef USE_VAC
       if(direction==2)
     //wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction);
wd[fencode_cd2(p,ix,iy,f1+direction)]= 0;
       else
       wd[fencode_cd2(p,ix,iy,f1+direction)]= transportflux_cd2(dw,wd,w,p,ix,iy,field,direction)+fluxb1(dw,wd,w,p,ix,iy,field,direction);
         #endif
       break;*/

    }
   
  }     
  return ( status);
}

__device__ __host__
int divflux_cd2(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field) {

  int direction;
  int status=0;
  real divflux=0;
  dw[fencode_cd2(p,ix,iy,field)]= grad_cd2(wd,p,ix,iy,f1,0)+grad_cd2(wd,p,ix,iy,f2,1); 


 #ifdef USE_SAC
  if(field==energy)     
     dw[fencode_cd2(p,ix,iy,field)]+=fluxe2(dw, wd, w, p,ix, iy);


 #endif
  return ( status);
}





//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux_cd2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case energy:
      computefluxe(dw,wd,w,p,ix,iy);
      // add the following terms for SAC
      // del((b bb+ bb b).v)+ptb del v - bb bb del v
     break;
     case b1:
      computefluxb(dw,wd,w,p,ix,iy,field);
     break;
     case b2:
       computefluxb(dw,wd,w,p,ix,iy,field);
     break;
     /*case b3:
      computefluxb(dw,wd,w,p,ix,iy,field);
     break;*/
  }
  //return ( status);
}



__global__ void centdiff2_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
 // real dt=p->dt;
  //real dy=p->dx[1];
 // real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/(p->n[0]);
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*(p->n[0]));


               //for(int f=energy; f<NVAR; f++)
               //{
			if(i<(ni) && j<(nj))
                        {
                            dwn1[fencode_cd2(p,i,j,f)]=0.0;

                 	    for(fid=0;fid<2;fid++)
                               dwn1[fencode_cd2(p,i,j,f1+fid)]=0.0;

                        }
                             __syncthreads();

                             
	
			if( i<(ni) && j<(nj))
                  		//computeflux_cd2(dwn1,wd,wmod,p,i,j,f);
                                computeflux_cd2(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f); 
               //}
                        //might need to set boundaries correctly 
                        __syncthreads();

        if( i<(ni) && j<(nj))
             for(fid=0;fid<2;fid++)
                  //bc_cont_cd2(dwn1,p,i,j,f1+fid);
                  bc_periodic1_cd2(dwn1,p,i,j,f1+fid);
                __syncthreads();

        if( i<(ni) && j<(nj))
             for(fid=0;fid<2;fid++)
                  //bc_cont_cd2(dwn1,p,i,j,f1+fid);
                  bc_periodic2_cd2(dwn1,p,i,j,f1+fid);
                __syncthreads();


             // for(int f=energy; f<NVAR; f++)
              // {
			if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                                divflux_cd2(dwn1,wd,wmod,p,i,j,f); 
               // }
                        __syncthreads();





             // for(int f=energy; f<=NVAR; f++)
               //{
			 if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                              //                                                                                  - sign here same as vac maybe a +
                             // wmod[fencode_cd2(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=wmod[fencode_cd2(p,i,j,f)]-dt*dwn1[fencode_cd2(p,i,j,f)];
                             wmod[fencode_cd2(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=wmod[fencode_cd2(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]-dt*dwn1[fencode_cd2(p,i,j,f)];  
               // }
                         __syncthreads(); 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff2(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt, int field)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     centdiff2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


