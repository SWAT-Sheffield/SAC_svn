#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_adv.cuh"
#include "dervfields_adv.cuh"


__global__ void advance_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
 
               float big=9999.0;
               for(int f=rho; f<NVAR; f++)
               {
                   
                   
                  if((p->rkon)==1)
                  {
                  //wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+(dt/6.0)*(dwn1[fencode_adv(p,i,j,f)]+2*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+2*dwn1[(2*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+dwn1[(3*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]);
                 // wnew[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt/6.0)*(dwn1[fencode_adv(p,i,j,f)]+2*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+2*dwn1[(2*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+dwn1[(3*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]);
wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+(dt/5.0)*(dwn1[fencode_adv(p,i,j,f)]+2*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+2*dwn1[(2*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]);
                   }
                  else
                  {
                  //if((dwn1[fencode_adv(p,i,j,f)]<(big/100)) && ( dwn1[fencode_adv(p,i,j,f)]>(-big/100)) )
                  //  if( j!=2)
                       wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+dt*dwn1[fencode_adv(p,i,j,f)];

                   //lax-friedrichs
                  //wnew[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt)*(dwn1[fencode_adv(p,i,j,f)]);
                   }
                  
                   if(isnan(wnew[fencode_adv(p,i,j,f)])) wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wnew[fencode_adv(p,i,j,f)]>big)
                           wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wnew[fencode_adv(p,i,j,f)]<-big)
                           wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];

                     if(f==rho)
                            if(wnew[fencode_adv(p,i,j,f)]<0)
                               wnew[fencode_adv(p,i,j,f)]=1.001;
               }
               //computej_adv(wnew,wd,p,i,j);
               //computepk_adv(wnew,wd,p,i,j);
               //computept_adv(wnew,wd,p,i,j);


	}
 __syncthreads();
  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, real **w, real **wnew,struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}



