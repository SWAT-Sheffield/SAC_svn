#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_adv.cuh"
#include "dervfields_adv.cuh"


__global__ void advance_parallel(struct params *p, real *wmod, real *w,  int order)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;

  int index,i,j,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];

    int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
	{		               
 
               float big=9999.0;
               for(int f=rho; f<NVAR; f++)
               {
                  
                   
                  if((p->rkon)==1)
                  {
                    switch(order)
                     {
                        case 0:
                       wmod[fencode3_adv(p,iia,f)+(2*dimp*NVAR)]=wmod[fencode3_adv(p,iia,f)];

                         break;
                        case 1:
                       wmod[fencode3_adv(p,iia,f)+(3*dimp*NVAR)]=wmod[fencode3_adv(p,iia,f)];
 
                         break;
                        case 2:
                       wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)]=(wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)]+2.0*wmod[fencode3_adv(p,iia,f)+(2*dimp*NVAR)]+wmod[fencode3_adv(p,iia,f)+(3*dimp*NVAR)]-4.0*wmod[fencode3_adv(p,iia,f)])/3;


                         break;
                        case 3:

                        wmod[fencode3_adv(p,iia,f)]=wmod[fencode3_adv(p,iia,f)]+wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)];

                         break;

                     }
                   }
                  else
                  {
                  //if((dwn1[fencode3_adv(p,iia,f)]<(big/100)) && ( dwn1[fencode3_adv(p,iia,f)]>(-big/100)) )
                  //  if( j!=2)
                       //wmod[fencode3_adv(p,iia,f)]=wmod[fencode3_adv(p,iia,f)+(order*(p->n[0])*(p->n[1])*NVAR)];
                      wmod[fencode3_adv(p,iia,f)]=wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)];
                   //lax-friedrichs
                  //wmod[fencode3_adv(p,iia,f)]=((w[fencode3_adv(p,i+1,j,f)]+w[fencode3_adv(p,i-1,j,f)]+w[fencode3_adv(p,iia+1,f)]+w[fencode3_adv(p,iia-1,f)])/4.0)+(dt)*(dwn1[fencode3_adv(p,iia,f)]);
                   }
                  
                   if(isnan(wmod[fencode3_adv(p,iia,f)])) wmod[fencode3_adv(p,iia,f)]=w[fencode3_adv(p,iia,f)];
                   if(wmod[fencode3_adv(p,iia,f)]>big)
                           wmod[fencode3_adv(p,iia,f)]=w[fencode3_adv(p,iia,f)];
                   if(wmod[fencode3_adv(p,iia,f)]<-big)
                           wmod[fencode3_adv(p,iia,f)]=w[fencode3_adv(p,iia,f)];

                     if(f==rho)
                            if(wmod[fencode3_adv(p,iia,f)]<0)
                               wmod[fencode3_adv(p,iia,f)]=1.00;
               }



	}
}
 __syncthreads();





  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, struct params **d_p,  real **d_wmod, real **d_w,  int order)
{

 dim3 dimBlock(dimblock, 1);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wmod, *d_w, order);
     hipDeviceSynchronize();
}



