#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_adv (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_adv (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
float evalgrad_adv(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
{
 //float valgrad;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   return((1.0/(1.0*(p->dx)))*(fi-fim1));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
      return((1.0/(1.0*(p->dy)))*(fi-fim1));
 }

 return -1;
}


__device__ __host__
float grad_adv(float *wmod,struct params *p,int i,int j,int field,int dir)
{
 //float valgrad;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
return((1.0/(1.0*(p->dx)))*(wmod[fencode_adv(p,i+1,j,field)]-wmod[fencode_adv(p,i-1,j,field)]));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
 return((1.0/(1.0*(p->dy)))*(wmod[fencode_adv(p,i,j+1,field)]-wmod[fencode_adv(p,i,j-1,field)]));

 }

 return -1;
}

__device__ __host__
void computej_adv(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;

 // float dbzdy, dbydz;
 // float dbzdx, dbxdz;
 // float dbydx, dbxdy;

 // dbzdy=grad(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad(wmod,p,i,j,b2,0);
 // dbxdy=grad(wmod,p,i,j,b1,1);

  wd[fencode_adv(p,i,j,0)]=(grad_adv(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_adv(p,i,j,1)]=(grad_adv(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_adv(p,i,j,2)]=(grad_adv(wmod,p,i,j,b2,0)-grad_adv(wmod,p,i,j,b1,1))/(p->mu);
 
  //return ( status);
}

__device__ __host__
void computebdotv_adv(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
//  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);

wd[fencode_adv(p,i,j,bdotv)]=(wmod[fencode_adv(p,i,j,b1)]*wmod[fencode_adv(p,i,j,mom1)]+wmod[fencode_adv(p,i,j,b2)]*wmod[fencode_adv(p,i,j,mom2)]+wmod[fencode_adv(p,i,j,b3)]*wmod[fencode_adv(p,i,j,mom3)])/wmod[fencode_adv(p,i,j,rho)];
 // return ( status);
}


__device__ __host__
void computepk_adv(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode_adv(p,i,j,4)]=  wd[fencode_adv(p,i,j,3)]+0.5*(wmod[fencode_adv(p,i,j,b1)]*wmod[fencode_adv(p,i,j,b1)]+wmod[fencode_adv(p,i,j,b2)]*wmod[fencode_adv(p,i,j,b2)]+wmod[fencode_adv(p,i,j,b3)]*wmod[fencode_adv(p,i,j,b3)]);
 // return ( status);
}
__device__ __host__
void computept_adv(float *wmod,float *wd,struct params *p,int i,int j)
{
  //int status=0;
  //float momsq=wmod[fencode(p,i,j,mom1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,mom2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,mom3)]*wmod[fencode(p,i,j,mom3)];
  //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode_adv(p,i,j,3)]=((p->gamma)-1)*(wmod[fencode_adv(p,i,j,energy)]- 0.5*(wmod[fencode_adv(p,i,j,mom1)]*wmod[fencode_adv(p,i,j,mom1)]+wmod[fencode_adv(p,i,j,mom2)]*wmod[fencode_adv(p,i,j,mom2)]+wmod[fencode_adv(p,i,j,mom3)]*wmod[fencode_adv(p,i,j,mom3)])/wmod[fencode_adv(p,i,j,rho)]-0.5*(wmod[fencode_adv(p,i,j,b1)]*wmod[fencode_adv(p,i,j,b1)]+wmod[fencode_adv(p,i,j,b2)]*wmod[fencode_adv(p,i,j,b2)]+wmod[fencode_adv(p,i,j,b3)]*wmod[fencode_adv(p,i,j,b3)]) );
  //return ( status);
}

__global__ void advance_parallel(struct params *p, float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
              /* for(int f=rho; f<=b3; f++)               
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode(p,i,j,f)]=1.0;
                  __syncthreads();
               }*/
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn1[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn2[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn3[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode(p,i,j,f)]+2.0*dwn2[fencode(p,i,j,f)]
                         +2.0*dwn3[fencode(p,i,j,f)]+dwn4[fencode(p,i,j,f)]);
               }*/
                __syncthreads();
               for(int f=rho; f<=b3; f++)
                   wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+dt*dwn1[fencode_adv(p,i,j,f)];
               computej_adv(wnew,wd,p,i,j);
               computepk_adv(wnew,wd,p,i,j);
               computept_adv(wnew,wd,p,i,j);


	}
 __syncthreads();
  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, float **w, float **wnew,struct params **d_p, float **d_w, float **d_wnew, float **d_wmod, float **d_dwn1, float **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}



