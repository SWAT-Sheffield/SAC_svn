#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_adv.cuh"
#include "dervfields_adv.cuh"


__global__ void advance_parallel(struct params *p, real *wmod, real *w,  int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
if( i<((p->n[0])) && j<((p->n[1])))
	{		               
 
               float big=9999.0;
               for(int f=rho; f<NVAR; f++)
               {
                  
                   
                  if((p->rkon)==1)
                  {
                     //wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];
                    switch(order)
                     {
                        case 0:
                       wmod[fencode_adv(p,i,j,f)+(2*(p->n[0])*(p->n[1])*NVAR)]=wmod[fencode_adv(p,i,j,f)];
                       // wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];

                         break;
                        case 1:
                       wmod[fencode_adv(p,i,j,f)+(3*(p->n[0])*(p->n[1])*NVAR)]=wmod[fencode_adv(p,i,j,f)];
                      // wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+(2*(p->n[0])*(p->n[1])*NVAR)];

                         break;
                        case 2:
                       wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)]=(wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)]+2.0*wmod[fencode_adv(p,i,j,f)+(2*(p->n[0])*(p->n[1])*NVAR)]+wmod[fencode_adv(p,i,j,f)+(3*(p->n[0])*(p->n[1])*NVAR)]-4.0*wmod[fencode_adv(p,i,j,f)])/3;


                         break;
                        case 3:
                      // wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)]+wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];
                        wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)]+wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];

                         break;

                     }
                   }
                  else
                  {
                  //if((dwn1[fencode_adv(p,i,j,f)]<(big/100)) && ( dwn1[fencode_adv(p,i,j,f)]>(-big/100)) )
                  //  if( j!=2)
                       //wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+(order*(p->n[0])*(p->n[1])*NVAR)];
                      wmod[fencode_adv(p,i,j,f)]=wmod[fencode_adv(p,i,j,f)+((p->n[0])*(p->n[1])*NVAR)];
                   //lax-friedrichs
                  //wmod[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt)*(dwn1[fencode_adv(p,i,j,f)]);
                   }
                  
                   if(isnan(wmod[fencode_adv(p,i,j,f)])) wmod[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wmod[fencode_adv(p,i,j,f)]>big)
                           wmod[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wmod[fencode_adv(p,i,j,f)]<-big)
                           wmod[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];

                     if(f==rho)
                            if(wmod[fencode_adv(p,i,j,f)]<0)
                               wmod[fencode_adv(p,i,j,f)]=1.00;
               }
               //computej_adv(wmod,wd,p,i,j);
               //computepk_adv(wmod,wd,p,i,j);
               //computept_adv(wmod,wd,p,i,j);


	}
}
 __syncthreads();





  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, struct params **d_p,  real **d_wmod, real **d_w,  int order)
{

 dim3 dimBlock(dimblock, 1);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wmod, *d_w, order);
     hipDeviceSynchronize();
}



