#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_adv (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_adv (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
real evalgrad_adv(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx)))*(fi-fim1)):((1.0/(12.0*(p->dx)))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
     // return((2.0/(1.0*(p->dy)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dy)))*(fi-fim1)):((1.0/(12.0*(p->dy)))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_adv(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx)))*(wmod[fencode_adv(p,i+1,j,field)]-wmod[fencode_adv(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_adv(p,i+1,j,field)]-8*wmod[fencode_adv(p,i-1,j,field)]+wmod[fencode_adv(p,i-1,j,field)]-wmod[fencode_adv(p,i+1,j,field)])/6.0):wmod[fencode_adv(p,i+1,j,field)]-wmod[fencode_adv(p,i-1,j,field)])/(2.0*(p->dx))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dy)))*(wmod[fencode_adv(p,i,j+1,field)]-wmod[fencode_adv(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_adv(p,i,j+1,field)]-8*wmod[fencode_adv(p,i,j-1,field)]+wmod[fencode_adv(p,i,j-1,field)]-wmod[fencode_adv(p,i,j+1,field)])/6.0):wmod[fencode_adv(p,i,j+1,field)]-wmod[fencode_adv(p,i,j-1,field)])/(2.0*(p->dy))    );

 }

 return -1;
}

__device__ __host__
void computej_adv(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad(wmod,p,i,j,b2,0);
 // dbxdy=grad(wmod,p,i,j,b1,1);

  wd[fencode_adv(p,i,j,0)]=(grad_adv(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_adv(p,i,j,1)]=(grad_adv(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_adv(p,i,j,2)]=(grad_adv(wmod,p,i,j,b2,0)-grad_adv(wmod,p,i,j,b1,1))/(p->mu);
 
  //return ( status);
}

__device__ __host__
void computebdotv_adv(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
//  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);

wd[fencode_adv(p,i,j,bdotv)]=(wmod[fencode_adv(p,i,j,b1)]*wmod[fencode_adv(p,i,j,mom1)]+wmod[fencode_adv(p,i,j,b2)]*wmod[fencode_adv(p,i,j,mom2)]+wmod[fencode_adv(p,i,j,b3)]*wmod[fencode_adv(p,i,j,mom3)])/wmod[fencode_adv(p,i,j,rho)];
 // return ( status);
}

__device__ __host__
void computedivb_adv(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
//  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);

wd[fencode_adv(p,i,j,divb)]=grad_adv(wmod,p,i,j,b1,0)+grad_adv(wmod,p,i,j,b2,1);
 // return ( status);
}


__device__ __host__
void computepk_adv(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

         #ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
wd[fencode_adv(p,i,j,4)]=(p->adiab)*pow(wmod[fencode_adv(p,i,j,rho)],p->gamma);


#else

 //real bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode_adv(p,i,j,4)]=  wd[fencode_adv(p,i,j,3)]+0.5*(wmod[fencode_adv(p,i,j,b1)]*wmod[fencode_adv(p,i,j,b1)]+wmod[fencode_adv(p,i,j,b2)]*wmod[fencode_adv(p,i,j,b2)]+wmod[fencode_adv(p,i,j,b3)]*wmod[fencode_adv(p,i,j,b3)]);

#endif


  if(wd[fencode_adv(p,i,j,4)]<0)
              wd[fencode_adv(p,i,j,3)]=0.001;


 // return ( status);
}
__device__ __host__
void computept_adv(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;



         #ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_adv(p,i,j,3)]=(p->adiab)*pow(wmod[fencode_adv(p,i,j,rho)],p->gamma);

#else

  //real momsq=wmod[fencode(p,i,j,mom1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,mom2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,mom3)]*wmod[fencode(p,i,j,mom3)];
  //real bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode_adv(p,i,j,3)]=((p->gamma)-1)*(wmod[fencode_adv(p,i,j,energy)]- 0.5*(wmod[fencode_adv(p,i,j,mom1)]*wmod[fencode_adv(p,i,j,mom1)]+wmod[fencode_adv(p,i,j,mom2)]*wmod[fencode_adv(p,i,j,mom2)]+wmod[fencode_adv(p,i,j,mom3)]*wmod[fencode_adv(p,i,j,mom3)])/wmod[fencode_adv(p,i,j,rho)]-0.5*(wmod[fencode_adv(p,i,j,b1)]*wmod[fencode_adv(p,i,j,b1)]+wmod[fencode_adv(p,i,j,b2)]*wmod[fencode_adv(p,i,j,b2)]+wmod[fencode_adv(p,i,j,b3)]*wmod[fencode_adv(p,i,j,b3)]) );

#endif

  if(wd[fencode_adv(p,i,j,3)]<0)
              wd[fencode_adv(p,i,j,3)]=0.001;

  //return ( status);
}

__global__ void advance_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-1) && j<((p->nj)-1))
	{		               
              /* for(int f=rho; f<=b3; f++)               
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode(p,i,j,f)]=1.0;
                  __syncthreads();
               }*/
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn1[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn2[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn3[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode(p,i,j,f)]+2.0*dwn2[fencode(p,i,j,f)]
                         +2.0*dwn3[fencode(p,i,j,f)]+dwn4[fencode(p,i,j,f)]);
               }*/
                __syncthreads();
               float big=9999.0;
               for(int f=rho; f<=b3; f++)
               {
                   
                  // if((dwn1[fencode_adv(p,i,j,f)]<(big/100)) && ( dwn1[fencode_adv(p,i,j,f)]>(-big/100)) )
                      // wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+dt*dwn1[fencode_adv(p,i,j,f)];

                   //lax-friedrichs
                  wnew[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt)*(dwn1[fencode_adv(p,i,j,f)]);
                  
                   if(isnan(wnew[fencode_adv(p,i,j,f)])) wnew[fencode_adv(p,i,j,f)]=0;
                   if(wnew[fencode_adv(p,i,j,f)]>big)
                           wnew[fencode_adv(p,i,j,f)]=big;
                   if(wnew[fencode_adv(p,i,j,f)]<-big)
                           wnew[fencode_adv(p,i,j,f)]=-big;

                     if(f==rho)
                            if(wnew[fencode_adv(p,i,j,f)]<0)
                               wnew[fencode_adv(p,i,j,f)]=0.001;
               }
               computej_adv(wnew,wd,p,i,j);
               computepk_adv(wnew,wd,p,i,j);
               computept_adv(wnew,wd,p,i,j);


	}
 __syncthreads();
  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, real **w, real **wnew,struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}



