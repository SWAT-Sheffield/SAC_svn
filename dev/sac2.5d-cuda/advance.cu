#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "iotypes.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////

__device__ __host__
int dimproduct_adv (struct params *dp) {

  int tot=1;
  for(int i=0;i<NDIM;i++)
    tot*=dp->n[i];
  return tot; 
}


__device__ __host__
int encode_adv (struct params *dp,int ix, int iy) {

  return (iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int encode3_adv (struct params *dp,int ix, int iy, int iz) {

  return (iz*((dp)->n[0])*((dp)->n[1])  + iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int fencode_adv (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return(( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1]))));
}


__device__ __host__
int fencode3_adv (struct params *dp,int ix, int iy, int iz, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return(  iz*((dp)->n[0])*((dp)->n[1])+ (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])*((dp)->n[2]))  );
}
__device__ __host__
real evalgrad_adv(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad;

   return(p->sodifon?((1.0/(2.0*(p->dx[dir])))*(fi-fim1)):((1.0/(12.0*(p->dx[dir])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 
}


__device__ __host__
real grad_adv(real *wmod,struct params *p,int *ix,int field,int dir)
{
 //real valgrad;

 if(dir == 0)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode_adv(p,ix[0]+1,ix[1],field)]-NVAR*wmod[fencode_adv(p,ix[0]-1,ix[1],field)]+wmod[fencode_adv(p,ix[0]-2,ix[1],field)]-wmod[fencode_adv(p,ix[0]+2,ix[1],field)])/6.0):wmod[fencode_adv(p,ix[0]+1,ix[1],field)]-wmod[fencode_adv(p,ix[0]-1,ix[1],field)])/(2.0*(p->dx[0]))    );
 }
 else if(dir == 1)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode_adv(p,ix[0],ix[1]+1,field)]-NVAR*wmod[fencode_adv(p,ix[0],ix[1]-1,field)]+wmod[fencode_adv(p,ix[0],ix[1]-2,field)]-wmod[fencode_adv(p,ix[0],ix[1]+2,field)])/6.0):wmod[fencode_adv(p,ix[0],ix[1]+1,field)]-wmod[fencode_adv(p,ix[0],ix[1]-1,field)])/(2.0*(p->dx[1]))    );

 }

 return -1;
}

__device__ __host__
real grad3_adv(real *wmod,struct params *p,int *ix,int field,int dir)
{
 //real valgrad;

 if(dir == 0)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode3_adv(p,ix[0]+1,ix[1],ix[2],field)]-NVAR*wmod[fencode3_adv(p,ix[0]-1,ix[1],ix[2],field)]+wmod[fencode3_adv(p,ix[0]-2,ix[1],ix[2],field)]-wmod[fencode3_adv(p,ix[0]+2,ix[1],ix[2],field)])/6.0):wmod[fencode3_adv(p,ix[0]+1,ix[1],ix[2],field)]-wmod[fencode3_adv(p,ix[0]-1,ix[1],ix[2],field)])/(2.0*(p->dx[0]))    );
 }
 else if(dir == 1)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode3_adv(p,ix[0],ix[1]+1,ix[2],field)]-NVAR*wmod[fencode3_adv(p,ix[0],ix[1]-1,ix[2],field)]+wmod[fencode3_adv(p,ix[0],ix[1]-2,ix[2],field)]-wmod[fencode3_adv(p,ix[0],ix[1]+2,ix[2],field)])/6.0):wmod[fencode3_adv(p,ix[0],ix[1]+1,ix[2],field)]-wmod[fencode3_adv(p,ix[0],ix[1]-1,ix[2],field)])/(2.0*(p->dx[1]))    );

 }
else if(dir == 2)
 {

 return(  ( (p->sodifon)?((NVAR*wmod[fencode3_adv(p,ix[0],ix[1],ix[2]+1,field)]-NVAR*wmod[fencode3_adv(p,ix[0],ix[1],ix[2]-1,field)]+wmod[fencode3_adv(p,ix[0],ix[1],ix[2]-2,field)]-wmod[fencode3_adv(p,ix[0],ix[1],ix[2]+2,field)])/6.0):wmod[fencode3_adv(p,ix[0],ix[1],ix[2]+1,field)]-wmod[fencode3_adv(p,ix[0],ix[1],ix[2]-1,field)])/(2.0*(p->dx[2]))    );

 }
 return -1;
}

__device__ __host__
void computej_adv(real *wmod,real *wd,struct params *p,int *ix)
{
  wd[fencode_adv(p,ix[0],ix[1],0)]=(grad_adv(wmod,p,ix,b3,1))/(p->mu);
  wd[fencode_adv(p,ix[0],ix[1],1)]=(grad_adv(wmod,p,ix,b3,0))/(p->mu);
  wd[fencode_adv(p,ix[0],ix[1],2)]=(grad_adv(wmod,p,ix,b2,0)-grad_adv(wmod,p,ix,b1,1))/(p->mu);
}

__device__ __host__
void computebdotv_adv(real *wmod,real *wd,struct params *p,int *ix)
{
wd[fencode_adv(p,ix[0],ix[1],bdotv)]=(wmod[fencode_adv(p,ix[0],ix[1],b1)]*wmod[fencode_adv(p,ix[0],ix[1],mom1)]+wmod[fencode_adv(p,ix[0],ix[1],b2)]*wmod[fencode_adv(p,ix[0],ix[1],mom2)]+wmod[fencode_adv(p,ix[0],ix[1],b3)]*wmod[fencode_adv(p,ix[0],ix[1],mom3)])/wmod[fencode_adv(p,ix[0],ix[1],rho)];
}

__device__ __host__
void computedivb_adv(real *wmod,real *wd,struct params *p,int *ix)
{
wd[fencode_adv(p,ix[0],ix[1],divb)]=grad_adv(wmod,p,ix,b1,0)+grad_adv(wmod,p,ix,b2,1);
}


__device__ __host__
void computepk_adv(real *wmod,real *wd,struct params *p,int *ix)
{
 // int status=0;

         #ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
wd[fencode_adv(p,ix[0],ix[1],pressurek)]=(p->adiab)*pow(wmod[fencode_adv(p,ix[0],ix[1],rho)],p->gamma);


#else


 wd[fencode_adv(p,ix[0],ix[1],pressurek)]=((p->gamma)-1)*(wmod[fencode_adv(p,ix[0],ix[1],energy)]- 0.5*(wmod[fencode_adv(p,ix[0],ix[1],mom1)]*wmod[fencode_adv(p,ix[0],ix[1],mom1)]+wmod[fencode_adv(p,ix[0],ix[1],mom2)]*wmod[fencode_adv(p,ix[0],ix[1],mom2)]+wmod[fencode_adv(p,ix[0],ix[1],mom3)]*wmod[fencode_adv(p,ix[0],ix[1],mom3)])/wmod[fencode_adv(p,ix[0],ix[1],rho)]-0.5*(wmod[fencode_adv(p,ix[0],ix[1],b1)]*wmod[fencode_adv(p,ix[0],ix[1],b1)]+wmod[fencode_adv(p,ix[0],ix[1],b2)]*wmod[fencode_adv(p,ix[0],ix[1],b2)]+wmod[fencode_adv(p,ix[0],ix[1],b3)]*wmod[fencode_adv(p,ix[0],ix[1],b3)]) );

#endif


  if(wd[fencode_adv(p,ix[0],ix[1],pressurek)]<0)
              wd[fencode_adv(p,ix[0],ix[1],pressurek)]=0.001;


}
__device__ __host__
void computept_adv(real *wmod,real *wd,struct params *p,int *ix)
{
         #ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_adv(p,ix[0],ix[1],pressuret)]=(p->adiab)*pow(wmod[fencode_adv(p,ix[0],ix[1],rho)],p->gamma);

#else


   wd[fencode_adv(p,ix[0],ix[1],pressuret)]=  wd[fencode_adv(p,ix[0],ix[1],pressurek)]+0.5*(wmod[fencode_adv(p,ix[0],ix[1],b1)]*wmod[fencode_adv(p,ix[0],ix[1],b1)]+wmod[fencode_adv(p,ix[0],ix[1],b2)]*wmod[fencode_adv(p,ix[0],ix[1],b2)]+wmod[fencode_adv(p,ix[0],ix[1],b3)]*wmod[fencode_adv(p,ix[0],ix[1],b3)]);

#endif

  if(wd[fencode_adv(p,ix[0],ix[1],pressuret)]<0)
              wd[fencode_adv(p,ix[0],ix[1],pressuret)]=0.001;

  //return ( status);
}

__global__ void advance_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
 
               float big=9999.0;
               for(int f=rho; f<=b3; f++)
               {
                   
                   
                  if((p->rkon)==1)
                  {
                  wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+(dt/6.0)*(dwn1[fencode_adv(p,i,j,f)]+2*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+2*dwn1[(2*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+dwn1[(3*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]);
                 // wnew[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt/6.0)*(dwn1[fencode_adv(p,i,j,f)]+2*dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+2*dwn1[(2*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]+dwn1[(3*NVAR*(p->n[0])*(p->n[1]))+fencode_adv(p,i,j,f)]);

                   }
                  else
                  {
                  //if((dwn1[fencode_adv(p,i,j,f)]<(big/100)) && ( dwn1[fencode_adv(p,i,j,f)]>(-big/100)) )
                  //  if( j!=2)
                       wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)]+dt*dwn1[fencode_adv(p,i,j,f)];

                   //lax-friedrichs
                  //wnew[fencode_adv(p,i,j,f)]=((w[fencode_adv(p,i+1,j,f)]+w[fencode_adv(p,i-1,j,f)]+w[fencode_adv(p,i,j+1,f)]+w[fencode_adv(p,i,j-1,f)])/4.0)+(dt)*(dwn1[fencode_adv(p,i,j,f)]);
                   }
                  
                   if(isnan(wnew[fencode_adv(p,i,j,f)])) wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wnew[fencode_adv(p,i,j,f)]>big)
                           wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];
                   if(wnew[fencode_adv(p,i,j,f)]<-big)
                           wnew[fencode_adv(p,i,j,f)]=w[fencode_adv(p,i,j,f)];

                     if(f==rho)
                            if(wnew[fencode_adv(p,i,j,f)]<0)
                               wnew[fencode_adv(p,i,j,f)]=1.001;
               }
               //computej_adv(wnew,wd,p,i,j);
               //computepk_adv(wnew,wd,p,i,j);
               //computept_adv(wnew,wd,p,i,j);


	}
 __syncthreads();
  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, real **w, real **wnew,struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}



