#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_cor (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_cor (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
real evalgrad_cor(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_cor;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx)))*(fi-fim1)):((1.0/(12.0*(p->dx)))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
     // return((2.0/(1.0*(p->dy)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dy)))*(fi-fim1)):((1.0/(12.0*(p->dy)))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_cor(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_cor;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx)))*(wmod[fencode_cor(p,i+1,j,field)]-wmod[fencode_cor(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_cor(p,i+1,j,field)]-8*wmod[fencode_cor(p,i-1,j,field)]+wmod[fencode_cor(p,i-2,j,field)]-wmod[fencode_cor(p,i+2,j,field)])/6.0):wmod[fencode_cor(p,i+1,j,field)]-wmod[fencode_cor(p,i-1,j,field)])/(2.0*(p->dx))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dy)))*(wmod[fencode_cor(p,i,j+1,field)]-wmod[fencode_cor(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_cor(p,i,j+1,field)]-8*wmod[fencode_cor(p,i,j-1,field)]+wmod[fencode_cor(p,i,j-2,field)]-wmod[fencode_cor(p,i,j+2,field)])/6.0):wmod[fencode_cor(p,i,j+1,field)]-wmod[fencode_cor(p,i,j-1,field)])/(2.0*(p->dy))    );  
}


 return 0;
}

__device__ __host__
void computej_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad_cor(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_cor(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_cor(wmod,p,i,j,b2,0);
 // dbxdy=grad_cor(wmod,p,i,j,b1,1);

  wd[fencode_cor(p,i,j,0)]=(grad_cor(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_cor(p,i,j,1)]=(grad_cor(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_cor(p,i,j,2)]=(grad_cor(wmod,p,i,j,b2,0)-grad_cor(wmod,p,i,j,b1,1))/(p->mu);
  
  

 
  //return ( status);
}

__device__ __host__
void computebdotv_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
//  wd[fencode_cor(p,i,j,4)]=  wd[fencode_cor(p,i,j,3)]+0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]);

wd[fencode_cor(p,i,j,bdotv)]=(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,mom1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,mom2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,mom3)])/wmod[fencode_cor(p,i,j,rho)];
 // return ( status);
}

__device__ __host__
void computedivb_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
//  wd[fencode_cor(p,i,j,4)]=  wd[fencode_cor(p,i,j,3)]+0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]);

wd[fencode_cor(p,i,j,divb)]=grad_cor(wmod,p,i,j,b1,0)+grad_cor(wmod,p,i,j,b2,1);
 // return ( status);
}


__device__ __host__
void computept_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode_cor(p,i,j,pressuret)]=(p->adiab)*pow(wmod[fencode_cor(p,i,j,rho)],p->gamma);


#else

 //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
  wd[fencode_cor(p,i,j,pressuret)]=  wd[fencode_cor(p,i,j,pressurek)]+0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]);

#endif



  if(wd[fencode_cor(p,i,j,pressuret)]<0)
              wd[fencode_cor(p,i,j,pressuret)]=0.001;


 // return ( status);
}
__device__ __host__
void computepk_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_cor(p,i,j,pressurek)]=(p->adiab)*pow(wmod[fencode_cor(p,i,j,rho)],p->gamma);

#else

  //real momsq=wmod[fencode_cor(p,i,j,mom1)]*wmod[fencode_cor(p,i,j,mom1)]+wmod[fencode_cor(p,i,j,mom2)]*wmod[fencode_cor(p,i,j,mom2)]+wmod[fencode_cor(p,i,j,mom3)]*wmod[fencode_cor(p,i,j,mom3)];
  //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
  wd[fencode_cor(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_cor(p,i,j,energy)]- 0.5*(wmod[fencode_cor(p,i,j,mom1)]*wmod[fencode_cor(p,i,j,mom1)]+wmod[fencode_cor(p,i,j,mom2)]*wmod[fencode_cor(p,i,j,mom2)]+wmod[fencode_cor(p,i,j,mom3)]*wmod[fencode_cor(p,i,j,mom3)])/wmod[fencode_cor(p,i,j,rho)]-0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]) );


#endif






  if(wd[fencode_cor(p,i,j,pressurek)]<0)
              wd[fencode_cor(p,i,j,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec_cor(real *wmod,real *wd,struct params *p,int i,int j)
{

  
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode_cor(p,i,j,soundspeed)]=sqrt((p->adiab)/wmod[fencode_cor(p,i,j,rho)]);

#else
wd[fencode_cor(p,i,j,soundspeed)]=sqrt(((p->gamma))*wd[fencode_cor(p,i,j,pressuret)]/wmod[fencode_cor(p,i,j,rho)]);

#endif



  
}

__device__ __host__
void computecmax_cor(real *wmod,real *wd,struct params *p,int i,int j)
{

       if(wd[fencode_cor(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_cor(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_cor(p,i,j,soundspeed)]);

}


__global__ void corrector_parallel(struct params *p,  real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
   if(order==1 || order==2)
     dt=(p->dt)/2.0;

  //advance the solution for one of the corrector steps
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{ 
   
		for(int f=rho; f<=b3; f++)           
 			//wmod[fencode_cor(p,i,j,f)]=((w[fencode_cor(p,i+1,j,f)]+w[fencode_cor(p,i-1,j,f)]+w[fencode_cor(p,i,j+1,f)]+w[fencode_cor(p,i,j-1,f)])/4.0)+dt*dwn1[(8*ni*nj*(order-1))+fencode_cor(p,i,j,f)];
wmod[fencode_cor(p,i,j,f)]=(w[fencode_cor(p,i,j,f)])+dt*dwn1[(8*ni*nj*(order-1))+fencode_cor(p,i,j,f)];
	}

 __syncthreads();

if(i<((p->ni)) && j<((p->nj)))
	{		
               //for(int f=rho; f<=b3; f++)
               //{               
               //   wmod[fencode_cor(p,i,j,f)]=w[fencode_cor(p,i,j,f)];
               //   wnew[fencode_cor(p,i,j,f)]=0.0;
               //}
               for(int f=current1; f<=divb; f++)
                  wd[fencode_cor(p,i,j,f)]=0; 
        }
               __syncthreads();


  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               computej_cor(wmod,wd,p,i,j);
               computepk_cor(wmod,wd,p,i,j);
               computept_cor(wmod,wd,p,i,j);

               computebdotv_cor(wmod,wd,p,i,j);
               computedivb_cor(wmod,wd,p,i,j);
         }
              __syncthreads();
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{
 //determin cmax
               computec_cor(wmod,wd,p,i,j);
        }
              __syncthreads();


  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cor(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucorrector(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     corrector_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
     hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






