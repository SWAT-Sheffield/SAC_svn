#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////


__device__ __host__
int dimproduct_cor (struct params *dp) {

  int tot=1;
  for(int i=0;i<NDIM;i++)
    tot*=dp->n[i];
  return tot; 
}


__device__ __host__
int encode_cor (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int fencode_cor (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])));
}

__device__ __host__
real evalgrad_cor(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_cor;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx[0])))*(fi-fim1)-(1.0/(12.0*(p->dx[0])))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx[0])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[0])))*(fi-fim1)):((1.0/(12.0*(p->dx[0])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(fi-fim1)-(1.0/(12.0*(p->dx[1])))*(fip2-fim2);
     // return((2.0/(1.0*(p->dx[1])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[1])))*(fi-fim1)):((1.0/(12.0*(p->dx[1])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_cor(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_cor;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx[0])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx[0])))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx[0])))*(wmod[fencode_cor(p,i+1,j,field)]-wmod[fencode_cor(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((NVAR*wmod[fencode_cor(p,i+1,j,field)]-NVAR*wmod[fencode_cor(p,i-1,j,field)]+wmod[fencode_cor(p,i-2,j,field)]-wmod[fencode_cor(p,i+2,j,field)])/6.0):wmod[fencode_cor(p,i+1,j,field)]-wmod[fencode_cor(p,i-1,j,field)])/(2.0*(p->dx[0]))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dx[1])))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dx[1])))*(wmod[fencode_cor(p,i,j+1,field)]-wmod[fencode_cor(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((NVAR*wmod[fencode_cor(p,i,j+1,field)]-NVAR*wmod[fencode_cor(p,i,j-1,field)]+wmod[fencode_cor(p,i,j-2,field)]-wmod[fencode_cor(p,i,j+2,field)])/6.0):wmod[fencode_cor(p,i,j+1,field)]-wmod[fencode_cor(p,i,j-1,field)])/(2.0*(p->dx[1]))    );  
}


 return 0;
}

__device__ __host__
void computej_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad_cor(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_cor(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_cor(wmod,p,i,j,b2,0);
 // dbxdy=grad_cor(wmod,p,i,j,b1,1);

  wd[fencode_cor(p,i,j,0)]=(grad_cor(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_cor(p,i,j,1)]=(grad_cor(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_cor(p,i,j,2)]=(grad_cor(wmod,p,i,j,b2,0)-grad_cor(wmod,p,i,j,b1,1))/(p->mu);
  
  

 
  //return ( status);
}

__device__ __host__
void computebdotv_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
//  wd[fencode_cor(p,i,j,4)]=  wd[fencode_cor(p,i,j,3)]+0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]);

wd[fencode_cor(p,i,j,bdotv)]=(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,mom1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,mom2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,mom3)])/wmod[fencode_cor(p,i,j,rho)];
 // return ( status);
}

__device__ __host__
void computedivb_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
//  wd[fencode_cor(p,i,j,4)]=  wd[fencode_cor(p,i,j,3)]+0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]);

wd[fencode_cor(p,i,j,divb)]=grad_cor(wmod,p,i,j,b1,0)+grad_cor(wmod,p,i,j,b2,1);
 // return ( status);
}


__device__ __host__
void computept_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode_cor(p,i,j,pressuret)]=(p->adiab)*pow(wmod[fencode_cor(p,i,j,rho)],p->gamma);


#else

 //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
  wd[fencode_cor(p,i,j,pressuret)]=  wd[fencode_cor(p,i,j,pressurek)]+0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]);

#endif



  if(wd[fencode_cor(p,i,j,pressuret)]<0)
              wd[fencode_cor(p,i,j,pressuret)]=0.001;


 // return ( status);
}
__device__ __host__
void computepk_cor(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_cor(p,i,j,pressurek)]=(p->adiab)*pow(wmod[fencode_cor(p,i,j,rho)],p->gamma);

#else

  //real momsq=wmod[fencode_cor(p,i,j,mom1)]*wmod[fencode_cor(p,i,j,mom1)]+wmod[fencode_cor(p,i,j,mom2)]*wmod[fencode_cor(p,i,j,mom2)]+wmod[fencode_cor(p,i,j,mom3)]*wmod[fencode_cor(p,i,j,mom3)];
  //real bsq=wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)];
  wd[fencode_cor(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_cor(p,i,j,energy)]- 0.5*(wmod[fencode_cor(p,i,j,mom1)]*wmod[fencode_cor(p,i,j,mom1)]+wmod[fencode_cor(p,i,j,mom2)]*wmod[fencode_cor(p,i,j,mom2)]+wmod[fencode_cor(p,i,j,mom3)]*wmod[fencode_cor(p,i,j,mom3)])/wmod[fencode_cor(p,i,j,rho)]-0.5*(wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)]) );


#endif






  if(wd[fencode_cor(p,i,j,pressurek)]<0)
              wd[fencode_cor(p,i,j,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec_cor(real *wmod,real *wd,struct params *p,int i,int j)
{

  
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode_cor(p,i,j,soundspeed)]=sqrt((p->adiab)/wmod[fencode_cor(p,i,j,rho)]);

#else
wd[fencode_cor(p,i,j,soundspeed)]=sqrt(((p->gamma))*wd[fencode_cor(p,i,j,pressuret)]/wmod[fencode_cor(p,i,j,rho)]);
wd[fencode_cor(p,i,j,cfast)]=sqrt(((wmod[fencode_cor(p,i,j,b1)]*wmod[fencode_cor(p,i,j,b1)]+wmod[fencode_cor(p,i,j,b2)]*wmod[fencode_cor(p,i,j,b2)]+wmod[fencode_cor(p,i,j,b3)]*wmod[fencode_cor(p,i,j,b3)])/wmod[fencode_cor(p,i,j,rho)])+(wd[fencode_cor(p,i,j,soundspeed)]*wd[fencode_cor(p,i,j,soundspeed)]));
#endif



  
}

__device__ __host__
void computecmax_cor(real *wmod,real *wd,struct params *p,int i,int j)
{

       if(wd[fencode_cor(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_cor(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_cor(p,i,j,soundspeed)]);
       if(wd[fencode_cor(p,i,j,cfast)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_cor(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_cor(p,i,j,cfast)]);

}


__global__ void corrector_parallel(struct params *p,  real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  int ix[NDIM];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
   ix[0]=i;
   ix[1]=j;
   if(order==1 || order==2)
     dt=(p->dt)/2.0;

  //advance the solution for one of the corrector steps
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{ 
   
		for(int f=rho; f<=b3; f++)           
 			wmod[fencode_cor(p,i,j,f)]=((w[fencode_cor(p,i+1,j,f)]+w[fencode_cor(p,i-1,j,f)]+w[fencode_cor(p,i,j+1,f)]+w[fencode_cor(p,i,j-1,f)])/4.0)+dt*dwn1[(NVAR*ni*nj*(order-1))+fencode_cor(p,i,j,f)];
//wmod[fencode_cor(p,i,j,f)]=(w[fencode_cor(p,i,j,f)])+dt*dwn1[(NVAR*ni*nj*(order-1))+fencode_cor(p,i,j,f)];
	}

 __syncthreads();

if(i<((p->n[0])) && j<((p->n[1])))
	{		
               //for(int f=rho; f<=b3; f++)
               //{               
               //   wmod[fencode_cor(p,i,j,f)]=w[fencode_cor(p,i,j,f)];
               //   wnew[fencode_cor(p,i,j,f)]=0.0;
               //}
               for(int f=current1; f<=divb; f++)
                  wd[fencode_cor(p,i,j,f)]=0; 
        }
               __syncthreads();


  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
               computej_cor(wmod,wd,p,i,j);
               computepk_cor(wmod,wd,p,i,j);
               computept_cor(wmod,wd,p,i,j);

               computebdotv_cor(wmod,wd,p,i,j);
               computedivb_cor(wmod,wd,p,i,j);
         }
              __syncthreads();
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec_cor(wmod,wd,p,i,j);
        }
              __syncthreads();


  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cor(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucorrector(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimproduct_cor(*p)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     corrector_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
 

  //  hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






