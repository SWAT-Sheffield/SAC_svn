#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdb1.cuh"






__global__ void hyperdifbsource4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb,real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         if(i<(ni) && j<(nj))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hdb1(p,i,j,b1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdb1(p,i,j,b1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdb1(p,i,j,b1+field)]; 
                             wmod[fencode_hdb1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdb1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdb1(p,i,j,energy)]; 

                         }
              //  }	
}
  __syncthreads();  
}




__global__ void hyperdifbsource3_parallel(struct params *p, real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb,real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   real rdx=((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1);

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  if( i<((p->n[0])) && j<((p->n[1])))
	{		               



dwn1[fencode_hdb1(p,i,j,b1+ii0)]=sb*(wtemp[fencode_hdb1(p,i,j,tmp5)]*wd[fencode_hdb1(p,i,j,hdnur)]-wtemp[fencode_hdb1(p,i,j,tmp4)]*wd[fencode_hdb1(p,i,j,hdnul)])/rdx;

dwn1[fencode_hdb1(p,i,j,energy)]=sb*(wtemp[fencode_hdb1(p,i,j,tmp3)]*wtemp[fencode_hdb1(p,i,j,tmp5)]*wd[fencode_hdb1(p,i,j,hdnur)]-wtemp[fencode_hdb1(p,i,j,tmp2)]*wtemp[fencode_hdb1(p,i,j,tmp4)]*wd[fencode_hdb1(p,i,j,hdnul)])/rdx;


   }
}
 __syncthreads();


 
}






__global__ void hyperdifbsource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   real rdx=((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1);


   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
	{		               

wtemp[fencode_hdb1(p,i,j,tmp4)]=grad1l_hdb1(wtemp,p,i,j,tmp1,dim);
wtemp[fencode_hdb1(p,i,j,tmp5)]=grad1r_hdb1(wtemp,p,i,j,tmp1,dim);

   }
}
 __syncthreads();   



   



}



__global__ void hyperdifbsource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int shift=(order*NVAR*(p->n[0])*(p->n[1]));
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode_hdb1(p,i,j,f)]=0.0;

   dwn1[fencode_hdb1(p,i,j,energy)]=0.0;
   dwn1[fencode_hdb1(p,i,j,b1+ii0)]=0.0;
  }

}
 __syncthreads();



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


       wtemp[fencode_hdb1(p,i,j,tmp2)]=(wmod[shift+fencode_hdb1(p,i,j,b1+jj)]+wmod[shift+fencode_hdb1(p,i-(dim==0),j-(dim==1),b1+jj)])/2;

       wtemp[fencode_hdb1(p,i,j,tmp3)]=(wmod[shift+fencode_hdb1(p,i,j,b1+jj)]+wmod[shift+fencode_hdb1(p,i+(dim==0),j+(dim==1),b1+jj)])/2;

wtemp[fencode_hdb1(p,i,j,tmp1)]=wmod[shift+fencode_hdb1(p,i,j,b1+field)];

   }

}
__syncthreads();






   

}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdb1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifbsource1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{

 dim3 dimBlock(dimblock, 1);

    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifbsource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb); 
     hipDeviceSynchronize();
     hyperdifbsource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb); 
     hipDeviceSynchronize();
     hyperdifbsource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt); 
     hipDeviceSynchronize();
     hyperdifbsource4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt); 
     hipDeviceSynchronize();

}







