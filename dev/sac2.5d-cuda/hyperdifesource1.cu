#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hde1.cuh"



__global__ void hyperdifesource3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

if(i<((p->n[0])) && j<((p->n[1])))
//  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


 
//dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wtemp[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             )/rdx;
//dwn1[fencode_hde1(p,i,j,field)]=( wd[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wd[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             );

//wtemp[fencode_hde1(p,i,j,tmp2)]= grad1r_hde1(wtemp,p,i,j,tmp1,dim) ;
//wtemp[fencode_hde1(p,i,j,tmp3)]= grad1l_hde1(wtemp,p,i,j,tmp1,dim) ;
dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] *wtemp[fencode_hde1(p,i,j,tmp3)] - wtemp[fencode_hde1(p,i,j,hdnul)] *wtemp[fencode_hde1(p,i,j,tmp2)])/rdx;

    wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hde1(p,i,j,field)]; 

  }
}
__syncthreads();



   
/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hde1(p,i,j,field)]; 
//wmod[fencode_hde1(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=dwn1[fencode_hde1(p,i,j,f2)];
                              //dwn1[fencode_hde1(p,i,j,f)]=0;
                         }
              //  }	
}
  __syncthreads();*/



 
}


__global__ void hyperdifesource2a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

 


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

if(i<((p->n[0])) && j<((p->n[1])))
//  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


 
//dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wtemp[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             )/rdx;
//dwn1[fencode_hde1(p,i,j,field)]=( wd[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wd[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             );

//wtemp[fencode_hde1(p,i,j,tmp2)]= grad1r_hde1(wtemp,p,i,j,tmp1,dim) ;
//wtemp[fencode_hde1(p,i,j,tmp3)]= grad1l_hde1(wtemp,p,i,j,tmp1,dim) ;
dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] *wtemp[fencode_hde1(p,i,j,tmp3)] - wtemp[fencode_hde1(p,i,j,hdnul)] *wtemp[fencode_hde1(p,i,j,tmp2)])/rdx;


  }
}
__syncthreads();



   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hde1(p,i,j,field)]; 
//wmod[fencode_hde1(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=dwn1[fencode_hde1(p,i,j,f2)];
                              //dwn1[fencode_hde1(p,i,j,f)]=0;
                         }
              //  }	
}
  __syncthreads();



 
}

__global__ void hyperdifesource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

//if(i<((p->n[0])) && j<((p->n[1])))
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


 
//dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wtemp[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             )/rdx;
//dwn1[fencode_hde1(p,i,j,field)]=( wd[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wd[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             );

wtemp[fencode_hde1(p,i,j,tmp2)]= grad1l_hde1(wtemp,p,i,j,tmp1,dim) ;
wtemp[fencode_hde1(p,i,j,tmp3)]= grad1r_hde1(wtemp,p,i,j,tmp1,dim) ;


  }
}
__syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

if(i<((p->n[0])) && j<((p->n[1])))
//  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


 
//dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wtemp[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             )/rdx;
//dwn1[fencode_hde1(p,i,j,field)]=( wd[fencode_hde1(p,i,j,hdnur)] * grad1r_hde1(wtemp,p,i,j,tmp1,dim) - wd[fencode_hde1(p,i,j,hdnul)] *grad1l_hde1(wtemp,p,i,j,tmp1,dim)             );

//wtemp[fencode_hde1(p,i,j,tmp2)]= grad1r_hde1(wtemp,p,i,j,tmp1,dim) ;
//wtemp[fencode_hde1(p,i,j,tmp3)]= grad1l_hde1(wtemp,p,i,j,tmp1,dim) ;
dwn1[fencode_hde1(p,i,j,field)]=( wtemp[fencode_hde1(p,i,j,hdnur)] *wtemp[fencode_hde1(p,i,j,tmp3)] - wtemp[fencode_hde1(p,i,j,hdnul)] *wtemp[fencode_hde1(p,i,j,tmp2)])/rdx;


  }
}
__syncthreads();*/



   
 /*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hde1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hde1(p,i,j,field)]; 
//wmod[fencode_hde1(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=dwn1[fencode_hde1(p,i,j,f2)];
                              //dwn1[fencode_hde1(p,i,j,f)]=0;
                         }
              //  }	
}
  __syncthreads();*/



 
}



__global__ void hyperdifesource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
int shift=order*NVAR*(p->n[0])*(p->n[1]);
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode_hde1(p,i,j,f)]=0.0;
    dwn1[fencode_hde1(p,i,j,field)]=0.0;
   }
}
 __syncthreads();

  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

if(i<((p->n[0])) && j<((p->n[1])))
 // if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  {

#ifdef USE_SAC
     wtemp[fencode_hde1(p,i,j,tmp1)]=wmod[shift+fencode_hde1(p,i,j,energy)]-0.5*(


(wmod[shift+fencode_hde1(p,i,j,b1)]*wmod[shift+fencode_hde1(p,i,j,b1)]+wmod[shift+fencode_hde1(p,i,j,b2)]*wmod[shift+fencode_hde1(p,i,j,b2)])

+((wmod[shift+fencode_hde1(p,i,j,mom1)]*wmod[shift+fencode_hde1(p,i,j,mom1)]+wmod[shift+fencode_hde1(p,i,j,mom2)]*wmod[shift+fencode_hde1(p,i,j,mom2)])/(wmod[shift+fencode_hde1(p,i,j,rho)]+wmod[shift+fencode_hde1(p,i,j,rhob)])));
#else

     wtemp[fencode_hde1(p,i,j,tmp1)]=wmod[shift+fencode_hde1(p,i,j,energy)]-0.5*((wmod[shift+fencode_hde1(p,i,j,b1)]*wmod[shift+fencode_hde1(p,i,j,b1)]+wmod[shift+fencode_hde1(p,i,j,b2)]*wmod[shift+fencode_hde1(p,i,j,b2)])

+((wmod[shift+fencode_hde1(p,i,j,mom1)]*wmod[shift+fencode_hde1(p,i,j,mom1)]+wmod[shift+fencode_hde1(p,i,j,mom2)]*wmod[shift+fencode_hde1(p,i,j,mom2)])/(wmod[shift+fencode_hde1(p,i,j,rho)]))
);

#endif
 


  }
}
__syncthreads();




 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hde1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifesource1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim,real dt)
{


 dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifesource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
      hipDeviceSynchronize();

     hyperdifesource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
      hipDeviceSynchronize();

     hyperdifesource2a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
      hipDeviceSynchronize();

     hyperdifesource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,dt);
      hipDeviceSynchronize();


}







