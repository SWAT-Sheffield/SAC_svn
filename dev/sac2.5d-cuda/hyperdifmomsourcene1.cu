#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdmne1.cuh"

__global__ void hyperdifmomsourcene6_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));



  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

			 //if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                        if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                              - sign here same as vac maybe a +
                             ;//wmod[fencode_hdmne1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdmne1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdmne1(p,i,j,mom1+ii0)]; 
wmod[fencode_hdmne1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdmne1(p,i,j,mom1+ii0)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdmne1(p,i,j,mom1+ii0)];
                             wmod[fencode_hdmne1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdmne1(p,i,j,energy)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdmne1(p,i,j,energy)]; 

                         }
              //  }	
}
  __syncthreads();


  



}



__global__ void hyperdifmomsourcene5_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


 

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));



  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
//if( i<((p->n[0])) && j<((p->n[1])))
{
 // if(i<((p->n[0])) && j<((p->n[1])))
    dwn1[fencode_hdmne1(p,i,j,mom1+ii0)]=(grad1_hdmne1(wtemp,p,i,j,tmp7,ii));
// dwn1[fencode_hdmne1(p,i,j,mom1+ii0)]=0.0;



  }
}
 __syncthreads();

  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
 if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
 // if(i<((p->n[0])) && j<((p->n[1])))
     dwn1[fencode_hdmne1(p,i,j,energy)]=(grad1_hdmne1(wtemp,p,i,j,tmp8,ii));



}
 __syncthreads();


  



}

__global__ void hyperdifmomsourcene4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  int shift=order*NVAR*(p->n[0])*(p->n[1]);

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));



  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  if(i<((p->n[0])) && j<((p->n[1])))
	{		               
     wtemp[fencode_hdmne1(p,i,j,tmp7)]=wtemp[fencode_hdmne1(p,i,j,tmp1)]*wtemp[fencode_hdmne1(p,i,j,tmp6)];

     wtemp[fencode_hdmne1(p,i,j,tmp8)]=wtemp[fencode_hdmne1(p,i,j,tmp6)]*wmod[(shift)+fencode_hdmne1(p,i,j,mom1+ii0)];



//dwn1[(NVAR*(p->n[0])*(p->n[1]))+fencode_hdmne1(p,i,j,energy)]=(grad_hdmne1(wtemp,p,i,j,tmp4,ii));

   }
}
 __syncthreads();




  



}


__global__ void hyperdifmomsourcene3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));



  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  if(i<((p->n[0])) && j<((p->n[1])))
  {

     wtemp[fencode_hdmne1(p,i,j,tmp6)]=wtemp[fencode_hdmne1(p,i,j,tmp5)]*((wd[fencode_hdmne1(p,i,j,hdnur)]+wd[fencode_hdmne1(p,i,j,hdnul)]))/4.0;





   }
}
__syncthreads();




  



}

__global__ void hyperdifmomsourcene2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;

   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));



  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  //if(i<((p->n[0])) && j<((p->n[1])))
      wtemp[fencode_hdmne1(p,i,j,tmp5)]=(grad1_hdmne1(wtemp,p,i,j,tmp4,dim));

}
__syncthreads();

}


__global__ void hyperdifmomsourcene1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  
int shift=order*NVAR*(p->n[0])*(p->n[1]);
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


  //init rhol and rhor
  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode_hdmne1(p,i,j,f)]=0.0;

     dwn1[fencode_hdmne1(p,i,j,energy)]=0.0;
     dwn1[fencode_hdmne1(p,i,j,mom1+ii0)]=0.0;

   }
}
 __syncthreads();


  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  if(i<((p->n[0])) && j<((p->n[1])))
  {

#ifdef USE_SAC
     wtemp[fencode_hdmne1(p,i,j,tmp1)]=wmod[(shift)+fencode_hdmne1(p,i,j,rho)]+wmod[(shift)+fencode_hdmne1(p,i,j,rhob)];

     wtemp[fencode_hdmne1(p,i,j,tmp4)]=wmod[(shift)+fencode_hdmne1(p,i,j,mom1+field)]/(wmod[(shift)+fencode_hdmne1(p,i,j,rho)]+wmod[(shift)+fencode_hdmne1(p,i,j,rhob)]);
#else
     wtemp[fencode_hdmne1(p,i,j,tmp1)]=wmod[(shift)+fencode_hdmne1(p,i,j,rho)];
     wtemp[fencode_hdmne1(p,i,j,tmp4)]=wmod[(shift)+fencode_hdmne1(p,i,j,mom1+field)]/wmod[fencode_hdmne1(p,i,j,rho)];
#endif



   }
}
__syncthreads();


}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdmne1ne(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsourcene1(struct params **p, struct params **d_p, real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{

 dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifmomsourcene1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene5_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene6_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();



}







