#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdv1.cuh"

__device__ __host__
void bc_hyperdif(real *wt, struct params *p,int i, int j, int f,int dir) {




 
   if(  (dir == 0) && (i==(p->n[0])-1)   && j>0   && j<(p->n[1])           )
         wt[fencode_hdv1(p,i+2,j,f)]=wt[fencode_hdv1(p,(p->n[0])-5,j+1,f)];
   else if((dir == 1) && (j==(p->n[1])-1)    && i>0   && i<((p->n[0]))  )
       wt[fencode_hdv1(p,i,j+2,f)]=wt[fencode_hdv1(p,i+1,(p->n[1])-5,f)];
  else if((dir == 0) && (i==0)    && j>0   && j<((p->n[1]))   )
       wt[fencode_hdv1(p,0,j+1,f)]=wt[fencode_hdv1(p,6,j+1,f)];
   else if((dir == 1) && (j==0)    && i>0   && i<((p->n[0]))   )
       wt[fencode_hdv1(p,i+1,0,f)]=wt[fencode_hdv1(p,i+1,6,f)];
 
}


__device__ __host__
void bc_periodic1_temp2(real *wt, struct params *p,int i, int j, int f) {

                if(i==1 )                
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,6,j,f)];
                else if((i==((p->n[0]))) )                
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i-4,j,f)];
                else if(j==1  )                
                  wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,6,f)];
                else if((j==((p->n[1]))) )                
                  wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,j-4,f)];
}

__device__ __host__
void bc_periodic2_temp2(real *wt, struct params *p,int i, int j, int f) {


               if(i<1 && j<1)
                {
                  if(i==j)
                    //wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,6,f)];
                  else                  
                    //wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,6,j,f)];                                    
                }
                else if(i<1 && j>((p->n[1])-1))
                {
                  if(i==(j-(p->n[1])-1))                  
                    //wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,6,j,f)];                                     
                  else                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,j-6,f)];                                     
                }
                else if(i>((p->n[0])-1) && j<1)
                {
                  if((i-(p->n[0])+1)==j)                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i-5,j,f)];                                    
                  else                  
                   wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,4,f)];                                    
                }
                else if(i>((p->n[0])-1) && j>((p->n[1])-1))
                {
                  if(i==j)                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,j-5,f)];                                    
                  else                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i-5,j,f)];                                    
                }                       
                 
                




}


__global__ void hyperdifvisc4_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


int shift=order*NVAR*(p->n[0])*(p->n[1]);


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l





 /*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv1(wtemp,p,i,j,tmp4);
                  bc_periodic1_hdv1(wtemp,p,i,j,tmp5);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv1(wtemp,p,i,j,tmp4);
                  bc_periodic2_hdv1(wtemp,p,i,j,tmp5);
             }
}
                __syncthreads();*/



   p->maxviscoef=0;


    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r
   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;


   if(i>1 && i<((p->n[0])-2) && j>1 && j<((p->n[1])-2))
   {
     //wd[fencode_hdv1(p,i,j,hdnur+hand)]=wtemp2[fencode_hdv1(p,i+1,j+1,tmpnui)];
     if(wtemp[fencode_hdv1(p,i,j,tmp5)]>0)
{
//p->cmax=1.0;
	wd[fencode_hdv1(p,i,j,hdnur+hand)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp[field])*wtemp[fencode_hdv1(p,i,j,tmp4)]/wtemp[fencode_hdv1(p,i,j,tmp5)];

          //wd[fencode_hdv1(p,i,j,hdnur+hand)]=wtemp[fencode_hdv1(p,i,j,tmp4)];
	//wd[fencode_hdv1(p,i,j,hdnul+hand)]=0.01;
}
     else
        wd[fencode_hdv1(p,i,j,hdnur+hand)]=0;

    

     //temporary trap for debugging
     //if(wd[fencode_hdv1(p,i,j,hdnul+hand)]>0.02 ||wd[fencode_hdv1(p,i,j,hdnul+hand)] < -0.02)
     //                                                     wd[fencode_hdv1(p,i,j,hdnul+hand)]=0.0;

   }
}
 __syncthreads();



 
}





__global__ void hyperdifvisc3_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt1=0,max3=0, maxt2=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


int shift=order*NVAR*(p->n[0])*(p->n[1]);


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l





  //compute md3r and md1r
//tmp4    md3r
//tmp5    md1r
  //js=0;
 // is=0;
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if( i>1 && j>1 && i<((p->n[0])-2) && j<((p->n[1])-2))            
   {
         maxt1=0;
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp1[fencode_hdv1(p,i+1+is,j+1+js,d3)]>maxt1)
                         maxt1=wtemp1[fencode_hdv1(p,i+1+is,j+1+js,d3)];

                }
          wtemp[fencode_hdv1(p,i,j,tmp4)]=maxt1;

         maxt2=0;
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp1[fencode_hdv1(p,i+1+is,j+1+js,d1)]>maxt2)
                        maxt2=wtemp1[fencode_hdv1(p,i+1+is,j+1+js,d1)];

                }
          wtemp[fencode_hdv1(p,i,j,tmp5)]=maxt2;
   }
}
   __syncthreads();







 
}




__global__ void hyperdifvisc2_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


int shift=order*NVAR*(p->n[0])*(p->n[1]);






   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
 
   if(i>1 && j>1 && i<((p->n[0])) && j<((p->n[1])))
   //if(i<((p->n[0])-1) && j<((p->n[1])-1))            
   { 
     if(hand==0)
     {
           wtemp1[fencode_hdv1(p,i,j,d3)]=fabs(3.0*(wtemp2[fencode_hdv1(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv1(p,i,j,tmpnui)] ) - (wtemp2[fencode_hdv1(p,i+2*(dim==0),j+2*(dim==1),tmpnui)] - wtemp2[fencode_hdv1(p,i-(dim==0),j-(dim==1),tmpnui)]    ));
     }
     else
     {
          // wtemp1[fencode_hdv1(p,i,j,d3)]=fabs(3.0*(wtemp2[fencode_hdv1(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv1(p,i,j,tmpnui)]) - (wtemp2[fencode_hdv1(p,i+2*(dim==0),j+2*(dim==1),tmpnui)] - wtemp2[fencode_hdv1(p,i-(dim==0),j-(dim==1),tmpnui)]    ));
           wtemp1[fencode_hdv1(p,i,j,d3)]=fabs(3.0*(wtemp2[fencode_hdv1(p,i,j,tmpnui)] - wtemp2[fencode_hdv1(p,i-(dim==0),j-(dim==1),tmpnui)]) - (wtemp2[fencode_hdv1(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv1(p,i-2*(dim==0),j-2*(dim==1),tmpnui)]    ));
     }
   }
}
   __syncthreads();
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

   if(i>0 && j>0 && i<=((p->n[0])) && j<=((p->n[1])))            
   { 
     if(hand==0)
     {

           wtemp1[fencode_hdv1(p,i,j,d1)]=fabs((wtemp2[fencode_hdv1(p,i+(dim==0),j+(dim==1),tmpnui)] - wtemp2[fencode_hdv1(p,i,j,tmpnui)] ));
     }
     else
     {
           wtemp1[fencode_hdv1(p,i,j,d1)]=fabs((wtemp2[fencode_hdv1(p,i,j,tmpnui)] - wtemp2[fencode_hdv1(p,i-(dim==0),j-(dim==1),tmpnui)] ));
     }
   }
}
   __syncthreads();



/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic1_hdv1(wtemp,p,i,j,tmp2);
                  bc_periodic1_hdv1(wtemp,p,i,j,tmp3);
             }

}
                __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
          if( i<(ni) && j<(nj))
            {
                  bc_periodic2_hdv1(wtemp,p,i,j,tmp2);
                  bc_periodic2_hdv1(wtemp,p,i,j,tmp3);
             }
}
                __syncthreads();*/








 
}



__global__ void hyperdifvisc1a_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
//int numBlocks = (ni*nj+numThreadsPerBlock-1) / numThreadsPerBlock;
  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);
  __shared__ real wts[512];
  __shared__ real wms[512];



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if( i<((p->n[0])) && j<((p->n[1])))
   {
     //tmp6 is tmp_nuI
     wtemp2[fencode_hdv1(p,i+1,j+1,tmpnui)]=wtemp[fencode_hdv1(p,i,j,tmp6)];

   }

   }
   __syncthreads();








 
}


__global__ void hyperdifvisc1_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
//int numBlocks = (ni*nj+numThreadsPerBlock-1) / numThreadsPerBlock;
  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*(p->n[0])*(p->n[1]);
  __shared__ real wts[512];
  __shared__ real wms[512];




//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {


        for(int f=tmp1; f<=tmp8; f++)
                 wtemp[fencode_hdv1(p,i,j,f)]=0;

        for(int f=d1; f<=d3; f++)
                 wtemp1[fencode_hdv1(p,i,j,f)]=0;
      wtemp2[fencode_hdv1(p,i,j,tmpnui)]=0;
      if(i==((p->n[0])-1))
      {
        for(int f=d1; f<=d3; f++)
                 wtemp1[fencode_hdv1(p,i+1,j,f)]=0;
        wtemp2[fencode_hdv1(p,i+1,j,tmpnui)]=0;
        wtemp2[fencode_hdv1(p,i+2,j,tmpnui)]=0;
      }
      if(j==((p->n[1])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[fencode_hdv1(p,i,j+1,f)]=0;
          wtemp2[fencode_hdv1(p,i,j+1,tmpnui)]=0;
          wtemp2[fencode_hdv1(p,i,j+2,tmpnui)]=0;
      }
      if(j==((p->n[1])-1)  && i==j)
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[fencode_hdv1(p,i+1,j+1,f)]=0;
          for(int di=0; di<2; di++)
             for(int dj=0; dj<2; dj++)
                wtemp2[fencode_hdv1(p,i+1+di,j+1+dj,tmpnui)]=0;

      }



   }



  }

   __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {

        //for(iv=0;iv<NVAR;iv++)
        //               wms[tid+iv*blockdim]=wmod[fencode_hdv1(p,i,j,iv)+shift];
        //wts[tid]=wtemp[fencode_hdv1(p,i,j,tmp6)];
        //temp value for viscosity

       //tmp6  tmpnu
#ifdef USE_SAC
	if((field ==mom1 || field == mom2))
		wtemp[fencode_hdv1(p,i,j,tmp6)]=wmod[fencode_hdv1(p,i,j,field)+shift]/(((wmod[fencode_hdv1(p,i,j,rho)+shift] +wmod[fencode_hdv1(p,i,j,rhob)+shift])));
               //wts[tid]=wms[tid+field*blockdim]/(((wms[tid+rho*blockdim] +wms[tid+rhob*blockdim])));
     	else if(field !=energy)
        	wtemp[fencode_hdv1(p,i,j,tmp6)]=wmod[fencode_hdv1(p,i,j,field)+shift];///(bfac2+bfac3*((wmod[fencode_hdv1(p,i,j,rho)+shift] +wmod[fencode_hdv1(p,i,j,rhob)+shift])));
               //wts[tid]=wms[tid+field*blockdim];///(bfac2+bfac3*((wmod[fencode_hdv1(p,i,j,rho)+shift] +wmod[fencode_hdv1(p,i,j,rhob)+shift])));
     	else
        wtemp[fencode_hdv1(p,i,j,tmp6)]=wmod[fencode_hdv1(p,i,j,energy)+shift]-0.5*(wmod[fencode_hdv1(p,i,j,b1)+shift]*wmod[fencode_hdv1(p,i,j,b1)+shift]+wmod[fencode_hdv1(p,i,j,b2)+shift]*wmod[fencode_hdv1(p,i,j,b2)+shift])+(wmod[fencode_hdv1(p,i,j,mom1)+shift]*wmod[fencode_hdv1(p,i,j,mom1)+shift]+wmod[fencode_hdv1(p,i,j,mom2)+shift]*wmod[fencode_hdv1(p,i,j,mom2)+shift])/(wmod[fencode_hdv1(p,i,j,rho)+shift]+wmod[fencode_hdv1(p,i,j,rhob)+shift] );
//wts[tid]=wms[tid+energy*blockdim]-0.5*(wms[tid+b1*blockdim]*wms[tid+b1*blockdim]+wms[tid+b2*blockdim]*wms[tid+b2*blockdim])+(wms[tid+mom1*blockdim]*wms[tid+mom1*blockdim]+wms[tid+mom2*blockdim]*wms[tid+mom2*blockdim])/(wms[tid+rho*blockdim]+wms[tid+rhob*blockdim] );

#else
	if((field ==mom1 || field == mom2))
		wtemp[fencode_hdv1(p,i,j,tmp6)]=wmod[fencode_hdv1(p,i,j,field)+shift]/(((wmod[fencode_hdv1(p,i,j,rho)+shift] )));
     else if(field !=energy)
        wtemp[fencode_hdv1(p,i,j,tmp6)]=wmod[fencode_hdv1(p,i,j,field)+shift]/(bfac2+bfac3*(wmod[fencode_hdv1(p,i,j,rho)+shift] ));

     else
        wtemp[fencode_hdv1(p,i,j,tmp6)]=wmod[fencode_hdv1(p,i,j,energy)+shift]-0.5*(wmod[fencode_hdv1(p,i,j,b1)+shift]*wmod[fencode_hdv1(p,i,j,b1)+shift]+wmod[fencode_hdv1(p,i,j,b2)+shift]*wmod[fencode_hdv1(p,i,j,b2)+shift])+(wmod[fencode_hdv1(p,i,j,mom1)+shift]*wmod[fencode_hdv1(p,i,j,mom1)+shift]+wmod[fencode_hdv1(p,i,j,mom2)+shift]*wmod[fencode_hdv1(p,i,j,mom2)+shift])/(wmod[fencode_hdv1(p,i,j,rho)+shift] );

#endif

        //for(iv=0;iv<NVAR;iv++)
        //               wmod[fencode_hdv1(p,i,j,iv)+shift]=wms[tid+iv*blockdim];
        //          wtemp[fencode_hdv1(p,i,j,tmp6)]=wts[tid];

       // wd[fencode_hdv1(p,i,j,hdnur)]=0;
        wd[fencode_hdv1(p,i,j,hdnul+hand)]=0;
   }

}
   __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if(i<((p->n[0])) && j<((p->n[1])))
   {
	
        bc_hyperdif(wtemp2, p,i, j, tmpnui,dim);

   }


    }
   __syncthreads();

/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
    //set viscosities
   if( i<((p->n[0])) && j<((p->n[1])))
   {
     //tmp6 is tmp_nuI
     wtemp2[fencode_hdv1(p,i+1,j+1,tmpnui)]=wtemp[fencode_hdv1(p,i,j,tmp6)];

   }

   }
   __syncthreads();
*/







 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc1(struct params **p,  struct params **d_p,   real **d_wmod,  real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim,int hand)
{



 dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifvisc1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();

}







