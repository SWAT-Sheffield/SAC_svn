#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int encode_u (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_u (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}



__global__ void update_parallel(struct params *p, float *b, float *w, float *wnew)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;


  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  float *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;
     j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  //if(i>2 && j >2 && i<((p->ni)-3) && j<((p->nj)-3))

  if(i<p->ni && j<p->nj)
	{
             for(int f=rho; f<=b3; f++)               
                  w[fencode_u(p,i,j,f)]=wnew[fencode_u(p,i,j,f)];
            // u[i+j*ni]=un[i+j*ni];
           // v[i+j*ni]=vn[i+j*ni];
	   // h[i+j*ni]=hn[i+j*ni];
	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_u(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}


int cuupdate(struct params **p, float **w, float **wnew, float **b,struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // prop_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     //hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
 hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


int cufinish(struct params **p, float **w, float **wnew, float **b, struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{
  

 hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  checkErrors_u("copy data from device");


  hipFree(*d_p);

  hipFree(*d_w);
  hipFree(*d_wnew);
  hipFree(*d_b);

  hipFree(*d_wmod);
  hipFree(*d_dwn1);
  hipFree(*d_wd);



}
