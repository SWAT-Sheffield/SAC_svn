#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int encode_db (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_db (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
real evalgrad_db(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_db;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx)))*(fi-fim1)):((1.0/(12.0*(p->dx)))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
     // return((2.0/(1.0*(p->dy)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dy)))*(fi-fim1)):((1.0/(12.0*(p->dy)))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_db(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_db;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx)))*(wmod[fencode_db(p,i+1,j,field)]-wmod[fencode_db(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_db(p,i+1,j,field)]-8*wmod[fencode_db(p,i-1,j,field)]+wmod[fencode_db(p,i-1,j,field)]-wmod[fencode_db(p,i+1,j,field)])/6.0):wmod[fencode_db(p,i+1,j,field)]-wmod[fencode_db(p,i-1,j,field)])/(2.0*(p->dx))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dy)))*(wmod[fencode_db(p,i,j+1,field)]-wmod[fencode_db(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_db(p,i,j+1,field)]-8*wmod[fencode_db(p,i,j-1,field)]+wmod[fencode_db(p,i,j-1,field)]-wmod[fencode_db(p,i,j+1,field)])/6.0):wmod[fencode_db(p,i,j+1,field)]-wmod[fencode_db(p,i,j-1,field)])/(2.0*(p->dy))    );

 }

 return -1;
}

__device__ __host__
real dbsourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  real src=0;

  
 
  return src;
}

__device__ __host__
real dbsourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,b1)];
	break;
	case 1:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,b2)];
	break;
	case 2:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,b3)];
	break;
  }

  return(isnan(src)?0:src);


}

__device__ __host__
real dbsourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,mom1)]/w[fencode_db(p,ix,iy,rho)];
	break;
	case 1:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,mom2)]/w[fencode_db(p,ix,iy,rho)];
	break;
	case 2:
         src= -wd[fencode_db(p,ix,iy,divb)]*w[fencode_db(p,ix,iy,mom3)]/w[fencode_db(p,ix,iy,rho)];
	break;
  }
   return(isnan(src)?0:src);
}

__device__ __host__
real dbsourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 real src=0;
    src= -wd[fencode_db(p,ix,iy,divb)]*wd[fencode_db(p,ix,iy,bdotv)];
 
  return ( src);
}


__device__ __host__
int dbderivsourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourcerho(dw,wd,w,p,ix,iy);
     	//dw[fencode_db(p,ix,iy,field)]=w[fencode_db(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int dbderivsourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode_db(p,ix,iy,field)]=w[fencode_db(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourcemom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_db(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int dbderivsourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourceb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int dbderivsourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode_db(p,ix,iy,field)]=dw[fencode_db(p,ix,iy,field)]+dbsourceenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void dbderivsource (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      dbderivsourcerho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      dbderivsourcemom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
      dbderivsourcemom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      dbderivsourcemom(dw,wd,w,p,ix,iy,field,2);
     break;
     case energy:
       dbderivsourceenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
      dbderivsourceb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      dbderivsourceb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      dbderivsourceb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}


__global__ void divb_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  __shared__ int ntot;

  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;
  real *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  real *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;
     j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  //if(i>2 && j >2 && i<((p->ni)-3) && j<((p->nj)-3))



  if(i<p->ni && j<p->nj)
	{
           if(p->divbfix)
           {    
               for(int f=rho; f<=b3; f++)
               {              
                  dbderivsource(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_ds(p,i,j,f)]=1.0;
                  __syncthreads();
               }
            }
            // u[i+j*ni]=un[i+j*ni];
           // v[i+j*ni]=vn[i+j*ni];
	   // h[i+j*ni]=hn[i+j*ni];
	}
 __syncthreads();



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_db(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int cudivb(struct params **p, real **w, real **wnew,  struct state **state,struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
    // prop_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     //hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
    divb_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
    //hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
   // hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);

//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}



