#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_b.cuh"
__device__ __host__
void bc_cont(real *wt, struct params *p,int i, int j, int f) {

                if(i<2 && j<2)
                {
                  if(i==j)
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i+2,j,f)];
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,j+2,f)];                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i+2,j,f)];                  
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,(j-3),f)];                  
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(i-3),j,f)];                  
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,j+2,f)];                  
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(i-3),j,f)];                   
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,(j-3),f)];                  
                }                       
                else if(i==0 || i==1)                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i+2,j,f)];                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(i-3),j,f)];                
                else if(j==0 || j==1)                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,j+2,f)];                
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,(j-3),f)];
                




}

__device__ __host__
void bc_fixed(real *wt, struct params *p,int i, int j, int f, real val) {


                //(UPPER or LOWER)*NDIM*NVAR+dim*NVAR+varnum = picks out correct value for fixed BC
                //for array of values for fixed BC's

                if(i<2 && j<2)
                {
                  if(i==j)
                    wt[fencode_b(p,i,j,f)]=val;
                  else                  
                    wt[fencode_b(p,i,j,f)]=val;                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    wt[fencode_b(p,i,j,f)]=val;                  
                  else                  
                    wt[fencode_b(p,i,j,f)]=val;                  
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_b(p,i,j,f)]=val;                  
                  else                  
                    wt[fencode_b(p,i,j,f)]=val;                  
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_b(p,i,j,f)]=val;                   
                  else                  
                    wt[fencode_b(p,i,j,f)]=val;                  
                }                       
                else if(i==0 || i==1)                
                  wt[fencode_b(p,i,j,f)]=val;                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                  wt[fencode_b(p,i,j,f)]=val;                
                else if(j==0 || j==1)                
                  wt[fencode_b(p,i,j,f)]=val;                
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))                
                  wt[fencode_b(p,i,j,f)]=val;
                




}

__device__ __host__
void bc_periodic(real *wt, struct params *p,int i, int j, int f) {

               if(i<2 && j<2)
                {
                  if(i==j)
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(p->n[0])-3+i,(p->n[1])-3+j,f)];
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(p->n[0])-3+i,(p->n[1])-3+j,f)];                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(p->n[0])-3+i,2+((p->n[1])-j),f)];                  
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(p->n[0])-3+i,2+((p->n[1])-j),f)];                  
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,2+((p->n[0])-i),(p->n[1])-3+j,f)];                  
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,2+((p->n[0])-i),(p->n[1])-3+j,f)];                  
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,2+((p->n[0])-i),2+((p->n[1])-j),f)];                   
                  else                  
                    wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,2+((p->n[0])-i),2+((p->n[1])-j),f)];                  
                }                       
                else if(i==0 || i==1)                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,(p->n[0])-3+i,j,f)];                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,2+((p->n[0])-i),j,f)];                
                else if(j==0 || j==1)                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,(p->n[1])-3+j,f)];                
               else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))                
                  wt[fencode_b(p,i,j,f)]=wt[fencode_b(p,i,2+((p->n[1])-j),f)];
                




}

__global__ void boundary_parallel(struct params *p, real *w, real *wnew, real *wd, real *wmod)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  



    j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->n[0] && j<p->n[1])
	{

               //default continuous BC for all
               //gradient kept zero by copying variable values from edge of mesh to ghost cells
                  bc_cont(wmod,p,i,j,rho);
                  bc_cont(wnew,p,i,j,rho);
               //   bc_fixed(wmod,p,i,j,rho,1.0);
               //   bc_fixed(wnew,p,i,j,rho,1.0);
               //   bc_periodic(wmod,p,i,j,rho);
               //   bc_periodic(wnew,p,i,j,rho);

               
               for(int f=rho+1; f<NVAR; f++)
               {

                  bc_cont(wmod,p,i,j,f);
                  bc_cont(wnew,p,i,j,f);

                 // bc_fixed(wmod,p,i,j,f,val);
                 // bc_fixed(wnew,p,i,j,f,val);

                //  bc_periodic(wmod,p,i,j,f);
                //  bc_periodic(wnew,p,i,j,f);


               }

               for(int f=vel1; f<NDERV; f++)
               {
                  bc_cont(wd,p,i,j,f);

                // bc_fixed(wd,p,i,j,f,val);
                 //   bc_periodic(wd,p,i,j,f);

                  
               }

		
               /*if(i==0 )
		{
			un[j*ni] = 2.5*un[1+j*ni] - 2*un[2+j*ni] + 0.5*un[3+j*ni];
			un[ni+j*ni] = 2.5*un[ni-1+j*ni] - 2*un[ni-2+ni*j] + 0.5*un[ni-3+j*ni];
			vn[j*ni] = 2.5*vn[1+j*ni] - 2*vn[2+j*ni] + 0.5*vn[3+j*ni];
		 	vn[ni+j*ni] = 2.5*vn[ni-1+j*ni] - 2*vn[ni-2+ni*j] + 0.5*vn[ni-3+j*ni];
		 	hn[j*ni] = 2.5*hn[1+j*ni] - 2*hn[2+j*ni] + 0.5*hn[3+j*ni];
			hn[ni+j*ni] = 2.5*hn[ni-1+j*ni] - 2*hn[ni-2+ni*j] + 0.5*hn[ni-3+j*ni];
		}

		if(j==0)
		{
			un[i+ni] = 2.5*un[i+1*ni] - 2*un[i+2*ni] + 0.5*un[i+3*ni];
			un[i+(nj )*ni] = 2.5*un[i+(nj-1)*ni] - 2*un[i+(nj-2)*ni] + 0.5*un[i+(nj-3)*ni];
			vn[i+ni] = 2.5*vn[i+1*ni] - 2*vn[i+2*ni] + 0.5*vn[i+3*ni];
			vn[i+(nj)*ni] = 2.5*vn[i+(nj-1)*ni] - 2*vn[i+(nj-2)*ni] + 0.5*vn[i+(nj-3)*ni];
			hn[i+ni] = 2.5*hn[i+1*ni] - 2*hn[i+2*ni] + 0.5*hn[i+3*ni];
			hn[i+(nj)*ni] = 2.5*hn[i+(nj-1)*ni] - 2*hn[i+(nj-2)*ni] + 0.5*hn[i+(nj-3)*ni];
		}*/
	}
 __syncthreads();
  
}

int cuboundary(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   //int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
int numBlocks = ((dimproduct_b(*p)+numThreadsPerBlock-1)) / numThreadsPerBlock;
//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
 	    //printf("called prop\n"); 
    // hipDeviceSynchronize();
    boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wd, *d_wmod);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
	    //printf("called update\n"); 
    hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

