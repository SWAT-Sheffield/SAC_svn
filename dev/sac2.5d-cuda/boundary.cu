#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_b.cuh"

__global__ void boundary_parallel(struct params *p, real *w, real *wnew, real *wd, real *wmod, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int f;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
                real val=0;
  



    j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->n[0] && j<p->n[1])
	{

               //default continuous BC for all
               //gradient kept zero by copying variable values from edge of mesh to ghost cells
                //  bc_cont_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho);
               
                 // bc_fixed_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho,1.0);
               //   bc_fixed(wnew,p,i,j,rho,1.0);
               //   bc_periodic_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,rho);
               

               
               for( f=rho; f<=b2; f++)
               {

                  //bc_cont_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
                

                //  bc_fixed_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,0.0);
                 // bc_fixed(wnew,p,i,j,f,val);

                  bc_periodic1_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
                


               }

               /*for(int f=vel1; f<NDERV; f++)
               {
                  bc_cont_b(wd,p,i,j,f);

                 //bc_fixed_b(wd,p,i,j,f,0.0);
                 //   bc_periodic(wd,p,i,j,f);

                  
               }*/

	}
 __syncthreads();


  //This second call makes sure corners are set correctly
  if(i<p->n[0] && j<p->n[1])
             for( f=rho; f<=b2; f++)
                  bc_periodic2_b(wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f); 
 __syncthreads();



  
}

int cuboundary(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   //int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
int numBlocks = ((dimproduct_b(*p)+numThreadsPerBlock-1)) / numThreadsPerBlock;
//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
 	    //printf("called prop\n"); 
    // hipDeviceSynchronize();
    boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wd, *d_wmod, order);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
	    //printf("called update\n"); 
    hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

