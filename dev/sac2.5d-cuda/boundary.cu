#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int dimproduct_b (struct params *dp) {

  int tot=1;
  for(int i=0;i<NDIM;i++)
    tot*=dp->n[i];
  return tot; 
}


__device__ __host__
int encode_b (struct params *dp,int ix, int iy) {

  return (iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int encode3_b (struct params *dp,int ix, int iy, int iz) {

  return (iz*((dp)->n[0])*((dp)->n[1])  + iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int fencode_b (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return(( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1]))));
}


__device__ __host__
int fencode3_b (struct params *dp,int ix, int iy, int iz, int field) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return(  iz*((dp)->n[0])*((dp)->n[1])+ (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])*((dp)->n[2]))  );
}


__global__ void boundary_parallel(struct params *p, real *w, real *wnew, real *wd, real *wmod)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  



    j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->n[0] && j<p->n[1])
	{

               //default continuous BC for all
               //gradient kept zero by copying variable values from edge of mesh to ghost cells
               
               for(int f=rho; f<=b3; f++)
               {
                   
                if(i==0 || i==1)
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,2,j,f)];
                  wmod[fencode_b(p,i,j,f)]=wmod[fencode_b(p,2,j,f)];
                }
                if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,((p->n[0])-3),j,f)];
                  wmod[fencode_b(p,i,j,f)]=wmod[fencode_b(p,((p->n[0])-3),j,f)];
                }
                if(j==0 || j==1)
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,i,2,f)];
                  wmod[fencode_b(p,i,j,f)]=wmod[fencode_b(p,i,2,f)];
                }
                if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,i,((p->n[1])-3),f)];
                  wmod[fencode_b(p,i,j,f)]=wnew[fencode_b(p,i,((p->n[1])-3),f)];
                }

                  
               }

               for(int f=current1; f<=cfast; f++)
               {
                                      
                if(i==0 || i==1)
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,2,j,f)];
                if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,((p->n[0])-3),j,f)];
                if(j==0 || j==1)
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,i,2,f)];
                if((j==((p->n[1])-1)) || (j==((p->n[1])-2)))
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,i,((p->n[1])-3),f)];
                
                  
               }

		
               /*if(i==0 )
		{
			un[j*ni] = 2.5*un[1+j*ni] - 2*un[2+j*ni] + 0.5*un[3+j*ni];
			un[ni+j*ni] = 2.5*un[ni-1+j*ni] - 2*un[ni-2+ni*j] + 0.5*un[ni-3+j*ni];
			vn[j*ni] = 2.5*vn[1+j*ni] - 2*vn[2+j*ni] + 0.5*vn[3+j*ni];
		 	vn[ni+j*ni] = 2.5*vn[ni-1+j*ni] - 2*vn[ni-2+ni*j] + 0.5*vn[ni-3+j*ni];
		 	hn[j*ni] = 2.5*hn[1+j*ni] - 2*hn[2+j*ni] + 0.5*hn[3+j*ni];
			hn[ni+j*ni] = 2.5*hn[ni-1+j*ni] - 2*hn[ni-2+ni*j] + 0.5*hn[ni-3+j*ni];
		}

		if(j==0)
		{
			un[i+ni] = 2.5*un[i+1*ni] - 2*un[i+2*ni] + 0.5*un[i+3*ni];
			un[i+(nj )*ni] = 2.5*un[i+(nj-1)*ni] - 2*un[i+(nj-2)*ni] + 0.5*un[i+(nj-3)*ni];
			vn[i+ni] = 2.5*vn[i+1*ni] - 2*vn[i+2*ni] + 0.5*vn[i+3*ni];
			vn[i+(nj)*ni] = 2.5*vn[i+(nj-1)*ni] - 2*vn[i+(nj-2)*ni] + 0.5*vn[i+(nj-3)*ni];
			hn[i+ni] = 2.5*hn[i+1*ni] - 2*hn[i+2*ni] + 0.5*hn[i+3*ni];
			hn[i+(nj)*ni] = 2.5*hn[i+(nj-1)*ni] - 2*hn[i+(nj-2)*ni] + 0.5*hn[i+(nj-3)*ni];
		}*/
	}
 __syncthreads();
  
}

int cuboundary(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   //int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
int numBlocks = ((dimproduct_b(*p)+numThreadsPerBlock-1)) / numThreadsPerBlock;
//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
 	    //printf("called prop\n"); 
    // hipDeviceSynchronize();
    boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wd, *d_wmod);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
	    //printf("called update\n"); 
    hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

