#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__device__ __host__
int encode_b (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_b (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}


__global__ void boundary_parallel(struct params *p, real *w, real *wnew, real *wd, real *wmod)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;

  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;

  real *u,  *v,  *h;
  real *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;

    j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{

               //default continuous BC for all
               //gradient kept zero by copying variable values from edge of mesh to ghost cells
               
               for(int f=rho; f<=b3; f++)
               {
                   
                if(i==0 || i==1)
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,2,j,f)];
                  wmod[fencode_b(p,i,j,f)]=wmod[fencode_b(p,2,j,f)];
                }
                if((i==((p->ni)-1)) || (i==((p->ni)-2)))
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,((p->ni)-3),j,f)];
                  wmod[fencode_b(p,i,j,f)]=wmod[fencode_b(p,((p->ni)-3),j,f)];
                }
                if(j==0 || j==1)
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,i,2,f)];
                  wmod[fencode_b(p,i,j,f)]=wmod[fencode_b(p,i,2,f)];
                }
                if((j==((p->nj)-1)) || (j==((p->nj)-2)))
                {
                  wnew[fencode_b(p,i,j,f)]=wnew[fencode_b(p,i,((p->nj)-3),f)];
                  wmod[fencode_b(p,i,j,f)]=wnew[fencode_b(p,i,((p->nj)-3),f)];
                }

                  
               }

               for(int f=current1; f<=divb; f++)
               {
                                      
                if(i==0 || i==1)
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,2,j,f)];
                if((i==((p->ni)-1)) || (i==((p->ni)-2)))
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,((p->ni)-3),j,f)];
                if(j==0 || j==1)
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,i,2,f)];
                if((j==((p->nj)-1)) || (j==((p->nj)-2)))
                  wd[fencode_b(p,i,j,f)]=wd[fencode_b(p,i,((p->nj)-3),f)];
                
                  
               }

		
               /*if(i==0 )
		{
			un[j*ni] = 2.5*un[1+j*ni] - 2*un[2+j*ni] + 0.5*un[3+j*ni];
			un[ni+j*ni] = 2.5*un[ni-1+j*ni] - 2*un[ni-2+ni*j] + 0.5*un[ni-3+j*ni];
			vn[j*ni] = 2.5*vn[1+j*ni] - 2*vn[2+j*ni] + 0.5*vn[3+j*ni];
		 	vn[ni+j*ni] = 2.5*vn[ni-1+j*ni] - 2*vn[ni-2+ni*j] + 0.5*vn[ni-3+j*ni];
		 	hn[j*ni] = 2.5*hn[1+j*ni] - 2*hn[2+j*ni] + 0.5*hn[3+j*ni];
			hn[ni+j*ni] = 2.5*hn[ni-1+j*ni] - 2*hn[ni-2+ni*j] + 0.5*hn[ni-3+j*ni];
		}

		if(j==0)
		{
			un[i+ni] = 2.5*un[i+1*ni] - 2*un[i+2*ni] + 0.5*un[i+3*ni];
			un[i+(nj )*ni] = 2.5*un[i+(nj-1)*ni] - 2*un[i+(nj-2)*ni] + 0.5*un[i+(nj-3)*ni];
			vn[i+ni] = 2.5*vn[i+1*ni] - 2*vn[i+2*ni] + 0.5*vn[i+3*ni];
			vn[i+(nj)*ni] = 2.5*vn[i+(nj-1)*ni] - 2*vn[i+(nj-2)*ni] + 0.5*vn[i+(nj-3)*ni];
			hn[i+ni] = 2.5*hn[i+1*ni] - 2*hn[i+2*ni] + 0.5*hn[i+3*ni];
			hn[i+(nj)*ni] = 2.5*hn[i+(nj-1)*ni] - 2*hn[i+(nj-2)*ni] + 0.5*hn[i+(nj-3)*ni];
		}*/
	}
 __syncthreads();
  
}

int cuboundary(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
 	    //printf("called prop\n"); 
    // hipDeviceSynchronize();
    boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wd, *d_wmod);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
	    //printf("called update\n"); 
    hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

