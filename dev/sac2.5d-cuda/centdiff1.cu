#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd1.cuh"
__device__ __host__
real transportflux (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
     //flux= wd[fencode_cd1(p,ix,iy,vel1)]*w[fencode_cd1(p,ix,iy,field)];
     flux= w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,field)]/w[fencode_cd1(p,ix,iy,rho)];
     break;
     case 1:
     //flux= wd[fencode_cd1(p,ix,iy,vel2)]*w[fencode_cd1(p,ix,iy,field)];
     flux= w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,field)]/w[fencode_cd1(p,ix,iy,rho)];
     break;
    /* case 2:
     flux= wd[fencode_cd1(p,ix,iy,vel3)]*w[fencode_cd1(p,ix,iy,field)];
     break;*/
   }
  return flux;


  //return ( ddc1-ddc2);
}






__device__ __host__
real fluxmom1 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
    switch(direction)
  {
     case 0:
        #ifdef USE_SAC
     		flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b1)]-w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b1b)]-w[fencode_cd1(p,ix,iy,field+7)]*w[fencode_cd1(p,ix,iy,b1)];
        #endif
        #ifdef USE_VAC
                flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b1)];
         #endif

     break;
     case 1:
        #ifdef USE_SAC
                flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b2)]-w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b2b)]-w[fencode_cd1(p,ix,iy,field+7)]*w[fencode_cd1(p,ix,iy,b2)];
         #endif
        #ifdef USE_VAC
              flux= -w[fencode_cd1(p,ix,iy,field+3)]*w[fencode_cd1(p,ix,iy,b2)];
         #endif
     break;

   }




  return flux;


  //return ( ddc1-ddc2);
}










__device__ __host__
int computefluxrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int direction) {

  int field;
  int status=0;

  // for(direction=0;direction<2;direction++)
         #ifdef USE_SAC
	      wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,rho,direction)+(w[fencode_cd1(p,ix,iy,rhob)]*w[fencode_cd1(p,ix,iy,mom1+direction)])/(w[fencode_cd1(p,ix,iy,rhob)]+w[fencode_cd1(p,ix,iy,rho)]);
         #else
             wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,rho,direction);
         #endif
  
  return ( status);
}

__device__ __host__
int computefluxmom (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field,int direction) {

 
  int status=0;
  //for(direction=0;direction<2;direction++)
  //{
    switch(field)
  {
     case mom1:
        #ifdef ADIABHYDRO
     		wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #endif
        #ifdef USE_VAC
    		wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
 
        #endif
        #ifdef USE_SAC
    		wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
               if(direction==0)
                  wd[fencode_cd1(p,ix,iy,f1+direction)]+=wd[fencode_cd1(p,ix,iy,ptb)];
 
        #endif
               if(direction==0)
               {
                  wd[fencode_cd1(p,ix,iy,pressuret)]=((p->gamma)-1.0)*w[fencode_cd1(p,ix,iy,energy)]+(1.0-0.5*(p->gamma))*(w[fencode_cd1(p,ix,iy,b1)]*w[fencode_cd1(p,ix,iy,b1)]+w[fencode_cd1(p,ix,iy,b2)]*w[fencode_cd1(p,ix,iy,b2)])+0.5*(1.0-(p->gamma))*(w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,mom1)]+w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,mom2)])/w[fencode_cd1(p,ix,iy,rho)];
                  wd[fencode_cd1(p,ix,iy,f1+direction)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
             //     wd[fencode_cd1(p,ix,iy,f1+direction)]+=((p->gamma)-1.0)*w[fencode_cd1(p,ix,iy,energy)]+(1.0-0.5*(p->gamma))*(w[fencode_cd1(p,ix,iy,b1)]*w[fencode_cd1(p,ix,iy,b1)]+w[fencode_cd1(p,ix,iy,b2)]*w[fencode_cd1(p,ix,iy,b2)])+0.5*(1.0-(p->gamma))*(w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,mom1)]+w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,mom2)])/w[fencode_cd1(p,ix,iy,rho)];
               }
 
     break;
     case mom2:
        #ifdef ADIABHYDRO
     		wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,field,direction);
        #endif
        #ifdef USE_VAC
    		wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
 
        #endif
        #ifdef USE_SAC
    		wd[fencode_cd1(p,ix,iy,f1+direction)]= transportflux(dw,wd,w,p,ix,iy,field,direction)+fluxmom1(dw,wd,w,p,ix,iy,field,direction);
               if(direction==1)
                  wd[fencode_cd1(p,ix,iy,f1+direction)]+=wd[fencode_cd1(p,ix,iy,ptb)];
 
        #endif
               if(direction==1)
               {
                  wd[fencode_cd1(p,ix,iy,pressuret)]=((p->gamma)-1.0)*w[fencode_cd1(p,ix,iy,energy)]+(1.0-0.5*(p->gamma))*(w[fencode_cd1(p,ix,iy,b1)]*w[fencode_cd1(p,ix,iy,b1)]+w[fencode_cd1(p,ix,iy,b2)]*w[fencode_cd1(p,ix,iy,b2)])+0.5*(1.0-(p->gamma))*(w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,mom1)]+w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,mom2)])/w[fencode_cd1(p,ix,iy,rho)];
                  wd[fencode_cd1(p,ix,iy,f1+direction)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
             //     wd[fencode_cd1(p,ix,iy,f1+direction)]+=((p->gamma)-1.0)*w[fencode_cd1(p,ix,iy,energy)]+(1.0-0.5*(p->gamma))*(w[fencode_cd1(p,ix,iy,b1)]*w[fencode_cd1(p,ix,iy,b1)]+w[fencode_cd1(p,ix,iy,b2)]*w[fencode_cd1(p,ix,iy,b2)])+0.5*(1.0-(p->gamma))*(w[fencode_cd1(p,ix,iy,mom1)]*w[fencode_cd1(p,ix,iy,mom1)]+w[fencode_cd1(p,ix,iy,mom2)]*w[fencode_cd1(p,ix,iy,mom2)])/w[fencode_cd1(p,ix,iy,rho)];
               }
 
     break;
 
  // }
}
        
  return ( status);
}

__device__ __host__
int divflux1(real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  dw[fencode_cd1(p,ix,iy,field)]= grad_cd1(wd,p,ix,iy,f1+dir,dir);//+grad_cd1(wd,p,ix,iy,f2,1);  
 /*switch(field)
  {
     case mom1:
       dw[fencode_cd1(p,ix,iy,field)]+= grad_cd1(wd,p,ix,iy,pressuret,0);
      break;

    case mom2:
      dw[fencode_cd1(p,ix,iy,field)]+= grad_cd1(wd,p,ix,iy,pressuret,1);
      break;


  }    */
 // dw[fencode_cd1(p,ix,iy,field)]= gradd0_cd1(wd,p,ix,iy,f1,0)+gradd1_cd1(wd,p,ix,iy,f2,1);    
  return ( status);
}





//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case rho:
      computefluxrho(dw,wd,w,p,ix,iy,dir);
     break;
     case mom1:
      computefluxmom(dw,wd,w,p,ix,iy,field,dir);
      //wd[fencode_cd1(p,ix,iy,f1)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
     break;
     case mom2:
       computefluxmom(dw,wd,w,p,ix,iy,field,dir);
       //wd[fencode_cd1(p,ix,iy,f2)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
     break;
     /*case mom3:
      computefluxmom(dw,wd,w,p,ix,iy,field);
      //wd[fencode_cd1(p,ix,iy,f3)]+=wd[fencode_cd1(p,ix,iy,pressuret)];
     break;*/
  }
  //return ( status);
}



__global__ void centdiff1_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);

             //  for(int f=rho; f<=mom3; f++)
             //  {
			if(i<(ni) && j<(nj))
                        {
                            dwn1[fencode_cd1(p,i,j,f)]=0.0;
                  	    for(fid=0;fid<2;fid++)
                               wd[fencode_cd1(p,i,j,f1+fid)]=0.0;
                        }
                        __syncthreads();

			//if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                        if(i<(ni) && j<(nj))
                        {
                            computeflux(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir); 
                        }
              //  }
                        //might need to set boundaries correctly 
                        __syncthreads();

          if( i<(ni) && j<(nj))
             for(fid=0;fid<2;fid++)
                  //bc_cont_cd1(dwn1,p,i,j,f1+fid);
                  bc_periodic1_cd1(wd,p,i,j,f1+fid);
                __syncthreads();

          if( i<(ni) && j<(nj))
             for(fid=0;fid<2;fid++)
                  //bc_cont_cd1(dwn1,p,i,j,f1+fid);
                  bc_periodic2_cd1(wd,p,i,j,f1+fid);
                __syncthreads();

			//if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                        //        divflux1(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f);
            //  for(int f=rho; f<=mom3; f++)
             //  {
			 if(i>1 && j >1 && i<(ni-2) && j<(nj-2))

                               divflux1(dwn1,wd,wmod+order*NVAR*(p->n[0])*(p->n[1]),p,i,j,f,dir);  

               // }
     __syncthreads();

         // if( i<(ni) && j<(nj))
          //        bc_cont_cd1(dwn1,p,i,j,f);
            //    __syncthreads();



             // for(int f=rho; f<=mom3; f++)
              // {
			 if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode_cd1(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_cd1(p,i,j,f)+(ordero*NVAR*(p->n[0])*(p->n[1]))]-dt*dwn1[fencode_cd1(p,i,j,f)]; 
//wmod[fencode_cd1(p,i,j,f)+ordero*NVAR*(p->n[0])*(p->n[1])]=dwn1[fencode_cd1(p,i,j,f2)];
                              //dwn1[fencode_cd1(p,i,j,f)]=0;
                         }
              //  }	

  __syncthreads();


         //if( i<(ni) && j<(nj))
         //         bc_cont_cd1(wmod+ordero*NVAR*(p->n[0])*(p->n[1]),p,i,j,f);
         //       __syncthreads();


}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff1(struct params **p, real **w, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt, int field, int dir)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     centdiff1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


