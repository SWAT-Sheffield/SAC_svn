#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_cd1.cuh"
#include "dervfields_cd1.cuh"
#include "usersource_cd1.cuh"

__device__ __host__
int divflux1(real *dw, real *wd, real *w, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;

dw[fencode3_cd1(p,ii,field)]= grad3d_cd1(wd,p,ii,flux,dir); 
//dw[fencode3_cd1(p,ii,field)]=0.0;
 switch(field)
  {

     case mom1:
       dw[fencode3_cd1(p,ii,field)]+= (p->g[dir])*w[fencode3_cd1(p,ii,rho)];
      break;
    case mom2:
      dw[fencode3_cd1(p,ii,field)]+= (p->g[dir])*w[fencode3_cd1(p,ii,rho)];
      break;
#if defined USE_SAC_3D
    case mom3:
      dw[fencode3_cd1(p,ii,field)]+= (p->g[dir])*w[fencode3_cd1(p,ii,rho)];
      break;
#endif
    case rho:
     ;// dw[fencode3_cd1(p,ii,field)]+= ix/800;
      break;

  }    
 // dw[fencode3_cd1(p,ii,field)]= gradd0_cd1(wd,p,ii,f1,0)+gradd1_cd1(wd,p,ii,f2,1);    
  return ( status);
}






__device__ __host__
real transportflux (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;

   //transport flux
   //this will work without the switch as follows
        #if defined USE_SAC || defined USE_SAC_3D
     //flux= w[fencode3_cd1(p,ii,mom1+direction)]*w[fencode3_cd1(p,ii,field)]/(w[fencode3_cd1(p,ii,rho)]+w[fencode3_cd1(p,ii,rhob)]);
flux= wd[fencode3_cd1(p,ii,vel1+direction)]*w[fencode3_cd1(p,ii,field)];
        #else
     //flux= w[fencode3_cd1(p,ii,mom1+direction)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];
flux= wd[fencode3_cd1(p,ii,vel1+direction)]*w[fencode3_cd1(p,ii,field)];
        #endif


  /*  switch(direction)
  {
     case 0:
     //flux= wd[fencode3_cd1(p,ii,vel1)]*w[fencode3_cd1(p,ii,field)];
        #if defined USE_SAC || defined USE_SAC_3D
     flux= w[fencode3_cd1(p,ii,mom1)]*w[fencode3_cd1(p,ii,field)]/(w[fencode3_cd1(p,ii,rho)]+w[fencode3_cd1(p,ii,rhob)]);
    // flux= w[fencode3_cd1(p,ii,mom1)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];

        #else
     flux= w[fencode3_cd1(p,ii,mom1)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];

        #endif
     break;
     case 1:
        #if defined USE_SAC || defined USE_SAC_3D
     flux= w[fencode3_cd1(p,ii,mom2)]*w[fencode3_cd1(p,ii,field)]/(w[fencode3_cd1(p,ii,rho)]+w[fencode3_cd1(p,ii,rhob)]);
     //flux= w[fencode3_cd1(p,ii,mom2)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];

        #else
     //flux= wd[fencode3_cd1(p,ii,vel2)]*w[fencode3_cd1(p,ii,field)];
     flux= w[fencode3_cd1(p,ii,mom2)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];

        #endif
     break;
     case 2:
        #ifdef USE_SAC
     flux= w[fencode3_cd1(p,ii,mom2)]*w[fencode3_cd1(p,ii,field)]/(w[fencode3_cd1(p,ii,rho)]+w[fencode3_cd1(p,ii,rhob)]);
     //flux= w[fencode3_cd1(p,ii,mom2)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];

        #else
     //flux= wd[fencode3_cd1(p,ii,vel2)]*w[fencode3_cd1(p,ii,field)];
     flux= w[fencode3_cd1(p,ii,mom2)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)];

        #endif
     break;
    /* case 2:
     flux= wd[fencode3_cd1(p,ii,vel3)]*w[fencode3_cd1(p,ii,field)];
     break;*/
   //}*/
  return flux;


  //return ( ddc1-ddc2);
}






__device__ __host__
real fluxmom1 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx=0,ddcy=0;

   real flux=0;


         #if defined USE_SAC || defined USE_SAC_3D
     		flux= -(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]-w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)];
        #endif




   //transport flux
  /*  switch(direction)
  {
     case 0:
         #if defined USE_SAC || defined USE_SAC_3D
     		flux= -w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b)]-w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1)]-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1)];
        #endif
     break;
     case 1:
         #if defined USE_SAC || defined USE_SAC_3D
                flux= -w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b2b)]-w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b2)]-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b2)];
         #endif
     break;
#ifdef USE_SAC_3D
     case 2:
         
                flux= -w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b3b)]-w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b3)]-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b3)];

     break;
#endif

   }*/




  return flux;


  //return ( ddc1-ddc2);
}










__device__ __host__
int computefluxrho (real *dw, real *wd, real *w, struct params *p,int *ii,int direction) {

  int field;
  int status=0;

  // for(direction=0;direction<2;direction++)
         #if defined USE_SAC || defined USE_SAC_3D
	      wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,rho,direction)+(w[fencode3_cd1(p,ii,rhob)]*w[fencode3_cd1(p,ii,mom1+direction)])/(w[fencode3_cd1(p,ii,rhob)]+w[fencode3_cd1(p,ii,rho)]);
         #else
             wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,rho,direction);
         #endif
  
  return ( status);
}


__device__ __host__
int computefluxmom3 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

#ifdef USE_SAC_3D


    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom1(dw,wd,w,p,ii,field,direction);
               //if(direction==1)
               //   wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,ptb)];
 
 
               if(direction==2)
               {
                //computept_cd1(w,wd,p,ii);
                //commented out to compare with vac
                 wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,pressuret)];


                //  wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,ptb)];


               }
 

#endif

  return ( status);
}



__device__ __host__
int computefluxmom2 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

        #ifdef ADIABHYDRO
     		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction);
        #endif
        #ifdef USE_SAC
    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom1(dw,wd,w,p,ii,field,direction);
               //if(direction==1)
               //   wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,ptb)];
 
        #endif
        #ifdef USE_SAC_3D
    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom1(dw,wd,w,p,ii,field,direction);
               //if(direction==1)
               //   wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,ptb)];
 
        #endif
               if(direction==1)
               {
                //computept_cd1(w,wd,p,ii);
                //commented out to compare with vac
                 wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,pressuret)];

        #ifdef USE_SAC

               //   wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,ptb)];
        #endif
        #ifdef USE_SAC_3D

               //   wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,ptb)];
        #endif

               }


  return ( status);
}



__device__ __host__
int computefluxmom1 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

        #ifdef ADIABHYDRO
     		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction);
        #endif
        #ifdef USE_SAC
    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom1(dw,wd,w,p,ii,field,direction);
               //if(direction==0)
               //   wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,ptb)];
 
        #endif
        #ifdef USE_SAC_3D
    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom1(dw,wd,w,p,ii,field,direction);
               //if(direction==0)
               //   wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,ptb)];
 
        #endif
               if(direction==0)
               {

                // computept_cd1(w,wd,p,ii);
                 //commented out to compare with vac 
                 wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,pressuret)];

        #ifdef USE_SAC

                //  wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,ptb)];
       #endif
        #ifdef USE_SAC_3D

                //  wd[fencode3_cd1(p,ii,flux)]+=wd[fencode3_cd1(p,ii,ptb)];
       #endif
               }


        
  return ( status);
}







//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case rho:
      //computevel3_cd1(w,wd,p,ii);
      //computept3_cd1(w,wd,p,ii);
      computefluxrho(dw,wd,w,p,ii,dir);
     break;
     case mom1:
      computefluxmom1(dw,wd,w,p,ii,field,dir);
      //wd[fencode3_cd1(p,ii,f1)]+=wd[fencode3_cd1(p,ii,pressuret)];
     break;
     case mom2:
       computefluxmom2(dw,wd,w,p,ii,field,dir);
       //wd[fencode3_cd1(p,ii,f2)]+=wd[fencode3_cd1(p,ii,pressuret)];
     break;
     #ifdef USE_SAC_3D
       case mom3:
        computefluxmom3(dw,wd,w,p,ii,field,dir);
        //wd[fencode3_cd1(p,ii,f3)]+=wd[fencode3_cd1(p,ii,pressuret)];
       break;
     #endif
  }
  //return ( status);
}



__global__ void centdiff1_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  

   fid=0;
   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd1(p,ii,f)]=0.0;
                  	    //for(fid=0;fid<2;fid++)
                               wd[fencode3_cd1(p,ii,flux)]=0.0;
                               //wmod[fencode_cd1(p,i,j,flux)+order*NVAR*(p->n[0])*(p->n[1])]=0.0;
                        }

   }
 __syncthreads();                       



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif


//1. 11/1/11 could swap cases below
                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 
                         break;
                         #endif
                        }
              //  }
                        //might need to set boundaries correctly
 
}
__syncthreads();                        



}










__global__ void centdiff1a_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp,kpg;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  

   fid=0;



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

			// if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
			     #ifdef USE_SAC
				   if(ii[0]>1 && ii[1] >1 && ii[0]<(ni-2) && ii[1]<(nj-2))
			     #endif
			     #ifdef USE_SAC_3D
				  if(ii[0]>1 && ii[1] >1 && ii[2] >1 && ii[0]<(ni-2) && ii[1]<(nj-2) && ii[2]<(nk-2))
			     #endif                        
                               divflux1(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir);  

}
 __syncthreads();

#if(defined(USE_USERSOURCE))
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
#endif
   #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
#if(defined(USE_USERSOURCE))
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
#endif
     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif


     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
       if(ii[0]<((p->n[0])-2) && ii[1]<((p->n[1])-2) && ii[2]<((p->n[2])-2)     && ii[0]>1    &&  ii[1]>1   && ii[2]>1   )
     #endif
     #if(defined(USE_SAC) && defined(USE_USERSOURCE))
       if(ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)
     #endif

                     #ifdef USE_USERSOURCE
                                addsourceterms1_cd2(dwn1,wd,wmod+ordero*NVAR*dimp,p,ii,f,dir); 


                      }
                    __syncthreads();
                     #endif



               // }
    

         // if( i<(ni) && j<(nj))
          //        bc_cont_cd1(dwn1,p,i,j,f);
            //    __syncthreads();



             // for(int f=rho; f<=mom3; f++)
              // {

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif


                        switch(dir)
                        {
                         case 0:

                         //if(i<(ni)  && j >1 &&  j<(nj-2))
			     #ifdef USE_SAC
				   if(ii[1]>1 && ii[1] <(nj-2) && ii[0]<(ni) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[1]>1 && ii[1] <(nj-2) && ii[0]<(ni) &&  ii[2]>1 && ii[2] <(nk-2) )
			     #endif                          
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)]; 
                         break;
                         case 1:
			     #ifdef USE_SAC
				   if(ii[0]>1 && ii[1] <(nj) && ii[0]<(ni-2) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[0]>1 && ii[1] <(nj) && ii[0]<(ni-2) &&  ii[2]>1 && ii[2] <(nk-2) )
			     #endif 
                         
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];
                         break;
                         case 2:

 			     #ifdef USE_SAC
				   if(ii[1]>1 && ii[0] <(ni) && ii[1]<(nj-2) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[0]>1 &&  ii[0]<(ni-2)  && ii[1]>1 &&  ii[1]<(nj-2) && ii[2] <(nk) )
			     #endif                         
                         //if(i>1 &&  i<(ni-2) && j<(nj))
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];
                         break;
                        }


              //  }
	
}
  __syncthreads();


}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int cucentdiff1(struct params **p, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt, int field, int dir)
{

 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

     centdiff1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     centdiff1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     hipDeviceSynchronize();
}


