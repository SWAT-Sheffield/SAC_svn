#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_dc2 (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int fencode_dc2 (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])));
}

__device__ __host__
real evalgrad_dc2(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_dc2;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx[0])))*(fi-fim1)-(1.0/(12.0*(p->dx[0])))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx[0])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[0])))*(fi-fim1)):((1.0/(12.0*(p->dx[0])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(fi-fim1)-(1.0/(12.0*(p->dx[1])))*(fip2-fim2);
     // return((2.0/(1.0*(p->dx[1])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[1])))*(fi-fim1)):((1.0/(12.0*(p->dx[1])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_dc2(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_dc2;

  if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx[0])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx[0])))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx[0])))*(wmod[fencode_dc2(p,i+1,j,field)]-wmod[fencode_dc2(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((NVAR*wmod[fencode_dc2(p,i+1,j,field)]-NVAR*wmod[fencode_dc2(p,i-1,j,field)]+wmod[fencode_dc2(p,i-2,j,field)]-wmod[fencode_dc2(p,i+2,j,field)])/6.0):wmod[fencode_dc2(p,i+1,j,field)]-wmod[fencode_dc2(p,i-1,j,field)])/(2.0*(p->dx[0]))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dx[1])))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dx[1])))*(wmod[fencode_dc2(p,i,j+1,field)]-wmod[fencode_dc2(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((NVAR*wmod[fencode_dc2(p,i,j+1,field)]-NVAR*wmod[fencode_dc2(p,i,j-1,field)]+wmod[fencode_dc2(p,i,j-2,field)]-wmod[fencode_dc2(p,i,j+2,field)])/6.0):wmod[fencode_dc2(p,i,j+1,field)]-wmod[fencode_dc2(p,i,j-1,field)])/(2.0*(p->dx[1]))    );
}
 return 0;
}



__device__ __host__
real ddotcurrentb (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  //real ddc=0;

  real fi, fim1, fip2=0, fim2=0;
  real ddc1,ddc2;
  real ddcx,ddcy;

  switch(direction)
  {
	case 0:
	       fi=w[fencode_dc2(p,ix+1,iy,mom1)]*w[fencode_dc2(p,ix+1,iy,b1)]/w[fencode_dc2(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc2(p,ix-1,iy,mom1)]*w[fencode_dc2(p,ix-1,iy,b1)]/w[fencode_dc2(p,ix-1,iy,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix+2,iy,mom1)]*w[fencode_dc2(p,ix+2,iy,b1)]/w[fencode_dc2(p,ix+2,iy,rho)];
	       fim2=w[fencode_dc2(p,ix-2,iy,mom1)]*w[fencode_dc2(p,ix-2,iy,b1)]/w[fencode_dc2(p,ix-2,iy,rho)];
               }
	       ddcx=evalgrad_dc2(fi,fim1,fip2,fim2,p,0);
	       fi=w[fencode_dc2(p,ix,iy+1,mom1)]*w[fencode_dc2(p,ix,iy+1,b2)]/w[fencode_dc2(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc2(p,ix,iy-1,mom1)]*w[fencode_dc2(p,ix,iy-1,b2)]/w[fencode_dc2(p,ix,iy-1,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix,iy+2,mom1)]*w[fencode_dc2(p,ix,iy+2,b2)]/w[fencode_dc2(p,ix,iy+2,rho)];
	       fim2=w[fencode_dc2(p,ix,iy-2,mom1)]*w[fencode_dc2(p,ix,iy-2,b2)]/w[fencode_dc2(p,ix,iy-2,rho)];
               }
	       ddcy=evalgrad_dc2(fi,fim1,fip2,fim2,p,1);
        break;
	case 1:
	       fi=w[fencode_dc2(p,ix+1,iy,mom2)]*w[fencode_dc2(p,ix+1,iy,b1)]/w[fencode_dc2(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc2(p,ix-1,iy,mom2)]*w[fencode_dc2(p,ix-1,iy,b1)]/w[fencode_dc2(p,ix-1,iy,rho)];

               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix+2,iy,mom2)]*w[fencode_dc2(p,ix+2,iy,b1)]/w[fencode_dc2(p,ix+2,iy,rho)];
	       fim2=w[fencode_dc2(p,ix-2,iy,mom2)]*w[fencode_dc2(p,ix-2,iy,b1)]/w[fencode_dc2(p,ix-2,iy,rho)];
               }
	       ddcx=evalgrad_dc2(fi,fim1,fip2,fim2,p,0);

	       fi=w[fencode_dc2(p,ix,iy+1,mom2)]*w[fencode_dc2(p,ix,iy+1,b2)]/w[fencode_dc2(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc2(p,ix,iy-1,mom2)]*w[fencode_dc2(p,ix,iy-1,b2)]/w[fencode_dc2(p,ix,iy-1,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix,iy+2,mom2)]*w[fencode_dc2(p,ix,iy+2,b2)]/w[fencode_dc2(p,ix,iy+2,rho)];
	       fim2=w[fencode_dc2(p,ix,iy-2,mom2)]*w[fencode_dc2(p,ix,iy-2,b2)]/w[fencode_dc2(p,ix,iy-2,rho)];
               }
	       ddcy=evalgrad_dc2(fi,fim1,fip2,fim2,p,1);
        break;
	case 2:
	       fi=w[fencode_dc2(p,ix+1,iy,mom3)]*w[fencode_dc2(p,ix+1,iy,b1)]/w[fencode_dc2(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc2(p,ix-1,iy,mom3)]*w[fencode_dc2(p,ix-1,iy,b1)]/w[fencode_dc2(p,ix-1,iy,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix+2,iy,mom3)]*w[fencode_dc2(p,ix+2,iy,b1)]/w[fencode_dc2(p,ix+2,iy,rho)];
	       fim2=w[fencode_dc2(p,ix-2,iy,mom3)]*w[fencode_dc2(p,ix-2,iy,b1)]/w[fencode_dc2(p,ix-2,iy,rho)];
               }
	       ddcx=evalgrad_dc2(fi,fim1,fip2,fim2,p,0);
	       fi=w[fencode_dc2(p,ix,iy+1,mom3)]*w[fencode_dc2(p,ix,iy+1,b2)]/w[fencode_dc2(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc2(p,ix,iy-1,mom3)]*w[fencode_dc2(p,ix,iy-1,b2)]/w[fencode_dc2(p,ix,iy-1,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix,iy+2,mom3)]*w[fencode_dc2(p,ix,iy+2,b2)]/w[fencode_dc2(p,ix,iy+2,rho)];
	       fim2=w[fencode_dc2(p,ix,iy-2,mom3)]*w[fencode_dc2(p,ix,iy-2,b2)]/w[fencode_dc2(p,ix,iy-2,rho)];
               }
	       ddcy=evalgrad_dc2(fi,fim1,fip2,fim2,p,1);

        break;
  }
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  switch(direction)
  {
	case 0:
	       fi=w[fencode_dc2(p,ix+1,iy,b1)]*w[fencode_dc2(p,ix+1,iy,mom1)]/w[fencode_dc2(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc2(p,ix-1,iy,b1)]*w[fencode_dc2(p,ix-1,iy,mom1)]/w[fencode_dc2(p,ix-1,iy,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix+2,iy,b1)]*w[fencode_dc2(p,ix+2,iy,mom1)]/w[fencode_dc2(p,ix+2,iy,rho)];
	       fim2=w[fencode_dc2(p,ix-2,iy,b1)]*w[fencode_dc2(p,ix-2,iy,mom1)]/w[fencode_dc2(p,ix-2,iy,rho)];
               }
	       ddcx=evalgrad_dc2(fi,fim1,fip2,fim2,p,0);
	       fi=w[fencode_dc2(p,ix,iy+1,b1)]*w[fencode_dc2(p,ix,iy+1,mom2)]/w[fencode_dc2(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc2(p,ix,iy-1,b1)]*w[fencode_dc2(p,ix,iy-1,mom2)]/w[fencode_dc2(p,ix,iy-1,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix,iy+2,b1)]*w[fencode_dc2(p,ix,iy+2,mom2)]/w[fencode_dc2(p,ix,iy+2,rho)];
	       fim2=w[fencode_dc2(p,ix,iy-2,b1)]*w[fencode_dc2(p,ix,iy-2,mom2)]/w[fencode_dc2(p,ix,iy-2,rho)];
               }
	       ddcy=evalgrad_dc2(fi,fim1,fip2,fim2,p,1);
        break;
	case 1:
	       fi=w[fencode_dc2(p,ix+1,iy,b2)]*w[fencode_dc2(p,ix+1,iy,mom1)]/w[fencode_dc2(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc2(p,ix-1,iy,b2)]*w[fencode_dc2(p,ix-1,iy,mom1)]/w[fencode_dc2(p,ix-1,iy,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix+2,iy,b2)]*w[fencode_dc2(p,ix+2,iy,mom1)]/w[fencode_dc2(p,ix+2,iy,rho)];
	       fim2=w[fencode_dc2(p,ix-2,iy,b2)]*w[fencode_dc2(p,ix-2,iy,mom1)]/w[fencode_dc2(p,ix-2,iy,rho)];
               }
	       ddcx=evalgrad_dc2(fi,fim1,fip2,fim2,p,0);
	       fi=w[fencode_dc2(p,ix,iy+1,b2)]*w[fencode_dc2(p,ix,iy+1,mom2)]/w[fencode_dc2(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc2(p,ix,iy-1,b2)]*w[fencode_dc2(p,ix,iy-1,mom2)]/w[fencode_dc2(p,ix,iy-1,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix,iy+2,b2)]*w[fencode_dc2(p,ix,iy+2,mom2)]/w[fencode_dc2(p,ix,iy+2,rho)];
	       fim2=w[fencode_dc2(p,ix,iy-2,b2)]*w[fencode_dc2(p,ix,iy-2,mom2)]/w[fencode_dc2(p,ix,iy-2,rho)];
               }
	       ddcy=evalgrad_dc2(fi,fim1,fip2,fim2,p,1);
        break;
	case 2:
	       fi=w[fencode_dc2(p,ix+1,iy,b3)]*w[fencode_dc2(p,ix+1,iy,mom1)]/w[fencode_dc2(p,ix+1,iy,rho)];
	       fim1=w[fencode_dc2(p,ix-1,iy,b3)]*w[fencode_dc2(p,ix-1,iy,mom1)]/w[fencode_dc2(p,ix-1,iy,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix+2,iy,b3)]*w[fencode_dc2(p,ix+2,iy,mom1)]/w[fencode_dc2(p,ix+2,iy,rho)];
	       fim2=w[fencode_dc2(p,ix-2,iy,b3)]*w[fencode_dc2(p,ix-2,iy,mom1)]/w[fencode_dc2(p,ix-2,iy,rho)];
               }
	       ddcx=evalgrad_dc2(fi,fim1,fip2,fim2,p,0);
	       fi=w[fencode_dc2(p,ix,iy+1,b3)]*w[fencode_dc2(p,ix,iy+1,mom2)]/w[fencode_dc2(p,ix,iy+1,rho)];
	       fim1=w[fencode_dc2(p,ix,iy-1,b3)]*w[fencode_dc2(p,ix,iy-1,mom2)]/w[fencode_dc2(p,ix,iy-1,rho)];
               if(p->sodifon)
               {
	       fip2=w[fencode_dc2(p,ix,iy+2,b3)]*w[fencode_dc2(p,ix,iy+2,mom2)]/w[fencode_dc2(p,ix,iy+2,rho)];
	       fim2=w[fencode_dc2(p,ix,iy-2,b3)]*w[fencode_dc2(p,ix,iy-2,mom2)]/w[fencode_dc2(p,ix,iy-2,rho)];
               }
	       ddcy=evalgrad_dc2(fi,fim1,fip2,fim2,p,1);
        break;
  }
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
  return(ddc1-ddc2);
  //return -ddc2;

}

__device__ __host__
real ddotcurrentenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 // real ddc=0;
  real dd1,dd2,dd3;
 
  real ddcx,ddcy;
  real fi, fim1,fip2=0, fim2=0;
  //real dpi, dpim1;//, dpip2=0, dpim2=0;


  //int field=energy;

  //fi=w[fencode_dc2(p,ix+1,iy,energy)]*w[fencode_dc2(p,ix+1,iy,mom1)]/w[fencode_dc2(p,ix,iy,rho)];
  //fim1=w[fencode_dc2(p,ix-1,iy,energy)]*w[fencode_dc2(p,ix-1,iy,mom1)]/w[fencode_dc2(p,ix-1,iy,rho)];
if(p->sodifon==1)
{
  fip2=w[fencode_dc2(p,ix+2,iy,energy)]*w[fencode_dc2(p,ix+2,iy,mom1)]/w[fencode_dc2(p,ix+2,iy,rho)];
  fim2=w[fencode_dc2(p,ix-2,iy,energy)]*w[fencode_dc2(p,ix-2,iy,mom1)]/w[fencode_dc2(p,ix-2,iy,rho)];
}
 // ddcx=evalgrad_dc2(fi,fim1,0,0,p,0);
  ddcx=evalgrad_dc2(w[fencode_dc2(p,ix+1,iy,energy)]*w[fencode_dc2(p,ix+1,iy,mom1)]/w[fencode_dc2(p,ix+1,iy,rho)],w[fencode_dc2(p,ix-1,iy,energy)]*w[fencode_dc2(p,ix-1,iy,mom1)]/w[fencode_dc2(p,ix-1,iy,rho)],fip2,fim2,p,0);

 // fi=w[fencode_dc2(p,ix,iy+1,energy)]*w[fencode_dc2(p,ix,iy+1,mom2)]/w[fencode_dc2(p,ix,iy+1,rho)];
 // fim1=w[fencode_dc2(p,ix,iy-1,energy)]*w[fencode_dc2(p,ix,iy-1,mom2)]/w[fencode_dc2(p,ix,iy-1,rho)];
if(p->sodifon==1)
{
  fip2=w[fencode_dc2(p,ix,iy+2,energy)]*w[fencode_dc2(p,ix,iy+2,mom2)]/w[fencode_dc2(p,ix,iy+2,rho)];
  fim2=w[fencode_dc2(p,ix,iy-2,energy)]*w[fencode_dc2(p,ix,iy-2,mom2)]/w[fencode_dc2(p,ix,iy-2,rho)];
}
  //ddcy=evalgrad_dc2(fi,fim1,0,0,p,1);
  ddcy=evalgrad_dc2(w[fencode_dc2(p,ix,iy+1,energy)]*w[fencode_dc2(p,ix,iy+1,mom2)]/w[fencode_dc2(p,ix,iy+1,rho)],w[fencode_dc2(p,ix,iy-1,energy)]*w[fencode_dc2(p,ix,iy-1,mom2)]/w[fencode_dc2(p,ix,iy-1,rho)],fip2,fim2,p,1);

  dd1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);


 // dpi=(w[fencode_dc2(p,ix+1,iy,b1)]*w[fencode_dc2(p,ix+1,iy,mom1)]+w[fencode_dc2(p,ix+1,iy,b2)]*w[fencode_dc2(p,ix+1,iy,mom2)]+w[fencode_dc2(p,ix+1,iy,b3)]*w[fencode_dc2(p,ix+1,iy,mom3)])/w[fencode_dc2(p,ix+1,iy,rho)];
 // dpim1=(w[fencode_dc2(p,ix-1,iy,b1)]*w[fencode_dc2(p,ix-1,iy,mom1)]+w[fencode_dc2(p,ix-1,iy,b2)]*w[fencode_dc2(p,ix-1,iy,mom2)]+w[fencode_dc2(p,ix-1,iy,b3)]*w[fencode_dc2(p,ix-1,iy,mom3)])/w[fencode_dc2(p,ix-1,iy,rho)];
  //dpip2=(w[fencode_dc2(p,ix+2,iy,b1)]*w[fencode_dc2(p,ix+2,iy,mom1)]+w[fencode_dc2(p,ix+2,iy,b2)]*w[fencode_dc2(p,ix+2,iy,mom2)]+w[fencode_dc2(p,ix+2,iy,b3)]*w[fencode_dc2(p,ix+2,iy,mom3)])/w[fencode_dc2(p,ix+2,iy,rho)];
 // dpim2=(w[fencode_dc2(p,ix-2,iy,b1)]*w[fencode_dc2(p,ix-2,iy,mom1)]+w[fencode_dc2(p,ix-2,iy,b2)]*w[fencode_dc2(p,ix-2,iy,mom2)]+w[fencode_dc2(p,ix-2,iy,b3)]*w[fencode_dc2(p,ix-2,iy,mom3)])/w[fencode_dc2(p,ix-2,iy,rho)];

 // fi=dpi*w[fencode_dc2(p,ix+1,iy,b1)];
 // fim1=dpim1*w[fencode_dc2(p,ix-1,iy,b1)];
  //fip2=dpip2*w[fencode_dc2(p,ix+2,iy,b1)];
 // fim2=dpim2*w[fencode_dc2(p,ix-2,iy,b1)];
 // ddcx=evalgrad_dc2(fi,fim1,0,0,p,0);
 //  ddcx=evalgrad_dc2(((w[fencode_dc2(p,ix+1,iy,b1)]*w[fencode_dc2(p,ix+1,iy,mom1)]+w[fencode_dc2(p,ix+1,iy,b2)]*w[fencode_dc2(p,ix+1,iy,mom2)]+w[fencode_dc2(p,ix+1,iy,b3)]*w[fencode_dc2(p,ix+1,iy,mom3)])/w[fencode_dc2(p,ix+1,iy,rho)])*w[fencode_dc2(p,ix+1,iy,b1)],((w[fencode_dc2(p,ix-1,iy,b1)]*w[fencode_dc2(p,ix-1,iy,mom1)]+w[fencode_dc2(p,ix-1,iy,b2)]*w[fencode_dc2(p,ix-1,iy,mom2)]+w[fencode_dc2(p,ix-1,iy,b3)]*w[fencode_dc2(p,ix-1,iy,mom3)])/w[fencode_dc2(p,ix-1,iy,rho)])*w[fencode_dc2(p,ix-1,iy,b1)],0,0,p,0);

if(p->sodifon==1)
{
  fip2=wd[fencode_dc2(p,ix+2,iy,bdotv)]*w[fencode_dc2(p,ix+2,iy,b1)];
  fim2=wd[fencode_dc2(p,ix-2,iy,bdotv)]*w[fencode_dc2(p,ix-2,iy,b1)];
}

  ddcx=evalgrad_dc2(wd[fencode_dc2(p,ix+1,iy,bdotv)]*w[fencode_dc2(p,ix+1,iy,b1)],wd[fencode_dc2(p,ix-1,iy,bdotv)]*w[fencode_dc2(p,ix-1,iy,b1)],fip2,fim2,p,0);

 // dpi=(w[fencode_dc2(p,ix,iy+1,b1)]*w[fencode_dc2(p,ix,iy+1,mom1)]+w[fencode_dc2(p,ix,iy+1,b2)]*w[fencode_dc2(p,ix,iy+1,mom2)]+w[fencode_dc2(p,ix,iy+1,b3)]*w[fencode_dc2(p,ix,iy+1,mom3)])/w[fencode_dc2(p,ix,iy+1,rho)];
 // dpim1=(w[fencode_dc2(p,ix,iy-1,b1)]*w[fencode_dc2(p,ix,iy-1,mom1)]+w[fencode_dc2(p,ix,iy-1,b2)]*w[fencode_dc2(p,ix,iy-1,mom2)]+w[fencode_dc2(p,ix,iy-1,b3)]*w[fencode_dc2(p,ix,iy-1,mom3)])/w[fencode_dc2(p,ix,iy-1,rho)];  
  //dpip2=(w[fencode_dc2(p,ix,iy+2,b1)]*w[fencode_dc2(p,ix,iy+2,mom1)]+w[fencode_dc2(p,ix,iy+2,b2)]*w[fencode_dc2(p,ix,iy+2,mom2)]+w[fencode_dc2(p,ix,iy+2,b3)]*w[fencode_dc2(p,ix,iy+2,mom3)])/w[fencode_dc2(p,ix,iy+2,rho)];
  //dpim2=(w[fencode_dc2(p,ix,iy-2,b1)]*w[fencode_dc2(p,ix,iy-2,mom1)]+w[fencode_dc2(p,ix,iy-2,b2)]*w[fencode_dc2(p,ix,iy-2,mom2)]+w[fencode_dc2(p,ix,iy-2,b3)]*w[fencode_dc2(p,ix,iy-2,mom3)])/w[fencode_dc2(p,ix,iy-2,rho)];

 // fi=dpi*w[fencode_dc2(p,ix,iy+1,b2)];
 // fim1=dpim1*w[fencode_dc2(p,ix,iy-1,b2)];
if(p->sodifon==1)
{
  fip2=wd[fencode_dc2(p,ix,iy+2,bdotv)]*w[fencode_dc2(p,ix,iy+2,b2)];
  fim2=wd[fencode_dc2(p,ix,iy-2,bdotv)]*w[fencode_dc2(p,ix,iy-2,b2)];
}

//fi=w[fencode_dc2(p,ix,iy+1,b2)];
//  fim1=w[fencode_dc2(p,ix,iy-1,b2)];
  ddcy=evalgrad_dc2(wd[fencode_dc2(p,ix,iy+1,bdotv)]*w[fencode_dc2(p,ix,iy+1,b2)],wd[fencode_dc2(p,ix,iy-1,bdotv)]*w[fencode_dc2(p,ix,iy-1,b2)],fip2,fim2,p,1);
//ddcx=0;
//ddcy=evalgrad_dc2(((w[fencode_dc2(p,ix,iy+1,b1)]*w[fencode_dc2(p,ix,iy+1,mom1)]+w[fencode_dc2(p,ix,iy+1,b2)]*w[fencode_dc2(p,ix,iy+1,mom2)]+w[fencode_dc2(p,ix,iy+1,b3)]*w[fencode_dc2(p,ix,iy+1,mom3)])/w[fencode_dc2(p,ix,iy+1,rho)])*w[fencode_dc2(p,ix,iy+1,b2)],((w[fencode_dc2(p,ix,iy-1,b1)]*w[fencode_dc2(p,ix,iy-1,mom1)]+w[fencode_dc2(p,ix,iy-1,b2)]*w[fencode_dc2(p,ix,iy-1,mom2)]+w[fencode_dc2(p,ix,iy-1,b3)]*w[fencode_dc2(p,ix,iy-1,mom3)])/w[fencode_dc2(p,ix,iy-1,rho)])*w[fencode_dc2(p,ix,iy-1,b2)],0,0,p,1);

  dd2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  ddcx=wd[fencode_dc2(p,ix,iy,pressuret)]*grad_dc2(w,p,ix,iy,mom1,0)/w[fencode_dc2(p,ix,iy,rho)];
  ddcy=wd[fencode_dc2(p,ix,iy,pressuret)]*grad_dc2(w,p,ix,iy,mom2,1)/w[fencode_dc2(p,ix,iy,rho)];


  dd3=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  
  return(dd1+dd2+dd3);
 //return dd1;
 // return ( ddc);
}


__device__ __host__
int derivcurrentb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode_dc2(p,ix,iy,field)]= -ddotcurrentb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivcurrentenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode_dc2(p,ix,iy,field)]= -ddotcurrentenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void derivcurrent2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case energy:
       derivcurrentenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
       derivcurrentb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      derivcurrentb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      derivcurrentb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}



__global__ void derivcurrent2_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
 // int index;
  //int ni=p->n[0];
  //int nj=p->n[1];
 // real dt=p->dt;
  //real dy=p->dx[1];
 // real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/(p->n[0]);
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*(p->n[0]));


  //if(i>(1+(p->sodifon==1)) && j >(1+(p->sodifon==1)) && i<((p->n[0])-1-(p->sodifon==1)) && j<((p->n[1])-1-(p->sodifon==1)))
if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
               /*for(int f=rho; f<=b3; f++)               
                  wmod[fencode_dc2(p,i,j,f)]=w[fencode_dc2(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);*/
          
               for(int f=energy; f<=b3; f++)
               {              
                  derivcurrent2(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f);
                 // dwn1[fencode_dc2(p,i,j,f)]=1.0;
                  //__syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc2(p,i,j,f)]=w[fencode_dc2(p,i,j,f)]+0.5*dt*dwn1[fencode_dc2(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc2(p,i,j,f)]=w[fencode_dc2(p,i,j,f)]+0.5*dt*dwn2[fencode_dc2(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc2(p,i,j,f)]=w[fencode_dc2(p,i,j,f)]+dt*dwn3[fencode_dc2(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_dc2(p,i,j,f)]=w[fencode_dc2(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_dc2(p,i,j,f)]+2.0*dwn2[fencode_dc2(p,i,j,f)]
                         +2.0*dwn3[fencode_dc2(p,i,j,f)]+dwn4[fencode_dc2(p,i,j,f)]);
               }*/
              //  __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_dc2(p,i,j,f)]=w[fencode_dc2(p,i,j,f)]+dt*dwn1[fencode_dc2(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_dc2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuderivcurrent2(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivcurrent2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


