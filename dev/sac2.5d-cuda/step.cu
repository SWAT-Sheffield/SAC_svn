#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
float evalgrad(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
{
 //float valgrad;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   return(1.0/(1.0*(p->dx)))*(fi-fim1);
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
      return(1.0/(1.0*(p->dy)))*(fi-fim1);
 }

 return -1;
}


__device__ __host__
float grad(float *wmod,struct params *p,int i,int j,int field,int dir)
{
 //float valgrad;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
return(1.0/(1.0*(p->dx)))*(wmod[fencode(p,i+1,j,field)]-wmod[fencode(p,i-1,j,field)]);
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
 return(1.0/(1.0*(p->dy)))*(wmod[fencode(p,i,j+1,field)]-wmod[fencode(p,i,j-1,field)]);

 }

 return -1;
}

__device__ __host__
void computej(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;

 // float dbzdy, dbydz;
 // float dbzdx, dbxdz;
 // float dbydx, dbxdy;

 // dbzdy=grad(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad(wmod,p,i,j,b2,0);
 // dbxdy=grad(wmod,p,i,j,b1,1);

  wd[fencode(p,i,j,0)]=(grad(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode(p,i,j,1)]=(grad(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode(p,i,j,2)]=(grad(wmod,p,i,j,b2,0)-grad(wmod,p,i,j,b1,1))/(p->mu);
 
  //return ( status);
}

__device__ __host__
void computebdotv(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
//  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);

wd[fencode(p,i,j,bdotv)]=(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,mom3)])/wmod[fencode(p,i,j,rho)];
 // return ( status);
}


__device__ __host__
void computepk(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode(p,i,j,4)]=  wd[fencode(p,i,j,3)]+0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]);
 // return ( status);
}
__device__ __host__
void computept(float *wmod,float *wd,struct params *p,int i,int j)
{
  //int status=0;
  //float momsq=wmod[fencode(p,i,j,mom1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,mom2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,mom3)]*wmod[fencode(p,i,j,mom3)];
  //float bsq=wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)];
  wd[fencode(p,i,j,3)]=((p->gamma)-1)*(wmod[fencode(p,i,j,energy)]- 0.5*(wmod[fencode(p,i,j,mom1)]*wmod[fencode(p,i,j,mom1)]+wmod[fencode(p,i,j,mom2)]*wmod[fencode(p,i,j,mom2)]+wmod[fencode(p,i,j,mom3)]*wmod[fencode(p,i,j,mom3)])/wmod[fencode(p,i,j,rho)]-0.5*(wmod[fencode(p,i,j,b1)]*wmod[fencode(p,i,j,b1)]+wmod[fencode(p,i,j,b2)]*wmod[fencode(p,i,j,b2)]+wmod[fencode(p,i,j,b3)]*wmod[fencode(p,i,j,b3)]) );
  //return ( status);
}

__device__ __host__
float sourcerho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float src=0;
 // int field=rho;
 
  return 0;
}

__device__ __host__
float sourcemom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float src=0;
  switch(direction)
  {
	case 0:
         return(w[fencode(p,ix,iy,rho)]*(p->g1))-grad(wd,p,ix,iy,pressuret,0);
	break;
	case 1:
         return(w[fencode(p,ix,iy,rho)]*(p->g2))-grad(wd,p,ix,iy,pressuret,1);
	break;
	case 2:
         return(w[fencode(p,ix,iy,rho)]*(p->g3))-grad(wd,p,ix,iy,pressuret,2);
	break;
  }
  return 0;
}

__device__ __host__
float sourceb (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float src=0;
  switch(direction)
  {
	case 0:
         return(p->eta)*grad(wd,p,ix,iy,current3,1);
	break;
	case 1:
         return -(p->eta)*grad(wd,p,ix,iy,current3,0);
	break;
	case 2:
         return (p->eta)*(grad(wd,p,ix,iy,current2,0)-grad(wd,p,ix,iy,current1,1));
	break;
  }
  return 0;
}

__device__ __host__
float sourceenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float src=0;
  float srcg,srcb;
  int field=energy;
  float ddcx,ddcy;
  float fi,fim1;//fip2,fim2;
      srcg=(p->g1)*w[fencode(p,ix,iy,mom1)]+(p->g2)*w[fencode(p,ix,iy,mom2)]+(p->g3)*w[fencode(p,ix,iy,mom3)];

       fi=(w[fencode(p,ix+1,iy,b2)]*wd[fencode(p,ix+1,iy,current3)]-w[fencode(p,ix+1,iy,b3)]*wd[fencode(p,ix+1,iy,current2)]);
       fim1=(w[fencode(p,ix-1,iy,b2)]*wd[fencode(p,ix-1,iy,current3)]-w[fencode(p,ix-1,iy,b3)]*wd[fencode(p,ix-1,iy,current2)]);
      // fip2=(w[fencode(p,ix+2,iy,b2)]*wd[fencode(p,ix+2,iy,current3)]-w[fencode(p,ix+2,iy,b3)]*wd[fencode(p,ix+2,iy,current2)]);
     //  fim2=(w[fencode(p,ix-2,iy,b2)]*wd[fencode(p,ix-2,iy,current3)]-w[fencode(p,ix-2,iy,b3)]*wd[fencode(p,ix-2,iy,current2)]);
      // ddcx=evalgrad(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad(fi,fim1,0,0,p,0);

       fi=(w[fencode(p,ix+1,iy,b3)]*wd[fencode(p,ix+1,iy,current1)]-w[fencode(p,ix+1,iy,b1)]*wd[fencode(p,ix+1,iy,current3)]);
       fim1=(w[fencode(p,ix,iy-1,b3)]*wd[fencode(p,ix,iy-1,current1)]-w[fencode(p,ix,iy-1,b1)]*wd[fencode(p,ix,iy-1,current3)]);
     //  fip2=(w[fencode(p,ix,iy+2,b3)]*wd[fencode(p,ix,iy+2,current1)]-w[fencode(p,ix,iy+2,b1)]*wd[fencode(p,ix,iy+2,current3)]);
     //  fim2=(w[fencode(p,ix,iy-2,b3)]*wd[fencode(p,ix,iy-2,current1)]-w[fencode(p,ix,iy-2,b1)]*wd[fencode(p,ix,iy-2,current3)]);
      // ddcx=evalgrad(fi,fim1,fip2,fim2,p,0);
      ddcy=evalgrad(fi,fim1,0,0,p,1);

      srcb=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

 // src=srcg+srcb;
  return ( srcg+srcb);
}


__device__ __host__
float ddotcurrentrho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float ddc=0;
//  int field=rho;

     // ddc= grad(w,p,ix,iy,mom1,0)+grad(w,p,ix,iy,mom2,1);
  return ( grad(w,p,ix,iy,mom1,0)+grad(w,p,ix,iy,mom2,1));
}

__device__ __host__
float ddotcurrentmom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  float ddc=0;
  float fi, fim1;
  //float  fip2=0, fim2=0;
  float ddc1,ddc2;
  float ddcx,ddcy;
   //     ddc= grad(w,p,ix,iy,mom1,0)+grad(w,p,ix,iy,mom2,1);
//evalgrad(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
  //fi=w(fencode(p,ix,iy,rho))
  //calculate momentum current

//w[fencode(p,ix,iy,rho)])=1;
//w[fencode(p,ix-1,iy,rho)])=1;
//w[fencode(p,ix+2,iy,rho)])=1;
//w[fencode(p,ix-2,iy,rho)])=1;
//w[fencode(p,ix,iy,rho)])=1;
//w[fencode(p,ix,iy-1,rho)])=1;
//w[fencode(p,ix,iy+2,rho)])=1;
//w[fencode(p,ix,iy-2,rho)])=1;

  switch(direction)
  {
    case 0:
       fi=(w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,mom1)];
       fim1=(w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,mom1)];
    //   fip2=(w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)])*w[fencode(p,ix+2,iy,mom1)];
     //  fim2=(w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)])*w[fencode(p,ix-2,iy,mom1)];
      // ddcx=evalgrad(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad(fi,fim1,0,0,p,0);
       //ddcx=fi-fim1;
       fi=(w[fencode(p,ix,iy+1,mom1)]/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,mom2)];
       fim1=(w[fencode(p,ix,iy-1,mom1)]/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,mom2)];
      // fip2=(w[fencode(p,ix,iy+2,mom1)]/w[fencode(p,ix,iy+2,rho)])*w[fencode(p,ix,iy+2,mom2)];
      // fim2=(w[fencode(p,ix,iy-2,mom1)]/w[fencode(p,ix,iy-2,rho)])*w[fencode(p,ix,iy-2,mom2)];
       //ddcy=fi;
       ddcy=evalgrad(fi,fim1,0,0,p,1);
       //ddcy=evalgrad(0,0,fip2,fim2,p,1);
    break;
    case 1:
       fi=(w[fencode(p,ix+1,iy,mom2)]/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,mom1)];
       fim1=(w[fencode(p,ix-1,iy,mom2)]/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,mom1)];
      // fip2=(w[fencode(p,ix+2,iy,mom2)]/w[fencode(p,ix+2,iy,rho)])*w[fencode(p,ix+2,iy,mom1)];
      // fim2=(w[fencode(p,ix-2,iy,mom2)]/w[fencode(p,ix-2,iy,rho)])*w[fencode(p,ix-2,iy,mom1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=(w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,mom2)];
       fim1=(w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,mom2)];
      // fip2=(w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)])*w[fencode(p,ix,iy+2,mom2)];
      // fim2=(w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)])*w[fencode(p,ix,iy-2,mom2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
    case 2:
       fi=(w[fencode(p,ix+1,iy,mom3)]/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,mom1)];
       fim1=(w[fencode(p,ix-1,iy,mom3)]/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,mom1)];
      // fip2=(w[fencode(p,ix+2,iy,mom3)]/w[fencode(p,ix+2,iy,rho)])*w[fencode(p,ix+2,iy,mom1)];
     //  fim2=(w[fencode(p,ix-2,iy,mom3)]/w[fencode(p,ix-2,iy,rho)])*w[fencode(p,ix-2,iy,mom1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=(w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,mom2)];
       fim1=(w[fencode(p,ix,iy-1,mom3)]/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,mom2)];
     //  fip2=(w[fencode(p,ix,iy+2,mom3)]/w[fencode(p,ix,iy+2,rho)])*w[fencode(p,ix,iy+2,mom2)];
     //  fim2=(w[fencode(p,ix,iy-2,mom3)]/w[fencode(p,ix,iy-2,rho)])*w[fencode(p,ix,iy-2,mom2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
  }
  
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
//fip2=0, fim2=0;
  //calculate bfield current
  switch(direction)
  {
    case 0:
       fi=w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,b1)];
       fim1=w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,b1)];
     // fip2=w[fencode(p,ix+2,iy,b1)]*w[fencode(p,ix+2,iy,b1)];
     //  fim2=w[fencode(p,ix-2,iy,b1)]*w[fencode(p,ix-2,iy,b1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,b2)];
       fim1=w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,b2)];
     //  fip2=w[fencode(p,ix,iy+2,b1)]*w[fencode(p,ix,iy+2,b2)];
      // fim2=w[fencode(p,ix,iy-2,b1)]*w[fencode(p,ix,iy-2,b2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
    case 1:
       fi=w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,b1)];
       fim1=w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,b1)];
     //  fip2=w[fencode(p,ix+2,iy,b2)]*w[fencode(p,ix+2,iy,b1)];
      // fim2=w[fencode(p,ix-2,iy,b2)]*w[fencode(p,ix-2,iy,b1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,b2)];
       fim1=w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,b2)];
      // fip2=w[fencode(p,ix,iy+2,b2)]*w[fencode(p,ix,iy+2,b2)];
      // fim2=w[fencode(p,ix,iy-2,b2)]*w[fencode(p,ix,iy-2,b2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
    case 2:
       fi=w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,b1)];
       fim1=w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,b1)];
      // fip2=w[fencode(p,ix+2,iy,b3)]*w[fencode(p,ix+2,iy,b1)];
      // fim2=w[fencode(p,ix-2,iy,b3)]*w[fencode(p,ix-2,iy,b1)];
       ddcx=evalgrad(fi,fim1,0,0,p,0);
       fi=w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,b2)];
       fim1=w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,b2)];
      // fip2=w[fencode(p,ix,iy+2,b3)]*w[fencode(p,ix,iy+2,b2)];
     //  fim2=w[fencode(p,ix,iy-2,b3)]*w[fencode(p,ix,iy-2,b2)];
       ddcy=evalgrad(fi,fim1,0,0,p,1);
    break;
  }
  //ddc2=ddcx+ddcy;
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  //ddc=ddc1-ddc2;

  return ( ddc1-ddc2);
}

__device__ __host__
float ddotcurrentb (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float ddc=0;

  float fi, fim1;// fip2=0, fim2=0;
  float ddc1,ddc2;
  float ddcx,ddcy;

  switch(direction)
  {
	case 0:
	       fi=w[fencode(p,ix+1,iy,mom1)]*w[fencode(p,ix+1,iy,b1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,mom1)]*w[fencode(p,ix-1,iy,b1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,mom1)]*w[fencode(p,ix+2,iy,b1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,mom1)]*w[fencode(p,ix-2,iy,b1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,mom1)]*w[fencode(p,ix,iy+1,b2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,mom1)]*w[fencode(p,ix,iy-1,b2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,mom1)]*w[fencode(p,ix,iy+2,b2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,mom1)]*w[fencode(p,ix,iy-2,b2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 1:
	       fi=w[fencode(p,ix+1,iy,mom2)]*w[fencode(p,ix+1,iy,b1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,mom2)]*w[fencode(p,ix-1,iy,b1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,mom2)]*w[fencode(p,ix+2,iy,b1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,mom2)]*w[fencode(p,ix-2,iy,b1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,mom2)]*w[fencode(p,ix,iy+1,b2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,mom2)]*w[fencode(p,ix,iy-1,b2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,mom2)]*w[fencode(p,ix,iy+2,b2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,mom2)]*w[fencode(p,ix,iy-2,b2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 2:
	       fi=w[fencode(p,ix+1,iy,mom3)]*w[fencode(p,ix+1,iy,b1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,mom3)]*w[fencode(p,ix-1,iy,b1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,mom3)]*w[fencode(p,ix+2,iy,b1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,mom3)]*w[fencode(p,ix-2,iy,b1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,mom3)]*w[fencode(p,ix,iy+1,b2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,mom3)]*w[fencode(p,ix,iy-1,b2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,mom3)]*w[fencode(p,ix,iy+2,b2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,mom3)]*w[fencode(p,ix,iy-2,b2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);

        break;
  }
  ddc1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  switch(direction)
  {
	case 0:
	       fi=w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,b1)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
	      // fim2=w[fencode(p,ix-2,iy,b1)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,b1)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,b1)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 1:
	       fi=w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,b2)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
	      // fim2=w[fencode(p,ix-2,iy,b2)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
	      // fip2=w[fencode(p,ix,iy+2,b2)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
	      // fim2=w[fencode(p,ix,iy-2,b2)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
	case 2:
	       fi=w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix+1,iy,rho)];
	       fim1=w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
	       //fip2=w[fencode(p,ix+2,iy,b3)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
	       //fim2=w[fencode(p,ix-2,iy,b3)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
	       ddcx=evalgrad(fi,fim1,0,0,p,0);
	       fi=w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
	       fim1=w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
	       //fip2=w[fencode(p,ix,iy+2,b3)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
	       //fim2=w[fencode(p,ix,iy-2,b3)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
	       ddcy=evalgrad(fi,fim1,0,0,p,1);
        break;
  }
  ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
  return(ddc1-ddc2);

}

__device__ __host__
float ddotcurrentenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float ddc=0;
  float dd1,dd2,dd3;
 
  float ddcx,ddcy;
  //float fi, fim1;//fip2=0, fim2=0;
  //float dpi, dpim1;//, dpip2=0, dpim2=0;


  //int field=energy;

  //fi=w[fencode(p,ix+1,iy,energy)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix,iy,rho)];
  //fim1=w[fencode(p,ix-1,iy,energy)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)];
  //fip2=w[fencode(p,ix+2,iy,energy)]*w[fencode(p,ix+2,iy,mom1)]/w[fencode(p,ix+2,iy,rho)];
 // fim2=w[fencode(p,ix-2,iy,energy)]*w[fencode(p,ix-2,iy,mom1)]/w[fencode(p,ix-2,iy,rho)];
 // ddcx=evalgrad(fi,fim1,0,0,p,0);
  ddcx=evalgrad(w[fencode(p,ix+1,iy,energy)]*w[fencode(p,ix+1,iy,mom1)]/w[fencode(p,ix,iy,rho)],w[fencode(p,ix-1,iy,energy)]*w[fencode(p,ix-1,iy,mom1)]/w[fencode(p,ix-1,iy,rho)],0,0,p,0);

 // fi=w[fencode(p,ix,iy+1,energy)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)];
 // fim1=w[fencode(p,ix,iy-1,energy)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)];
 // fip2=w[fencode(p,ix,iy+2,energy)]*w[fencode(p,ix,iy+2,mom2)]/w[fencode(p,ix,iy+2,rho)];
  //fim2=w[fencode(p,ix,iy-2,energy)]*w[fencode(p,ix,iy-2,mom2)]/w[fencode(p,ix,iy-2,rho)];
  //ddcy=evalgrad(fi,fim1,0,0,p,1);
  ddcy=evalgrad(w[fencode(p,ix,iy+1,energy)]*w[fencode(p,ix,iy+1,mom2)]/w[fencode(p,ix,iy+1,rho)],w[fencode(p,ix,iy-1,energy)]*w[fencode(p,ix,iy-1,mom2)]/w[fencode(p,ix,iy-1,rho)],0,0,p,1);

  dd1=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);


 // dpi=(w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,mom1)]+w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,mom2)]+w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,mom3)])/w[fencode(p,ix+1,iy,rho)];
 // dpim1=(w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,mom1)]+w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,mom2)]+w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,mom3)])/w[fencode(p,ix-1,iy,rho)];
  //dpip2=(w[fencode(p,ix+2,iy,b1)]*w[fencode(p,ix+2,iy,mom1)]+w[fencode(p,ix+2,iy,b2)]*w[fencode(p,ix+2,iy,mom2)]+w[fencode(p,ix+2,iy,b3)]*w[fencode(p,ix+2,iy,mom3)])/w[fencode(p,ix+2,iy,rho)];
 // dpim2=(w[fencode(p,ix-2,iy,b1)]*w[fencode(p,ix-2,iy,mom1)]+w[fencode(p,ix-2,iy,b2)]*w[fencode(p,ix-2,iy,mom2)]+w[fencode(p,ix-2,iy,b3)]*w[fencode(p,ix-2,iy,mom3)])/w[fencode(p,ix-2,iy,rho)];

 // fi=dpi*w[fencode(p,ix+1,iy,b1)];
 // fim1=dpim1*w[fencode(p,ix-1,iy,b1)];
  //fip2=dpip2*w[fencode(p,ix+2,iy,b1)];
 // fim2=dpim2*w[fencode(p,ix-2,iy,b1)];
 // ddcx=evalgrad(fi,fim1,0,0,p,0);
 //  ddcx=evalgrad(((w[fencode(p,ix+1,iy,b1)]*w[fencode(p,ix+1,iy,mom1)]+w[fencode(p,ix+1,iy,b2)]*w[fencode(p,ix+1,iy,mom2)]+w[fencode(p,ix+1,iy,b3)]*w[fencode(p,ix+1,iy,mom3)])/w[fencode(p,ix+1,iy,rho)])*w[fencode(p,ix+1,iy,b1)],((w[fencode(p,ix-1,iy,b1)]*w[fencode(p,ix-1,iy,mom1)]+w[fencode(p,ix-1,iy,b2)]*w[fencode(p,ix-1,iy,mom2)]+w[fencode(p,ix-1,iy,b3)]*w[fencode(p,ix-1,iy,mom3)])/w[fencode(p,ix-1,iy,rho)])*w[fencode(p,ix-1,iy,b1)],0,0,p,0);
  ddcx=evalgrad(wd[fencode(p,ix+1,iy,bdotv)]*w[fencode(p,ix+1,iy,b1)],wd[fencode(p,ix-1,iy,bdotv)]*w[fencode(p,ix-1,iy,b1)],0,0,p,1);

 // dpi=(w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,mom1)]+w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,mom2)]+w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,mom3)])/w[fencode(p,ix,iy+1,rho)];
 // dpim1=(w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,mom1)]+w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,mom2)]+w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,mom3)])/w[fencode(p,ix,iy-1,rho)];  
  //dpip2=(w[fencode(p,ix,iy+2,b1)]*w[fencode(p,ix,iy+2,mom1)]+w[fencode(p,ix,iy+2,b2)]*w[fencode(p,ix,iy+2,mom2)]+w[fencode(p,ix,iy+2,b3)]*w[fencode(p,ix,iy+2,mom3)])/w[fencode(p,ix,iy+2,rho)];
  //dpim2=(w[fencode(p,ix,iy-2,b1)]*w[fencode(p,ix,iy-2,mom1)]+w[fencode(p,ix,iy-2,b2)]*w[fencode(p,ix,iy-2,mom2)]+w[fencode(p,ix,iy-2,b3)]*w[fencode(p,ix,iy-2,mom3)])/w[fencode(p,ix,iy-2,rho)];

 // fi=dpi*w[fencode(p,ix,iy+1,b2)];
 // fim1=dpim1*w[fencode(p,ix,iy-1,b2)];
  //fip2=dpip2*w[fencode(p,ix,iy+2,b2)];
  //fim2=dpim2*w[fencode(p,ix,iy-2,b2)];

//fi=w[fencode(p,ix,iy+1,b2)];
//  fim1=w[fencode(p,ix,iy-1,b2)];
  ddcy=evalgrad(wd[fencode(p,ix,iy+1,bdotv)]*w[fencode(p,ix,iy+1,b2)],wd[fencode(p,ix,iy-1,bdotv)]*w[fencode(p,ix,iy-1,b2)],0,0,p,1);
//ddcx=0;
//ddcy=evalgrad(((w[fencode(p,ix,iy+1,b1)]*w[fencode(p,ix,iy+1,mom1)]+w[fencode(p,ix,iy+1,b2)]*w[fencode(p,ix,iy+1,mom2)]+w[fencode(p,ix,iy+1,b3)]*w[fencode(p,ix,iy+1,mom3)])/w[fencode(p,ix,iy+1,rho)])*w[fencode(p,ix,iy+1,b2)],((w[fencode(p,ix,iy-1,b1)]*w[fencode(p,ix,iy-1,mom1)]+w[fencode(p,ix,iy-1,b2)]*w[fencode(p,ix,iy-1,mom2)]+w[fencode(p,ix,iy-1,b3)]*w[fencode(p,ix,iy-1,mom3)])/w[fencode(p,ix,iy-1,rho)])*w[fencode(p,ix,iy-1,b2)],0,0,p,1);

  dd2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);



  ddcx=wd[fencode(p,ix,iy,pressuret)]*grad(w,p,ix,iy,mom1,0)/w[fencode(p,ix,iy,rho)];
  ddcy=wd[fencode(p,ix,iy,pressuret)]*grad(w,p,ix,iy,mom2,1)/w[fencode(p,ix,iy,rho)];


  dd3=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

  
  return(dd1+dd2+dd3);
 //return dd1;
 // return ( ddc);
}

__device__ __host__
int derivrho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode(p,ix,iy,field)]=sourcerho(dw,wd,w,p,ix,iy)-ddotcurrentrho(dw,wd,w,p,ix,iy);
     	//dw[fencode(p,ix,iy,field)]=w[fencode(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivmom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode(p,ix,iy,field)]=w[fencode(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode(p,ix,iy,field)]=sourcemom(dw,wd,w,p,ix,iy,field,direction)-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivb (float *dw, float *wd, float *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode(p,ix,iy,field)]=sourceb(dw,wd,w,p,ix,iy,field,direction)-ddotcurrentb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode(p,ix,iy,field)]=sourceenergy(dw,wd,w,p,ix,iy)-ddotcurrentenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void deriv (float *dw, float *wd, float *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivrho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      derivmom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
      derivmom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      derivmom(dw,wd,w,p,ix,iy,field,2);
     break;
     case energy:
       derivenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
      derivb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      derivb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      derivb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}



__global__ void init_parallel(struct params *p, float *w, float *wnew, float *b, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->ni;
  int nj=p->nj;

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

 int nli = 0.45*(p->ni-1)+1;
  int nui = 0.55*(p->ni-1)+1;
  int nlj = 0.45*(p->nj-1)+1;
  int nuj = 0.55*(p->nj-1)+1; 
  int i,j;
   
   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{
		b[i+j*(p->ni)]=0;

                 //Define b	
		if((i*(p->dx)) >20001)
		      b[j*(p->ni)+i]=0;
		else if((i*(p->dx)) <20000)
			//b[j*(p->ni)+i]=(5000/20000)*(20000-(i*(p->dx)));
                        b[j*(p->ni)+i]=0;
                        // b[j*(p->ni)+i]=5000*(1.0-(((float)i)/30.0));		



		//initialise the arrays here
               for(k=0;k<1;++k)
      		{
                    index=j*(p->ni)+i+k*(p->ni)*(p->nj);
                    //index=i+j*(p->ni)+(k*(p->nj)*(p->ni));
		    u[index]=0;
		    v[index]=0;
		    h[index]=5;
                    w[index+mom3*(p->ni)*(p->nj)]=0;
                    w[index+energy*(p->ni)*(p->nj)]=0;
                    w[index+b1*(p->ni)*(p->nj)]=0;
                    w[index+b2*(p->ni)*(p->nj)]=0;
                    w[index+b3*(p->ni)*(p->nj)]=0;

//float *wmod, 
//    float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)


      		}
		//h[iindex]=5000;
	
        __syncthreads();
        if(i>=nli && i<=nui && j>=nlj && j<=nuj)
	{
	   //j*(p->ni)+i;
           h[j*(p->ni)+i]=5.030;	
	}

       for(int f=0; f<=5; f++)
        { 
                  wd[fencode(p,i,j,f)]=0;
        }

        for(int f=rho; f<=b3; f++)
        {               
                  wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)];
                  dwn1[fencode(p,i,j,f)]=0;
                  //dwn2[fencode(p,i,j,f)]=0;
                 // dwn3[fencode(p,i,j,f)]=0;
                  //dwn4[fencode(p,i,j,f)]=0;
                 
        }

	 __syncthreads();

			}	
	 __syncthreads();
  
}



__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               for(int f=rho; f<=b3; f++)               
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode(p,i,j,f)]=1.0;
                  __syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn1[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+0.5*dt*dwn2[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn3[fencode(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode(p,i,j,f)]+2.0*dwn2[fencode(p,i,j,f)]
                         +2.0*dwn3[fencode(p,i,j,f)]+dwn4[fencode(p,i,j,f)]);
               }*/
                __syncthreads();
               for(int f=rho; f<=b3; f++)
                   wnew[fencode(p,i,j,f)]=w[fencode(p,i,j,f)]+dt*dwn1[fencode(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);


	}
 __syncthreads();
  
}

__global__ void boundary_parallel(struct params *p, float *b, float *w, float *wnew)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;

  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;

  float *u,  *v,  *h;
  float *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;

    j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{

		if(i==0 )
		{
			un[j*ni] = 2.5*un[1+j*ni] - 2*un[2+j*ni] + 0.5*un[3+j*ni];
			un[ni+j*ni] = 2.5*un[ni-1+j*ni] - 2*un[ni-2+ni*j] + 0.5*un[ni-3+j*ni];
			vn[j*ni] = 2.5*vn[1+j*ni] - 2*vn[2+j*ni] + 0.5*vn[3+j*ni];
		 	vn[ni+j*ni] = 2.5*vn[ni-1+j*ni] - 2*vn[ni-2+ni*j] + 0.5*vn[ni-3+j*ni];
		 	hn[j*ni] = 2.5*hn[1+j*ni] - 2*hn[2+j*ni] + 0.5*hn[3+j*ni];
			hn[ni+j*ni] = 2.5*hn[ni-1+j*ni] - 2*hn[ni-2+ni*j] + 0.5*hn[ni-3+j*ni];
		}

		if(j==0)
		{
			un[i+ni] = 2.5*un[i+1*ni] - 2*un[i+2*ni] + 0.5*un[i+3*ni];
			un[i+(nj )*ni] = 2.5*un[i+(nj-1)*ni] - 2*un[i+(nj-2)*ni] + 0.5*un[i+(nj-3)*ni];
			vn[i+ni] = 2.5*vn[i+1*ni] - 2*vn[i+2*ni] + 0.5*vn[i+3*ni];
			vn[i+(nj)*ni] = 2.5*vn[i+(nj-1)*ni] - 2*vn[i+(nj-2)*ni] + 0.5*vn[i+(nj-3)*ni];
			hn[i+ni] = 2.5*hn[i+1*ni] - 2*hn[i+2*ni] + 0.5*hn[i+3*ni];
			hn[i+(nj)*ni] = 2.5*hn[i+(nj-1)*ni] - 2*hn[i+(nj-2)*ni] + 0.5*hn[i+(nj-3)*ni];
		}
	}
 __syncthreads();
  
}

__global__ void update_parallel(struct params *p, float *b, float *w, float *wnew)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;


  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  float *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;
     j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  //if(i>2 && j >2 && i<((p->ni)-3) && j<((p->nj)-3))

  if(i<p->ni && j<p->nj)
	{
             for(int f=rho; f<=b3; f++)               
                  w[fencode(p,i,j,f)]=wnew[fencode(p,i,j,f)];
            // u[i+j*ni]=un[i+j*ni];
           // v[i+j*ni]=vn[i+j*ni];
	   // h[i+j*ni]=hn[i+j*ni];
	}
 __syncthreads();
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}



int cuinit(struct params **p, float **w, float **wnew,  float **b, struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{



/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
  float *adb;
  float *adw, *adwnew;
  struct params *adp;

  hipMalloc((void**)d_wmod, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_dwn1, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)d_wd, 6*((*p)->ni)* ((*p)->nj)*sizeof(float));

  hipMalloc((void**)&adw, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adwnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adb, 1*(((*p)->ni)* ((*p)->nj))*sizeof(float));
  hipMalloc((void**)&adp, sizeof(struct params));
  checkErrors("memory allocation");

printf("ni is %d\n",(*p)->nj);

    *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;


    hipMemcpy(*d_w, *w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_wnew, *wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_b, *b, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
    // init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b, *d_wmod, *d_dwn1,  *d_wd);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);




  return 0;



}


int cuprop(struct params **p, float **w, float **wnew, float **b,struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     prop_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
 hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

int cufinish(struct params **p, float **w, float **wnew, float **b, struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{
  

 hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  checkErrors("copy data from device");


  hipFree(*d_p);

  hipFree(*d_w);
  hipFree(*d_wnew);
  hipFree(*d_b);

  hipFree(*d_wmod);
  hipFree(*d_dwn1);
  hipFree(*d_wd);



}
