#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////

__device__ __host__
int encode (struct params *dp,int ix, int iy) {

  int kSizeX=(dp)->ni;
  int kSizeY=(dp)->nj;
  
  return ( iy * kSizeX + ix);
}



__global__ void init_parallel(struct params *p, float *w, float *wnew, float *b)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;

 int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int index,k;
int ni=p->ni;
  int nj=p->nj;

// Block index
    int bx = blockIdx.x;
   // int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
   // int ty = threadIdx.y;
    
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

 int nli = 0.45*(p->ni-1)+1;
  int nui = 0.55*(p->ni-1)+1;
  int nlj = 0.45*(p->nj-1)+1;
  int nuj = 0.55*(p->nj-1)+1; 
  int i,j;
   
   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{
		b[i+j*(p->ni)]=0;

                 //Define b	
		if((i*(p->dx)) >20001)
		      b[j*(p->ni)+i]=0;
		else if((i*(p->dx)) <20000)
			//b[j*(p->ni)+i]=(5000/20000)*(20000-(i*(p->dx)));
                        b[j*(p->ni)+i]=0;
                        // b[j*(p->ni)+i]=5000*(1.0-(((float)i)/30.0));		



		//initialise the arrays here
               for(k=0;k<1;++k)
      		{
                    index=j*(p->ni)+i+k*(p->ni)*(p->nj);
                    //index=i+j*(p->ni)+(k*(p->nj)*(p->ni));
		    u[index]=0;
		    v[index]=0;
		    h[index]=5000;
      		}
		//h[iindex]=5000;
	
        __syncthreads();
        if(i>=nli && i<=nui && j>=nlj && j<=nuj)
	{
	   //j*(p->ni)+i;
           h[j*(p->ni)+i]=5030;	
	}
	 __syncthreads();

			}	
	 __syncthreads();
  
}



__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;


  float *u,  *v,  *h;
  float *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>0 && j >0 && i<((p->ni)-1) && j<((p->nj)-1))
	{
		//update the arrays here
               un[encode(p,i,j)] = (  (u[encode(p,i+1,j)] + u[i-1+j*ni] + u[i+(j+1)*ni] + u[i+(j-1)*ni])/4)- 0.5*(dt/dx)*(   (u[(i+1)+ni*j]*u[(i+1)+ni*j]/2) - (u[i-1+j*ni]*u[i-1+j*ni]/2 )  )- 0.5*(dt/dy)*(   v[i+j*ni] *(u[i+(j+1)*ni] - u[i+(j-1)*ni])   ) - 0.5*g*(dt/dx)*(h[i+1+j*ni]-h[i-1+j*ni]);

vn[i+j*ni] = ((v[i+1+j*ni] + v[i-1+j*ni] + v[i+(j+1)*ni] + v[i+(j-1)*ni])/4)- 0.5*(dt/dy)*(   (v[i+ni*(j+1)]*v[(i)+ni*(j+1)])/2 - (v[i+(j-1)*ni]*v[i+(j-1)*ni])/2) - 0.5*(dt/dx)*  (u[i+j*ni])*(v[i+1+j*ni] - v[i-1+j*ni]) - 0.5*g*(dt/dy)*(h[i+(j+1)*ni]-h[i+(j-1)*ni]);

hn[i+j*ni] = ((h[i+1+j*ni] + h[i-1+j*ni] + h[i+(j+1)*ni] + h[i+(j-1)*ni])/4)
- 0.5*(dt/dx)*(u[i+j*ni])*((h[i+1+j*ni]-b[i+1+j*ni]) - (h[i-1+j*ni]-b[i-1+j*ni])) 
- 0.5*(dt/dy)*(v[i+j*ni])*((h[i+(j+1)*ni]-b[i+(j+1)*ni]) - (h[i+(j-1)*ni]-b[i+(j-1)*ni])) 
- 0.5*(dt/dx)*(h[i+j*ni]-b[i+j*ni])*(u[i+1+j*ni]- u[i-1+j*ni])
- 0.5*(dt/dy)*(h[i+j*ni]-b[i+j*ni])*(v[i+(j+1)*ni] - v[i+(j-1)*ni]);

	}
 __syncthreads();
  
}

__global__ void boundary_parallel(struct params *p, float *b, float *w, float *wnew)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;

  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;

  float *u,  *v,  *h;
  float *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;

    j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i<p->ni && j<p->nj)
	{

		if(i==0 )
		{
			un[j*ni] = 2.5*un[1+j*ni] - 2*un[2+j*ni] + 0.5*un[3+j*ni];
			un[ni+j*ni] = 2.5*un[ni-1+j*ni] - 2*un[ni-2+ni*j] + 0.5*un[ni-3+j*ni];
			vn[j*ni] = 2.5*vn[1+j*ni] - 2*vn[2+j*ni] + 0.5*vn[3+j*ni];
		 	vn[ni+j*ni] = 2.5*vn[ni-1+j*ni] - 2*vn[ni-2+ni*j] + 0.5*vn[ni-3+j*ni];
		 	hn[j*ni] = 2.5*hn[1+j*ni] - 2*hn[2+j*ni] + 0.5*hn[3+j*ni];
			hn[ni+j*ni] = 2.5*hn[ni-1+j*ni] - 2*hn[ni-2+ni*j] + 0.5*hn[ni-3+j*ni];
		}

		if(j==0)
		{
			un[i+ni] = 2.5*un[i+1*ni] - 2*un[i+2*ni] + 0.5*un[i+3*ni];
			un[i+(nj )*ni] = 2.5*un[i+(nj-1)*ni] - 2*un[i+(nj-2)*ni] + 0.5*un[i+(nj-3)*ni];
			vn[i+ni] = 2.5*vn[i+1*ni] - 2*vn[i+2*ni] + 0.5*vn[i+3*ni];
			vn[i+(nj)*ni] = 2.5*vn[i+(nj-1)*ni] - 2*vn[i+(nj-2)*ni] + 0.5*vn[i+(nj-3)*ni];
			hn[i+ni] = 2.5*hn[i+1*ni] - 2*hn[i+2*ni] + 0.5*hn[i+3*ni];
			hn[i+(nj)*ni] = 2.5*hn[i+(nj-1)*ni] - 2*hn[i+(nj-2)*ni] + 0.5*hn[i+(nj-3)*ni];
		}
	}
 __syncthreads();
  
}

__global__ void update_parallel(struct params *p, float *b, float *w, float *wnew)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;


  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
  float *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  h=w+(p->ni)*(p->nj)*rho;
  u=w+(p->ni)*(p->nj)*mom1;
  v=w+(p->ni)*(p->nj)*mom2;

  float *un,  *vn,  *hn;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  hn=wnew+(p->ni)*(p->nj)*rho;
  un=wnew+(p->ni)*(p->nj)*mom1;
  vn=wnew+(p->ni)*(p->nj)*mom2;
     j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);

  if(i<p->ni && j<p->nj)
	{
            u[i+j*ni]=un[i+j*ni];
            v[i+j*ni]=vn[i+j*ni];
	    h[i+j*ni]=hn[i+j*ni];
	}
 __syncthreads();
  
}
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void saxpy_parallel(int n, float alpha, float *x, float *y)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // except for special cases, the total number of threads in all blocks
  // adds up to more than the vector length n, so this conditional is
  // EXTREMELY important to avoid writing past the allocated memory for
  // the vector y.
  if (i<n)
    y[i] = alpha*x[i] + y[i];
}

/////////////////////////////////////
// kernel function (CPU)
/////////////////////////////////////
void saxpy_serial(int n, float alpha, float *x, float *y)
{
  int i;
  for (i=0; i<n; i++)
    y[i] = alpha*x[i] + y[i];
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int stepfunc()
{
 /////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
 // int deviceCount;
 // hipGetDeviceCount(&deviceCount);
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
 // }
//  if (selectedDevice >= deviceCount)
  //{
  //  fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
  //  return 1;
  //}
 // hipSetDevice(selectedDevice);
  //checkErrors("initialisations");
  

  
  /////////////////////////////////////
  // (2) allocate memory on host (main CPU memory) and device,
  //     h_ denotes data residing on the host, d_ on device
  /////////////////////////////////////
  float *h_x = (float*)malloc(N*sizeof(float));
  float *h_y = (float*)malloc(N*sizeof(float));
  float *d_x;
  hipMalloc((void**)&d_x, N*sizeof(float));
  float *d_y;
  hipMalloc((void**)&d_y, N*sizeof(float));
  checkErrors("memory allocation");



  /////////////////////////////////////
  // (3) initialise data on the CPU
  /////////////////////////////////////
  int i;
  for (i=0; i<N; i++)
  {
    h_x[i] = 1.0f + i;
    h_y[i] = (float)(N-i+1);
  }



  /////////////////////////////////////
  // (4) copy data to device
  /////////////////////////////////////
  hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);
  checkErrors("copy data to device");



  /////////////////////////////////////
  // (5) perform computation on host (to enable result comparison later)
  /////////////////////////////////////
  saxpy_serial(N, 2.0f, h_x, h_y);



  /////////////////////////////////////
  // (6) perform computation on device
  //     - we use numThreadsPerBlock threads per block
  //     - the total number of blocks is obtained by rounding the
  //       vector length N up to the next multiple of numThreadsPerBlock
  /////////////////////////////////////
  int numBlocks = (N+numThreadsPerBlock-1) / numThreadsPerBlock;
  saxpy_parallel<<<numBlocks, numThreadsPerBlock>>>(N, 2.0, d_x, d_y);
  checkErrors("compute on device");



  /////////////////////////////////////
  // (7) read back result from device into temp vector
  /////////////////////////////////////
  float *h_z = (float*)malloc(N*sizeof(float));
  hipMemcpy(h_z, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");

  
  /////////////////////////////////////
  // (8) perform result comparison
  /////////////////////////////////////
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_y[i]-h_z[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");



  /////////////////////////////////////
  // (9) clean up, free memory
  /////////////////////////////////////
  free(h_x);
  free(h_y);
  free(h_z);
  hipFree(d_x);
  hipFree(d_y);
  return 0;

}

int cuinit(struct params **p, float **w, float **wnew,  float **b, struct params **d_p, float **d_w, float **d_wnew, float **d_b)
{
/////////////////////////////////////
  // (1) initialisations:
  //     - perform basic sanity checks
  //     - set device
  /////////////////////////////////////
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
   
 // if (deviceCount == 0)
 // {
 //   fprintf(stderr, "Sorry, no CUDA device fount");
 //   return 1;
//  }
  if (selectedDevice >= deviceCount)
  {
    fprintf(stderr, "Choose device ID between 0 and %d\n", deviceCount-1);
    return 1;
  }
  //hipSetDevice(selectedDevice);
  printf("device count %d selected %d\n", deviceCount,selectedDevice);
  checkErrors("initialisations");
  
	// Build empty u, v, b matrices

  printf("in cuinit\n");
  float *adb;
  float *adw, *adwnew;
  struct params *adp;

  hipMalloc((void**)&adw, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adwnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float));
  hipMalloc((void**)&adb, 1*(((*p)->ni)* ((*p)->nj))*sizeof(float));
  hipMalloc((void**)&adp, sizeof(struct params));
  checkErrors("memory allocation");

printf("ni is %d\n",(*p)->nj);

    *d_b=adb;
    *d_p=adp;
    *d_w=adw;
    *d_wnew=adwnew;


    hipMemcpy(*d_w, *w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_wnew, *wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_b, *b, ((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
    
    dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;
   

    printf("calling initialiser\n");
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
    // init_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
     init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wnew, *d_b);
     hipDeviceSynchronize();
	    printf("called initialiser\n");
	hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);




  return 0;



}


int cuprop(struct params **p, float **w, float **wnew, float **b,struct params **d_p, float **d_w, float **d_wnew, float **d_b)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(16, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     prop_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
    hipDeviceSynchronize();
 hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}

int cufinish(struct params **p, float **w, float **wnew, float **b, struct params **d_p, float **d_w, float **d_wnew, float **d_b)
{
  

 hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  checkErrors("copy data from device");


  hipFree(*d_p);

  hipFree(*d_w);
  hipFree(*d_wnew);
  hipFree(*d_b);


}
