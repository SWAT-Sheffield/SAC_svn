#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdr1.cuh"
__global__ void hyperdifrhosource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;  
   
     #ifdef USE_SAC_3D
	  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1)+(p->dx[2])*(dim==2));
	#else
	  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));
	#endif
  
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  {
     



dwn1[fencode3_hdr1(p,ii,field)]=( wd[fencode3_hdr1(p,ii,hdnur)] * wtemp[fencode3_hdr1(p,ii,tmp1)] - wd[fencode3_hdr1(p,ii,hdnul)] *wtemp[fencode3_hdr1(p,ii,tmp2)]            )/rdx;

                              wmod[fencode3_hdr1(p,ii,field)+(ordero*NVAR*dimp)]=wmod[fencode3_hdr1(p,ii,field)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdr1(p,ii,field)]; 
  }
}
__syncthreads();




 
}



__global__ void hyperdifrhosource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;  

 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //init rhol and rhor
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    //for(int f=tmp1; f<=tmprhor; f++)	
    //    wtemp[fencode_hdr1(p,i,j,f)]=0.0;
    dwn1[fencode3_hdr1(p,ii,field)]=0.0;
    wtemp[fencode3_hdr1(p,ii,tmp1)]=0.0;
    wtemp[fencode3_hdr1(p,ii,tmp2)]=0.0;
    //wtemp[fencode_hdr1(p,i,j,tmp3)]=0.0;
   }
}
 __syncthreads();

     #ifdef USE_SAC_3D
	  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1)+(p->dx[2])*(dim==2));
	#else
	  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));
	#endif

 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif

  
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {
     

    wtemp[fencode3_hdr1(p,ii,tmp1)]=grad1r3_hdr1(wmod+shift,p,ii,rho,dim);
    wtemp[fencode3_hdr1(p,ii,tmp2)]=grad1l3_hdr1(wmod+shift,p,ii,rho,dim);
  }
}
__syncthreads();




 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdr1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifrhosource1(struct params **p, struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero,real **d_wtemp, int field, int dim, real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifrhosource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
     hipDeviceSynchronize();
    hyperdifrhosource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,dt);
     hipDeviceSynchronize();


}







