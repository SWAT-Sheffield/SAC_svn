#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_hdr1.cuh"
__global__ void hyperdifrhosource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

int shift=order*NVAR*(p->n[0])*(p->n[1]);
   
  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

  
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  //if(i>1 && j >1 && i<((p->n[0])-1) && j<((p->n[1])-1))
  //if(i>32 && j >32 && i<((p->n[0])-32) && j<((p->n[1])-32))
  if(i<((p->n[0])) && j<((p->n[1])))
  {
     

//dwn1[fencode_hdr1(p,i,j,field)]=( wd[fencode_hdr1(p,i,j,hdnur)] * grad1r_hdr1(wmod+shift,p,i,j,rho,dim) - wd[fencode_hdr1(p,i,j,hdnul)] *grad1l_hdr1(wmod+shift,p,i,j,rho,dim)             )/rdx;
//dwn1[fencode_hdr1(p,i,j,field)]=( wtemp[fencode_hdr1(p,i,j,hdnur)] * grad1r_hdr1(wmod+shift,p,i,j,rho,dim) - wtemp[fencode_hdr1(p,i,j,hdnul)] *grad1l_hdr1(wmod+shift,p,i,j,rho,dim)             );

dwn1[fencode_hdr1(p,i,j,field)]=( wd[fencode_hdr1(p,i,j,hdnur)] * wtemp[fencode_hdr1(p,i,j,tmp1)] - wd[fencode_hdr1(p,i,j,hdnul)] *wtemp[fencode_hdr1(p,i,j,tmp2)]            )/rdx;

                              wmod[fencode_hdr1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]=wmod[fencode_hdr1(p,i,j,field)+(ordero*NVAR*(p->n[0])*(p->n[1]))]+dt*dwn1[fencode_hdr1(p,i,j,field)]; 
  }
}
__syncthreads();




 
}



__global__ void hyperdifrhosource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii,ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));

   int shift=order*NVAR*(p->n[0])*(p->n[1]);
   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //init rhol and rhor
  if(i<((p->n[0])) && j<((p->n[1])))
  {
    //for(int f=tmp1; f<=tmprhor; f++)	
    //    wtemp[fencode_hdr1(p,i,j,f)]=0.0;
    dwn1[fencode_hdr1(p,i,j,field)]=0.0;
    wtemp[fencode_hdr1(p,i,j,tmp1)]=0.0;
    wtemp[fencode_hdr1(p,i,j,tmp2)]=0.0;
    //wtemp[fencode_hdr1(p,i,j,tmp3)]=0.0;
   }
}
 __syncthreads();


  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));

 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;

  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
  if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  //if(i>32 && j >32 && i<((p->n[0])-32) && j<((p->n[1])-32))
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
     

//dwn1[fencode_hdr1(p,i,j,field)]=( wd[fencode_hdr1(p,i,j,hdnur)] * grad1r_hdr1(wmod+shift,p,i,j,rho,dim) - wd[fencode_hdr1(p,i,j,hdnul)] *grad1l_hdr1(wmod+shift,p,i,j,rho,dim)             )/rdx;
//dwn1[fencode_hdr1(p,i,j,field)]=( wtemp[fencode_hdr1(p,i,j,hdnur)] * grad1r_hdr1(wmod+shift,p,i,j,rho,dim) - wtemp[fencode_hdr1(p,i,j,hdnul)] *grad1l_hdr1(wmod+shift,p,i,j,rho,dim)             );

    wtemp[fencode_hdr1(p,i,j,tmp1)]=grad1r_hdr1(wmod+shift,p,i,j,rho,dim);
    wtemp[fencode_hdr1(p,i,j,tmp2)]=grad1l_hdr1(wmod+shift,p,i,j,rho,dim);
  }
}
__syncthreads();




 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdr1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifrhosource1(struct params **p, struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero,real **d_wtemp, int field, int dim, real dt)
{

 dim3 dimBlock(dimblock, 1);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifrhosource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
     hipDeviceSynchronize();
    hyperdifrhosource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,dt);
     hipDeviceSynchronize();


}







