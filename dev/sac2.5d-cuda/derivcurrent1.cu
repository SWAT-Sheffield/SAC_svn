#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "gradops_dc1.cuh"


__device__ __host__
real ddotcurrentrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx,ddcy;
  real ddc=0;
//  int field=rho;
        #ifdef USE_SAC
		fi=(w[fencode_dc1(p,ix+1,iy,rhob)]*w[fencode_dc1(p,ix+1,iy,mom1)]/(w[fencode_dc1(p,ix+1,iy,rho)]+w[fencode_dc1(p,ix+1,iy,rhob)]  )  );
		fim1=(w[fencode_dc1(p,ix-1,iy,rhob)]*w[fencode_dc1(p,ix-1,iy,mom1)]/(w[fencode_dc1(p,ix-1,iy,rho)]+w[fencode_dc1(p,ix-1,iy,rhob)]  ));
		if(p->sodifon)
		{
		fip2=(w[fencode_dc1(p,ix+2,iy,rhob)]*w[fencode_dc1(p,ix+2,iy,mom1)]/(w[fencode_dc1(p,ix+2,iy,rho)]+w[fencode_dc1(p,ix,iy+1,rhob)]));
		fim2=(w[fencode_dc1(p,ix-2,iy,rhob)]*w[fencode_dc1(p,ix-2,iy,mom1)]/(w[fencode_dc1(p,ix-2,iy,rho)]+w[fencode_dc1(p,ix-2,iy,rhob)]));
		}
		ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);

		fi=(w[fencode_dc1(p,ix,iy+1,rhob)]*w[fencode_dc1(p,ix,iy+1,mom1)]/(w[fencode_dc1(p,ix,iy+1,rho)]+w[fencode_dc1(p,ix,iy+1,rhob)]));
		fim1=(w[fencode_dc1(p,ix,iy-1,rhob)]*w[fencode_dc1(p,ix,iy-1,mom1)]/(w[fencode_dc1(p,ix,iy-1,rho)]+w[fencode_dc1(p,ix,iy-1,rhob)]));
		if(p->sodifon)
		{
		fip2=(w[fencode_dc1(p,ix,iy+2,rhob)]*w[fencode_dc1(p,ix,iy+2,mom1)]/(w[fencode_dc1(p,ix,iy+2,rho)]+w[fencode_dc1(p,ix+2,iy,rhob)]));
		fim2=(w[fencode_dc1(p,ix,iy-2,rhob)]*w[fencode_dc1(p,ix,iy-2,mom1)]/(w[fencode_dc1(p,ix,iy-2,rho)]+w[fencode_dc1(p,ix-2,iy,rhob)]));
		}
		ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);

		ddc= ddcx+ddcy;
         #else
           ddc= grad_dc1(w,p,ix,iy,mom1,0)+grad_dc1(w,p,ix,iy,mom2,1);

   

         #endif

  return ( isnan(ddc)?0:ddc);
}

//compute momentum contributions for momentum current
__device__ __host__
real ddotcurrentmom1 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
 // real ddc1;
  real ddcx,ddcy;
   //     ddc= grad_dc1(w,p,ix,iy,mom1,0)+grad_dc1(w,p,ix,iy,mom2,1);
//evalgrad_dc1(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
  //fi=w(fencode_dc1(p,ix,iy,rho))
  //calculate momentum current

//w[fencode_dc1(p,ix,iy,rho)])=1;
//w[fencode_dc1(p,ix-1,iy,rho)])=1;
//w[fencode_dc1(p,ix+2,iy,rho)])=1;
//w[fencode_dc1(p,ix-2,iy,rho)])=1;
//w[fencode_dc1(p,ix,iy,rho)])=1;
//w[fencode_dc1(p,ix,iy-1,rho)])=1;
//w[fencode_dc1(p,ix,iy+2,rho)])=1;
//w[fencode_dc1(p,ix,iy-2,rho)])=1;

  switch(direction)
  {
    case 0:
       #ifdef USE_SAC
       fi=(w[fencode_dc1(p,ix+1,iy,mom1)]/(w[fencode_dc1(p,ix+1,iy,rho)]+w[fencode_dc1(p,ix+1,iy,rhob)]))*w[fencode_dc1(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc1(p,ix-1,iy,mom1)]/(w[fencode_dc1(p,ix-1,iy,rho)]+w[fencode_dc1(p,ix-1,iy,rhob)]))*w[fencode_dc1(p,ix-1,iy,mom1)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix+2,iy,mom1)]/(w[fencode_dc1(p,ix+2,iy,rho)]+w[fencode_dc1(p,ix+2,iy,rhob)]))*w[fencode_dc1(p,ix+2,iy,mom1)];
       fim2=(w[fencode_dc1(p,ix-2,iy,mom1)]/(w[fencode_dc1(p,ix-2,iy,rho)]+w[fencode_dc1(p,ix-2,iy,rhob)]))*w[fencode_dc1(p,ix-2,iy,mom1)];
       }

       #else
       fi=(w[fencode_dc1(p,ix+1,iy,mom1)]/w[fencode_dc1(p,ix+1,iy,rho)])*w[fencode_dc1(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc1(p,ix-1,iy,mom1)]/w[fencode_dc1(p,ix-1,iy,rho)])*w[fencode_dc1(p,ix-1,iy,mom1)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix+2,iy,mom1)]/w[fencode_dc1(p,ix+2,iy,rho)])*w[fencode_dc1(p,ix+2,iy,mom1)];
       fim2=(w[fencode_dc1(p,ix-2,iy,mom1)]/w[fencode_dc1(p,ix-2,iy,rho)])*w[fencode_dc1(p,ix-2,iy,mom1)];
       }
       #endif
       ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);
      //ddcx=evalgrad_dc1(fi,fim1,0,0,p,0);
       //ddcx=fi-fim1;

       #ifdef USE_SAC
       fi=(w[fencode_dc1(p,ix,iy+1,mom1)]/(w[fencode_dc1(p,ix,iy+1,rho)]+w[fencode_dc1(p,ix,iy+1,rhob)]))*w[fencode_dc1(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc1(p,ix,iy-1,mom1)]/(w[fencode_dc1(p,ix,iy-1,rho)]+w[fencode_dc1(p,ix,iy-1,rhob)]))*w[fencode_dc1(p,ix,iy-1,mom2)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix,iy+2,mom1)]/(w[fencode_dc1(p,ix,iy+2,rho)]+w[fencode_dc1(p,ix,iy+2,rhob)]))*w[fencode_dc1(p,ix,iy+2,mom2)];
       fim2=(w[fencode_dc1(p,ix,iy-2,mom1)]/(w[fencode_dc1(p,ix,iy-2,rho)]+w[fencode_dc1(p,ix,iy-2,rhob)]))*w[fencode_dc1(p,ix,iy-2,mom2)];
       }
       #else
       fi=(w[fencode_dc1(p,ix,iy+1,mom1)]/w[fencode_dc1(p,ix,iy+1,rho)])*w[fencode_dc1(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc1(p,ix,iy-1,mom1)]/w[fencode_dc1(p,ix,iy-1,rho)])*w[fencode_dc1(p,ix,iy-1,mom2)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix,iy+2,mom1)]/w[fencode_dc1(p,ix,iy+2,rho)])*w[fencode_dc1(p,ix,iy+2,mom2)];
       fim2=(w[fencode_dc1(p,ix,iy-2,mom1)]/w[fencode_dc1(p,ix,iy-2,rho)])*w[fencode_dc1(p,ix,iy-2,mom2)];
       }
       #endif
       //ddcy=fi;
       ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);
       //ddcy=evalgrad_dc1(0,0,fip2,fim2,p,1);
    break;
    case 1:
       #ifdef USE_SAC
       fi=(w[fencode_dc1(p,ix+1,iy,mom2)]/(w[fencode_dc1(p,ix+1,iy,rho)]+w[fencode_dc1(p,ix+1,iy,rhob)]))*w[fencode_dc1(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc1(p,ix-1,iy,mom2)]/(w[fencode_dc1(p,ix-1,iy,rho)]+w[fencode_dc1(p,ix-1,iy,rhob)]))*w[fencode_dc1(p,ix-1,iy,mom1)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix+2,iy,mom2)]/(w[fencode_dc1(p,ix+2,iy,rho)]+w[fencode_dc1(p,ix+2,iy,rhob)]))*w[fencode_dc1(p,ix+2,iy,mom1)];
       fim2=(w[fencode_dc1(p,ix-2,iy,mom2)]/(w[fencode_dc1(p,ix-2,iy,rho)]+w[fencode_dc1(p,ix-2,iy,rhob)]))*w[fencode_dc1(p,ix-2,iy,mom1)];
       }
       #else
       fi=(w[fencode_dc1(p,ix+1,iy,mom2)]/w[fencode_dc1(p,ix+1,iy,rho)])*w[fencode_dc1(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc1(p,ix-1,iy,mom2)]/w[fencode_dc1(p,ix-1,iy,rho)])*w[fencode_dc1(p,ix-1,iy,mom1)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix+2,iy,mom2)]/w[fencode_dc1(p,ix+2,iy,rho)])*w[fencode_dc1(p,ix+2,iy,mom1)];
       fim2=(w[fencode_dc1(p,ix-2,iy,mom2)]/w[fencode_dc1(p,ix-2,iy,rho)])*w[fencode_dc1(p,ix-2,iy,mom1)];
       }
       #endif
       ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);

       #ifdef USE_SAC
       fi=(w[fencode_dc1(p,ix,iy+1,mom2)]/(w[fencode_dc1(p,ix,iy+1,rho)]+w[fencode_dc1(p,ix,iy+1,rhob)]))*w[fencode_dc1(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc1(p,ix,iy-1,mom2)]/(w[fencode_dc1(p,ix,iy-1,rho)]+w[fencode_dc1(p,ix,iy-1,rhob)]))*w[fencode_dc1(p,ix,iy-1,mom2)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix,iy+2,mom2)]/(w[fencode_dc1(p,ix,iy+2,rho)]+w[fencode_dc1(p,ix,iy+2,rhob)]))*w[fencode_dc1(p,ix,iy+2,mom2)];
       fim2=(w[fencode_dc1(p,ix,iy-2,mom2)]/(w[fencode_dc1(p,ix,iy-2,rho)]+w[fencode_dc1(p,ix,iy-2,rhob)]))*w[fencode_dc1(p,ix,iy-2,mom2)];
       }
 
       #else
       fi=(w[fencode_dc1(p,ix,iy+1,mom2)]/w[fencode_dc1(p,ix,iy+1,rho)])*w[fencode_dc1(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc1(p,ix,iy-1,mom2)]/w[fencode_dc1(p,ix,iy-1,rho)])*w[fencode_dc1(p,ix,iy-1,mom2)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix,iy+2,mom2)]/w[fencode_dc1(p,ix,iy+2,rho)])*w[fencode_dc1(p,ix,iy+2,mom2)];
       fim2=(w[fencode_dc1(p,ix,iy-2,mom2)]/w[fencode_dc1(p,ix,iy-2,rho)])*w[fencode_dc1(p,ix,iy-2,mom2)];
       }
       #endif
       ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);
    break;
    case 2:
       #ifdef USE_SAC
       fi=(w[fencode_dc1(p,ix+1,iy,mom3)]/(w[fencode_dc1(p,ix+1,iy,rho)]+w[fencode_dc1(p,ix+1,iy,rhob)]))*w[fencode_dc1(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc1(p,ix-1,iy,mom3)]/(w[fencode_dc1(p,ix-1,iy,rho)]+w[fencode_dc1(p,ix-1,iy,rhob)]))*w[fencode_dc1(p,ix-1,iy,mom1)];
       if(p->sodifon)
       {
         fip2=(w[fencode_dc1(p,ix+2,iy,mom3)]/(w[fencode_dc1(p,ix+2,iy,rho)]+w[fencode_dc1(p,ix+2,iy,rhob)]))*w[fencode_dc1(p,ix+2,iy,mom1)];
         fim2=(w[fencode_dc1(p,ix-2,iy,mom3)]/(w[fencode_dc1(p,ix-2,iy,rho)]+w[fencode_dc1(p,ix-2,iy,rhob)]))*w[fencode_dc1(p,ix-2,iy,mom1)];
       }

       #else
       fi=(w[fencode_dc1(p,ix+1,iy,mom3)]/w[fencode_dc1(p,ix+1,iy,rho)])*w[fencode_dc1(p,ix+1,iy,mom1)];
       fim1=(w[fencode_dc1(p,ix-1,iy,mom3)]/w[fencode_dc1(p,ix-1,iy,rho)])*w[fencode_dc1(p,ix-1,iy,mom1)];
       if(p->sodifon)
       {
         fip2=(w[fencode_dc1(p,ix+2,iy,mom3)]/w[fencode_dc1(p,ix+2,iy,rho)])*w[fencode_dc1(p,ix+2,iy,mom1)];
         fim2=(w[fencode_dc1(p,ix-2,iy,mom3)]/w[fencode_dc1(p,ix-2,iy,rho)])*w[fencode_dc1(p,ix-2,iy,mom1)];
       }
       #endif
       ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);

       #ifdef USE_SAC
       fi=(w[fencode_dc1(p,ix,iy+1,mom3)]/(w[fencode_dc1(p,ix,iy+1,rho)]+w[fencode_dc1(p,ix,iy+1,rhob)]))*w[fencode_dc1(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc1(p,ix,iy-1,mom3)]/(w[fencode_dc1(p,ix,iy-1,rho)]+w[fencode_dc1(p,ix,iy-1,rhob)]))*w[fencode_dc1(p,ix,iy-1,mom2)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix,iy+2,mom3)]/(w[fencode_dc1(p,ix,iy+2,rho)]+w[fencode_dc1(p,ix,iy+2,rhob)]))*w[fencode_dc1(p,ix,iy+2,mom2)];
       fim2=(w[fencode_dc1(p,ix,iy-2,mom3)]/(w[fencode_dc1(p,ix,iy-2,rho)]+w[fencode_dc1(p,ix,iy-2,rhob)]))*w[fencode_dc1(p,ix,iy-2,mom2)];
       }

       #else
       fi=(w[fencode_dc1(p,ix,iy+1,mom3)]/w[fencode_dc1(p,ix,iy+1,rho)])*w[fencode_dc1(p,ix,iy+1,mom2)];
       fim1=(w[fencode_dc1(p,ix,iy-1,mom3)]/w[fencode_dc1(p,ix,iy-1,rho)])*w[fencode_dc1(p,ix,iy-1,mom2)];
       if(p->sodifon)
       {
       fip2=(w[fencode_dc1(p,ix,iy+2,mom3)]/w[fencode_dc1(p,ix,iy+2,rho)])*w[fencode_dc1(p,ix,iy+2,mom2)];
       fim2=(w[fencode_dc1(p,ix,iy-2,mom3)]/w[fencode_dc1(p,ix,iy-2,rho)])*w[fencode_dc1(p,ix,iy-2,mom2)];
       }
       #endif
       ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);
    break;
  }
  
  return(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);


  //return ( ddc1-ddc2);
}


//compute field contributions for momentum current
__device__ __host__
real ddotcurrentmom2 (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {
  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
  //real ddc1,ddc2;
  real ddcx,ddcy;
   //     ddc= grad_dc1(w,p,ix,iy,mom1,0)+grad_dc1(w,p,ix,iy,mom2,1);
//evalgrad_dc1(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
  //fi=w(fencode_dc1(p,ix,iy,rho))
  //calculate momentum current

//w[fencode_dc1(p,ix,iy,rho)])=1;
//w[fencode_dc1(p,ix-1,iy,rho)])=1;
//w[fencode_dc1(p,ix+2,iy,rho)])=1;
//w[fencode_dc1(p,ix-2,iy,rho)])=1;
//w[fencode_dc1(p,ix,iy,rho)])=1;
//w[fencode_dc1(p,ix,iy-1,rho)])=1;
//w[fencode_dc1(p,ix,iy+2,rho)])=1;
//w[fencode_dc1(p,ix,iy-2,rho)])=1;

//fip2=0, fim2=0;
  //calculate bfield current
  switch(direction)
  {
    case 0:
 
	       fi=w[fencode_dc1(p,ix+1,iy,b1)]*w[fencode_dc1(p,ix+1,iy,b1)];
	       fim1=w[fencode_dc1(p,ix-1,iy,b1)]*w[fencode_dc1(p,ix-1,iy,b1)];
	      if(p->sodifon)
	      {
	      fip2=w[fencode_dc1(p,ix+2,iy,b1)]*w[fencode_dc1(p,ix+2,iy,b1)];
	       fim2=w[fencode_dc1(p,ix-2,iy,b1)]*w[fencode_dc1(p,ix-2,iy,b1)];
	      }
  
       ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);

	       fi=w[fencode_dc1(p,ix,iy+1,b1)]*w[fencode_dc1(p,ix,iy+1,b2)];
	       fim1=w[fencode_dc1(p,ix,iy-1,b1)]*w[fencode_dc1(p,ix,iy-1,b2)];
		   if(p->sodifon)
	      {
	       fip2=w[fencode_dc1(p,ix,iy+2,b1)]*w[fencode_dc1(p,ix,iy+2,b2)];
	       fim2=w[fencode_dc1(p,ix,iy-2,b1)]*w[fencode_dc1(p,ix,iy-2,b2)];
	      }
  
       ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);
    break;
    case 1:

	       fi=w[fencode_dc1(p,ix+1,iy,b2)]*w[fencode_dc1(p,ix+1,iy,b1)];
	       fim1=w[fencode_dc1(p,ix-1,iy,b2)]*w[fencode_dc1(p,ix-1,iy,b1)];

	     if(p->sodifon)
	      {
	       fip2=w[fencode_dc1(p,ix+2,iy,b2)]*w[fencode_dc1(p,ix+2,iy,b1)];
	       fim2=w[fencode_dc1(p,ix-2,iy,b2)]*w[fencode_dc1(p,ix-2,iy,b1)];
	      }
 
	       ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);


	       fi=w[fencode_dc1(p,ix,iy+1,b2)]*w[fencode_dc1(p,ix,iy+1,b2)];
	       fim1=w[fencode_dc1(p,ix,iy-1,b2)]*w[fencode_dc1(p,ix,iy-1,b2)];

	      if(p->sodifon)
	      {
	       fip2=w[fencode_dc1(p,ix,iy+2,b2)]*w[fencode_dc1(p,ix,iy+2,b2)];
	       fim2=w[fencode_dc1(p,ix,iy-2,b2)]*w[fencode_dc1(p,ix,iy-2,b2)];
	      }
 
       ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);
    break;
    case 2:
	       fi=w[fencode_dc1(p,ix+1,iy,b3)]*w[fencode_dc1(p,ix+1,iy,b1)];
	       fim1=w[fencode_dc1(p,ix-1,iy,b3)]*w[fencode_dc1(p,ix-1,iy,b1)];

	      if(p->sodifon)
	      {
	       fip2=w[fencode_dc1(p,ix+2,iy,b3)]*w[fencode_dc1(p,ix+2,iy,b1)];
	       fim2=w[fencode_dc1(p,ix-2,iy,b3)]*w[fencode_dc1(p,ix-2,iy,b1)];
	       }
       ddcx=evalgrad_dc1(fi,fim1,fip2,fim2,p,0);

	       fi=w[fencode_dc1(p,ix,iy+1,b3)]*w[fencode_dc1(p,ix,iy+1,b2)];
	       fim1=w[fencode_dc1(p,ix,iy-1,b3)]*w[fencode_dc1(p,ix,iy-1,b2)];
		    if(p->sodifon)
	      {
	       fip2=w[fencode_dc1(p,ix,iy+2,b3)]*w[fencode_dc1(p,ix,iy+2,b2)];
	       fim2=w[fencode_dc1(p,ix,iy-2,b3)]*w[fencode_dc1(p,ix,iy-2,b2)];
	       }

       ddcy=evalgrad_dc1(fi,fim1,fip2,fim2,p,1);
    break;
  }
  //ddc2=ddcx+ddcy;
 // ddc2=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);
return(isnan(ddcx)?0:-ddcx)-(isnan(ddcy)?0:ddcy);
  //ddc=ddc1-ddc2;

  //return ( ddc1-ddc2);


}



__device__ __host__
int derivcurrentrho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode_dc1(p,ix,iy,field)]= -ddotcurrentrho(dw,wd,w,p,ix,iy);
     	//dw[fencode_dc1(p,ix,iy,field)]=w[fencode_dc1(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivcurrentmom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode_dc1(p,ix,iy,field)]=w[fencode_dc1(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode_dc1(p,ix,iy,field)]= -ddotcurrentmom1(dw,wd,w,p,ix,iy,field,direction)-ddotcurrentmom2(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_dc1(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}


//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void derivcurrent1 (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivcurrentrho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      derivcurrentmom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
       derivcurrentmom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      derivcurrentmom(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}



__global__ void derivcurrent1_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);


  //if(i>(1+(p->sodifon==1)) && j >(1+(p->sodifon==1)) && i<((p->n[0])-1-(p->sodifon==1)) && j<((p->n[1])-1-(p->sodifon==1)))
if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
	{		               
               /*for(int f=rho; f<=b3; f++)               
                  wmod[fencode_dc1(p,i,j,f)]=w[fencode_dc1(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);*/
          
               for(int f=rho; f<=mom3; f++)
               {              
                  derivcurrent1(dwn1+(NVAR*(p->n[0])*(p->n[1])*order),wd,wmod,p,i,j,f);
                  //dwn1[fencode_dc1(p,i,j,f)]=1.0;
                  //__syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc1(p,i,j,f)]=w[fencode_dc1(p,i,j,f)]+0.5*dt*dwn1[fencode_dc1(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc1(p,i,j,f)]=w[fencode_dc1(p,i,j,f)]+0.5*dt*dwn2[fencode_dc1(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_dc1(p,i,j,f)]=w[fencode_dc1(p,i,j,f)]+dt*dwn3[fencode_dc1(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_dc1(p,i,j,f)]=w[fencode_dc1(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_dc1(p,i,j,f)]+2.0*dwn2[fencode_dc1(p,i,j,f)]
                         +2.0*dwn3[fencode_dc1(p,i,j,f)]+dwn4[fencode_dc1(p,i,j,f)]);
               }*/
              //  __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_dc1(p,i,j,f)]=w[fencode_dc1(p,i,j,f)]+dt*dwn1[fencode_dc1(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_dc1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuderivcurrent1(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;
 //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
 // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivcurrent1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


