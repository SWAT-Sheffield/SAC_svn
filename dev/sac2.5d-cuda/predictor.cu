#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_pre (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_pre (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
float evalgrad_pre(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
{
 //float valgrad_pre;

 if(dir == 0)
 {
     //valgrad_pre=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   return((1.0/(1.0*(p->dx)))*(fi-fim1));
 }
 else if(dir == 1)
 {
    // valgrad_pre=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
      return((1.0/(1.0*(p->dy)))*(fi-fim1));
 }

 return -1;
}


__device__ __host__
float grad_pre(float *wmod,struct params *p,int i,int j,int field,int dir)
{
 //float valgrad_pre;

 if(dir == 0)
 {
    // valgrad_pre=(2.0/(3.0*(p->dx)))*(wmod[fencode_pre(p,i,j,field)]-wmod[fencode_pre(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode_pre(p,i+2,j,field)]-wmod[fencode_pre(p,i-2,j,field)]);
return((1.0/(1.0*(p->dx)))*(wmod[fencode_pre(p,i+1,j,field)]-wmod[fencode_pre(p,i-1,j,field)]));
 }
 else if(dir == 1)
 {
    // valgrad_pre=(2.0/(3.0*(p->dy)))*(wmod[fencode_pre(p,i,j,field)]-wmod[fencode_pre(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode_pre(p,i,j+2,field)]-wmod[fencode_pre(p,i,j-2,field)]);
 return((1.0/(1.0*(p->dy)))*(wmod[fencode_pre(p,i,j+1,field)]-wmod[fencode_pre(p,i,j-1,field)]));

 }

 return -1;
}

__device__ __host__
void computej(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;

 // float dbzdy, dbydz;
 // float dbzdx, dbxdz;
 // float dbydx, dbxdy;

 // dbzdy=grad_pre(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_pre(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_pre(wmod,p,i,j,b2,0);
 // dbxdy=grad_pre(wmod,p,i,j,b1,1);

  wd[fencode_pre(p,i,j,0)]=(grad_pre(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_pre(p,i,j,1)]=(grad_pre(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_pre(p,i,j,2)]=(grad_pre(wmod,p,i,j,b2,0)-grad_pre(wmod,p,i,j,b1,1))/(p->mu);
 
  //return ( status);
}

__device__ __host__
void computebdotv(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
//  wd[fencode_pre(p,i,j,4)]=  wd[fencode_pre(p,i,j,3)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

wd[fencode_pre(p,i,j,bdotv)]=(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,mom3)])/wmod[fencode_pre(p,i,j,rho)];
 // return ( status);
}


__device__ __host__
void computepk(float *wmod,float *wd,struct params *p,int i,int j)
{
 // int status=0;
 //float bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
  wd[fencode_pre(p,i,j,4)]=  wd[fencode_pre(p,i,j,3)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);
 // return ( status);
}
__device__ __host__
void computept(float *wmod,float *wd,struct params *p,int i,int j)
{
  //int status=0;
  //float momsq=wmod[fencode_pre(p,i,j,mom1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,mom2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,mom3)]*wmod[fencode_pre(p,i,j,mom3)];
  //float bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
  wd[fencode_pre(p,i,j,3)]=((p->gamma)-1)*(wmod[fencode_pre(p,i,j,energy)]- 0.5*(wmod[fencode_pre(p,i,j,mom1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,mom2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,mom3)]*wmod[fencode_pre(p,i,j,mom3)])/wmod[fencode_pre(p,i,j,rho)]-0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]) );
  //return ( status);
}

__device__ __host__
void computec(float *wmod,float *wd,struct params *p,int i,int j)
{

  wd[fencode_pre(p,i,j,soundspeed)]=sqrt(((p->gamma)-1)*wd[fencode_pre(p,i,j,pressuret)]/wmod[fencode_pre(p,i,j,rho)]);
}

__device__ __host__
void computecmax(float *wmod,float *wd,struct params *p,int i,int j)
{

       if(wd[fencode_pre(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_pre(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_pre(p,i,j,soundspeed)]);

}


__global__ void predictor_parallel(struct params *p,  float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               for(int f=rho; f<=b3; f++)               
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);
 //determin cmax
               computec(wmod,wd,p,i,j);
               __syncthreads();
               computecmax(wmod,wd,p,i,j);

               /*for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_pre(p,i,j,f)]=1.0;
                  __syncthreads();
               }*/
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+0.5*dt*dwn1[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+0.5*dt*dwn2[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+dt*dwn3[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_pre(p,i,j,f)]+2.0*dwn2[fencode_pre(p,i,j,f)]
                         +2.0*dwn3[fencode_pre(p,i,j,f)]+dwn4[fencode_pre(p,i,j,f)]);
               }*/
                __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+dt*dwn1[fencode_pre(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_pre(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cupredictor(struct params **p, float **w, float **wnew, struct params **d_p, float **d_w, float **d_wnew, float **d_wmod, float **d_dwn1, float **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     predictor_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
     hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






