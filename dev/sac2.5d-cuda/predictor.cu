#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_pre (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( iy * ((dp)->n[0]) + ix);
}

__device__ __host__
int fencode_pre (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->n[0];
  //int kSizeY=(dp)->n[1];
  
  return ( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])));
}

__device__ __host__
real evalgrad_pre(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_pre;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx[0])))*(fi-fim1)-(1.0/(12.0*(p->dx[0])))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx[0])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[0])))*(fi-fim1)):((1.0/(12.0*(p->dx[0])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(fi-fim1)-(1.0/(12.0*(p->dx[1])))*(fip2-fim2);
     // return((2.0/(1.0*(p->dx[1])))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx[1])))*(fi-fim1)):((1.0/(12.0*(p->dx[1])))*((NVAR*fi-NVAR*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_pre(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_pre;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx[0])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx[0])))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx[0])))*(wmod[fencode_pre(p,i+1,j,field)]-wmod[fencode_pre(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((NVAR*wmod[fencode_pre(p,i+1,j,field)]-NVAR*wmod[fencode_pre(p,i-1,j,field)]+wmod[fencode_pre(p,i-2,j,field)]-wmod[fencode_pre(p,i+2,j,field)])/6.0):wmod[fencode_pre(p,i+1,j,field)]-wmod[fencode_pre(p,i-1,j,field)])/(2.0*(p->dx[0]))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dx[1])))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dx[1])))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dx[1])))*(wmod[fencode_pre(p,i,j+1,field)]-wmod[fencode_pre(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((NVAR*wmod[fencode_pre(p,i,j+1,field)]-NVAR*wmod[fencode_pre(p,i,j-1,field)]+wmod[fencode_pre(p,i,j-2,field)]-wmod[fencode_pre(p,i,j+2,field)])/6.0):wmod[fencode_pre(p,i,j+1,field)]-wmod[fencode_pre(p,i,j-1,field)])/(2.0*(p->dx[1]))    );  
}


 return 0;
}

__device__ __host__
void computej(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad_pre(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_pre(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_pre(wmod,p,i,j,b2,0);
 // dbxdy=grad_pre(wmod,p,i,j,b1,1);

  wd[fencode_pre(p,i,j,0)]=(grad_pre(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_pre(p,i,j,1)]=(grad_pre(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_pre(p,i,j,2)]=(grad_pre(wmod,p,i,j,b2,0)-grad_pre(wmod,p,i,j,b1,1))/(p->mu);
  
  

 
  //return ( status);
}

__device__ __host__
void computebdotv(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
//  wd[fencode_pre(p,i,j,4)]=  wd[fencode_pre(p,i,j,3)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

wd[fencode_pre(p,i,j,bdotv)]=(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,mom3)])/wmod[fencode_pre(p,i,j,rho)];
 // return ( status);
}

__device__ __host__
void computedivb(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
//  wd[fencode_pre(p,i,j,4)]=  wd[fencode_pre(p,i,j,3)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

wd[fencode_pre(p,i,j,divb)]=grad_pre(wmod,p,i,j,b1,0)+grad_pre(wmod,p,i,j,b2,1);
 // return ( status);
}


__device__ __host__
void computept(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode_pre(p,i,j,pressuret)]=(p->adiab)*pow(wmod[fencode_pre(p,i,j,rho)],p->gamma);


#else

 //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
  wd[fencode_pre(p,i,j,pressuret)]=  wd[fencode_pre(p,i,j,pressurek)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

#endif



  if(wd[fencode_pre(p,i,j,pressuret)]<0)
              wd[fencode_pre(p,i,j,pressuret)]=0.001;


 // return ( status);
}
__device__ __host__
void computepk(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_pre(p,i,j,pressurek)]=(p->adiab)*pow(wmod[fencode_pre(p,i,j,rho)],p->gamma);

#else

  //real momsq=wmod[fencode_pre(p,i,j,mom1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,mom2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,mom3)]*wmod[fencode_pre(p,i,j,mom3)];
  //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
  wd[fencode_pre(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_pre(p,i,j,energy)]- 0.5*(wmod[fencode_pre(p,i,j,mom1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,mom2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,mom3)]*wmod[fencode_pre(p,i,j,mom3)])/wmod[fencode_pre(p,i,j,rho)]-0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]) );


#endif






  if(wd[fencode_pre(p,i,j,pressurek)]<0)
              wd[fencode_pre(p,i,j,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec(real *wmod,real *wd,struct params *p,int i,int j)
{

  
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode_pre(p,i,j,soundspeed)]=sqrt((p->adiab)/wmod[fencode_pre(p,i,j,rho)]);

#else
wd[fencode_pre(p,i,j,soundspeed)]=sqrt(((p->gamma))*wd[fencode_pre(p,i,j,pressuret)]/wmod[fencode_pre(p,i,j,rho)]);
wd[fencode_pre(p,i,j,cfast)]=sqrt(((wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)])/wmod[fencode_pre(p,i,j,rho)])+(wd[fencode_pre(p,i,j,soundspeed)]*wd[fencode_pre(p,i,j,soundspeed)]));
#endif



  
}

__device__ __host__
void computecmax(real *wmod,real *wd,struct params *p,int i,int j)
{

       if(wd[fencode_pre(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_pre(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_pre(p,i,j,soundspeed)]);
       if(wd[fencode_pre(p,i,j,cfast)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_pre(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_pre(p,i,j,cfast)]);

}


__global__ void predictor_parallel(struct params *p,  real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
if(i<((p->n[0])) && j<((p->n[1])))
	{		
               for(int f=rho; f<=b3; f++)
               {               
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)];
                  wnew[fencode_pre(p,i,j,f)]=0.0;
               }
               for(int f=current1; f<=divb; f++)
                  wd[fencode_pre(p,i,j,f)]=0; 
        }
               __syncthreads();


  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);

               computebdotv(wmod,wd,p,i,j);
               computedivb(wmod,wd,p,i,j);
         }
              __syncthreads();
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec(wmod,wd,p,i,j);
        }
              __syncthreads();

  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{ 
               computecmax(wmod,wd,p,i,j);

               /*for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_pre(p,i,j,f)]=1.0;
                  __syncthreads();
               }*/
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+0.5*dt*dwn1[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+0.5*dt*dwn2[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+dt*dwn3[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_pre(p,i,j,f)]+2.0*dwn2[fencode_pre(p,i,j,f)]
                         +2.0*dwn3[fencode_pre(p,i,j,f)]+dwn4[fencode_pre(p,i,j,f)]);
               }*/
           //     __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+dt*dwn1[fencode_pre(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_pre(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cupredictor(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     predictor_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






