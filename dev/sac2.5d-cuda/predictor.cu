#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_pre (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_pre (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
real evalgrad_pre(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_pre;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx)))*(fi-fim1)):((1.0/(12.0*(p->dx)))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
     // return((2.0/(1.0*(p->dy)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dy)))*(fi-fim1)):((1.0/(12.0*(p->dy)))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_pre(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_pre;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx)))*(wmod[fencode_pre(p,i+1,j,field)]-wmod[fencode_pre(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_pre(p,i+1,j,field)]-8*wmod[fencode_pre(p,i-1,j,field)]+wmod[fencode_pre(p,i-1,j,field)]-wmod[fencode_pre(p,i+1,j,field)])/6.0):wmod[fencode_pre(p,i+1,j,field)]-wmod[fencode_pre(p,i-1,j,field)])/(2.0*(p->dx))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dy)))*(wmod[fencode_pre(p,i,j+1,field)]-wmod[fencode_pre(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_pre(p,i,j+1,field)]-8*wmod[fencode_pre(p,i,j-1,field)]+wmod[fencode_pre(p,i,j-1,field)]-wmod[fencode_pre(p,i,j+1,field)])/6.0):wmod[fencode_pre(p,i,j+1,field)]-wmod[fencode_pre(p,i,j-1,field)])/(2.0*(p->dy))    );

 }

 return 0;
}

__device__ __host__
void computej(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

 // real dbzdy, dbydz;
 // real dbzdx, dbxdz;
 // real dbydx, dbxdy;

 // dbzdy=grad_pre(wmod,p,i,j,b3,1);
 // dbydz=0.0;
 // dbzdx=grad_pre(wmod,p,i,j,b3,0);
//  dbxdz=0.0;
 // dbydx=grad_pre(wmod,p,i,j,b2,0);
 // dbxdy=grad_pre(wmod,p,i,j,b1,1);

  wd[fencode_pre(p,i,j,0)]=(grad_pre(wmod,p,i,j,b3,1))/(p->mu);
  wd[fencode_pre(p,i,j,1)]=(grad_pre(wmod,p,i,j,b3,0))/(p->mu);
  wd[fencode_pre(p,i,j,2)]=(grad_pre(wmod,p,i,j,b2,0)-grad_pre(wmod,p,i,j,b1,1))/(p->mu);
  
  

 
  //return ( status);
}

__device__ __host__
void computebdotv(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
//  wd[fencode_pre(p,i,j,4)]=  wd[fencode_pre(p,i,j,3)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

wd[fencode_pre(p,i,j,bdotv)]=(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,mom3)])/wmod[fencode_pre(p,i,j,rho)];
 // return ( status);
}

__device__ __host__
void computedivb(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;
 //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
//  wd[fencode_pre(p,i,j,4)]=  wd[fencode_pre(p,i,j,3)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

wd[fencode_pre(p,i,j,divb)]=grad_pre(wmod,p,i,j,b1,0)+grad_pre(wmod,p,i,j,b2,1);
 // return ( status);
}


__device__ __host__
void computept(real *wmod,real *wd,struct params *p,int i,int j)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode_pre(p,i,j,pressuret)]=(p->adiab)*pow(wmod[fencode_pre(p,i,j,rho)],p->gamma);


#else

 //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
  wd[fencode_pre(p,i,j,pressuret)]=  wd[fencode_pre(p,i,j,pressurek)]+0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]);

#endif



  if(wd[fencode_pre(p,i,j,pressuret)]<0)
              wd[fencode_pre(p,i,j,pressuret)]=0.001;


 // return ( status);
}
__device__ __host__
void computepk(real *wmod,real *wd,struct params *p,int i,int j)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode_pre(p,i,j,pressurek)]=(p->adiab)*pow(wmod[fencode_pre(p,i,j,rho)],p->gamma);

#else

  //real momsq=wmod[fencode_pre(p,i,j,mom1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,mom2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,mom3)]*wmod[fencode_pre(p,i,j,mom3)];
  //real bsq=wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)];
  wd[fencode_pre(p,i,j,pressurek)]=((p->gamma)-1)*(wmod[fencode_pre(p,i,j,energy)]- 0.5*(wmod[fencode_pre(p,i,j,mom1)]*wmod[fencode_pre(p,i,j,mom1)]+wmod[fencode_pre(p,i,j,mom2)]*wmod[fencode_pre(p,i,j,mom2)]+wmod[fencode_pre(p,i,j,mom3)]*wmod[fencode_pre(p,i,j,mom3)])/wmod[fencode_pre(p,i,j,rho)]-0.5*(wmod[fencode_pre(p,i,j,b1)]*wmod[fencode_pre(p,i,j,b1)]+wmod[fencode_pre(p,i,j,b2)]*wmod[fencode_pre(p,i,j,b2)]+wmod[fencode_pre(p,i,j,b3)]*wmod[fencode_pre(p,i,j,b3)]) );


#endif






  if(wd[fencode_pre(p,i,j,pressurek)]<0)
              wd[fencode_pre(p,i,j,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec(real *wmod,real *wd,struct params *p,int i,int j)
{

  
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode_pre(p,i,j,soundspeed)]=sqrt((p->adiab)/wmod[fencode_pre(p,i,j,rho)]);

#else
wd[fencode_pre(p,i,j,soundspeed)]=sqrt(((p->gamma))*wd[fencode_pre(p,i,j,pressuret)]/wmod[fencode_pre(p,i,j,rho)]);

#endif



  
}

__device__ __host__
void computecmax(real *wmod,real *wd,struct params *p,int i,int j)
{

       if(wd[fencode_pre(p,i,j,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode_pre(p,i,j,soundspeed)]));
                    p->cmax=(wd[fencode_pre(p,i,j,soundspeed)]);

}


__global__ void predictor_parallel(struct params *p,  real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
if(i<((p->ni)) && j<((p->nj)))
	{		
               for(int f=rho; f<=b3; f++)
               {               
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)];
                  wnew[fencode_pre(p,i,j,f)]=0.0;
               }
               for(int f=current1; f<=divb; f++)
                  wd[fencode_pre(p,i,j,f)]=0; 
        }
               __syncthreads();


  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);

              // if(j==2)
              // {
              //   wd[fencode_pre(p,i,0,3)]=wd[fencode_pre(p,i,j,4)];
              //   wd[fencode_pre(p,i,1,4)]=wd[fencode_pre(p,i,j,4)];

              // }
               computebdotv(wmod,wd,p,i,j);
               computedivb(wmod,wd,p,i,j);
         }
              __syncthreads();
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{
 //determin cmax
               computec(wmod,wd,p,i,j);
        }
              __syncthreads();

  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{ 
               computecmax(wmod,wd,p,i,j);

               /*for(int f=rho; f<=b3; f++)
               {              
                  deriv(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_pre(p,i,j,f)]=1.0;
                  __syncthreads();
               }*/
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+0.5*dt*dwn1[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+0.5*dt*dwn2[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+dt*dwn3[fencode_pre(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_pre(p,i,j,f)]+2.0*dwn2[fencode_pre(p,i,j,f)]
                         +2.0*dwn3[fencode_pre(p,i,j,f)]+dwn4[fencode_pre(p,i,j,f)]);
               }*/
           //     __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_pre(p,i,j,f)]=w[fencode_pre(p,i,j,f)]+dt*dwn1[fencode_pre(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_pre(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cupredictor(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     predictor_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
     hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






