#include "hip/hip_runtime.h"
//#define MODID pre


#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"
#include "gradops_cdf.cuh"
#include "dervfields_cdf.cuh"
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void computevels_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  








  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         #endif
                        }


         }

}
              __syncthreads();











  
}


__global__ void computepres_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  








  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #else
	       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #endif         
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


         }

}
              __syncthreads();











  
}


__global__ void computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  



 /*  for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
        }

}*/
              __syncthreads();



if(iindex==0)
{
   
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    //p->cmax=0.0;
    for(ii[0]=2;ii[0]<((p->n[0])-2);ii[0]++)
      for(ii[1]=2;ii[1]<((p->n[1])-2);ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]>1;ii[2]<((p->n[2])-2);ii[2]++)
     #endif
	{ 
               computecmax3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);




	}

 //  }
}
 __syncthreads(); 





  
}

__global__ void computec_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  






 p->cmax=0.0;
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
        }

}
              __syncthreads();












  
}


__global__ void computedervfields_parallel(struct params *p,   real *wmod, real *wd, int order)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  





if(order == 0)
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		
              #ifdef ADIABHYDRO
              for(int f=vel1; f<NDERV; f++)
             #else 
               for(int f=vel1; f<=pkb; f++)
             #endif
                        wd[fencode3_cdf(p,ii,f)]=0; 
		#ifdef USE_SAC_3D
		  for(int f=rho; f<NVAR; f++)
                  	wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]=wmod[fencode3_cdf(p,ii,f)]; 

		#else
		  for(int f=rho; f<NVAR; f++)
                  	wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]=wmod[fencode3_cdf(p,ii,f)]; 
		#endif               

        }

}
               __syncthreads();



  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if( ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               
             #ifdef ADIABHYDRO
               //computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
             #else
               //computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computej3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

               computebdotv3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computedivb3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

             #endif

         }

}
              __syncthreads();

  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cdf(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucomputedervfields(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   ////hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computedervfields_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputevels(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   ////hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computevels_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputemaxc(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}


int cucomputec(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computec_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputepres(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computepres_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}







