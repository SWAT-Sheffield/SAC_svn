#include "hip/hip_runtime.h"
//#define MODID pre


#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"
#include "gradops_cdf.cuh"
#include "dervfields_cdf.cuh"
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////


__global__ void computedervfields_parallel(struct params *p,   real *wmod, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   int ip,jp,ipg,jpg;
   jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));


if(order == 0)
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;




if(i<((p->n[0])) && j<((p->n[1])))
	{		
 
               for(int f=rho; f<=b2; f++)
                  wmod[fencode_cdf(p,i,j,f)+((p->n[0]))*((p->n[1]))*NVAR]=wmod[fencode_cdf(p,i,j,f)]; 
        }

}
               __syncthreads();

/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
if(i<((p->n[0])) && j<((p->n[1])))
	{		

               for(int f=vel1; f<NDERV; f++)
                 ;// wd[fencode_cdf(p,i,j,f)]=0; 
               for(int f=rho; f<NVAR; f++)
                 ;// dwn1[fencode_cdf(p,i,j,f)]=0; 
        }

}
               __syncthreads();*/

//if(i>20 && j >20 && i<90 && j<90)
//	{
//               computepk_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
//              computept_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
//}
//              __syncthreads();


/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
#ifdef USE_VAC
 if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
                    computej_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
#endif

#ifdef USE_SAC
 if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
                    computej_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
#endif

}
__syncthreads(); */


  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<((p->n[0])) && j<((p->n[1])))
	{		               
             #ifdef ADIABHYDRO
               computepk_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computept_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
             #else
               //computej_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computepk_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               computept_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);

               computebdotv_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               //computedivb_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);

             #endif

         }

}
              __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
  if(i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);
               p->cmax=0.0;
        }

}
              __syncthreads();

if(iindex==0)
{
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    p->cmax=0.0;
    for(i>1;i<((p->n[0])-2);i++)
      for(j>1;j<((p->n[1])-2);j++)
	{ 
               computecmax_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);




	}

 //  }
}
 __syncthreads(); 




  /* for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
     i=2*i;
     j=2*j;

  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
               computecmax_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);

	

   }

 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   {

     i=ip*(p->npgp[0])+ipg;
     j=jp*(p->npgp[1])+jpg;
     i=2*i+1;
     j=2*j+1;

  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
               computecmax_cdf(wmod+(order*((p->n[0]))*((p->n[1]))*NVAR),wd,p,i,j);

	

   }

 __syncthreads();*/
  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cdf(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucomputedervfields(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order)
{


 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


     computedervfields_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}






