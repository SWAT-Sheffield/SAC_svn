#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_ds (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_ds (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
float evalgrad_ds(float fi, float fim1, float fip2, float fim2,struct params *p,int dir)
{
 //float valgrad_ds;

 if(dir == 0)
 {
     //valgrad_ds=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   return((1.0/(1.0*(p->dx)))*(fi-fim1));
 }
 else if(dir == 1)
 {
    // valgrad_ds=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
      return((1.0/(1.0*(p->dy)))*(fi-fim1));
 }

 return -1;
}


__device__ __host__
float grad_ds(float *wmod,struct params *p,int i,int j,int field,int dir)
{
 //float valgrad_ds;

 if(dir == 0)
 {
    // valgrad_ds=(2.0/(3.0*(p->dx)))*(wmod[fencode_ds(p,i,j,field)]-wmod[fencode_ds(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode_ds(p,i+2,j,field)]-wmod[fencode_ds(p,i-2,j,field)]);
return((1.0/(1.0*(p->dx)))*(wmod[fencode_ds(p,i+1,j,field)]-wmod[fencode_ds(p,i-1,j,field)]));
 }
 else if(dir == 1)
 {
    // valgrad_ds=(2.0/(3.0*(p->dy)))*(wmod[fencode_ds(p,i,j,field)]-wmod[fencode_ds(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode_ds(p,i,j+2,field)]-wmod[fencode_ds(p,i,j-2,field)]);
 return((1.0/(1.0*(p->dy)))*(wmod[fencode_ds(p,i,j+1,field)]-wmod[fencode_ds(p,i,j-1,field)]));

 }

 return -1;
}

__device__ __host__
float sourcerho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float src=0;
 // int field=rho;
 
  return 0;
}

__device__ __host__
float sourcemom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float src=0;
  switch(direction)
  {
	case 0:
         return(w[fencode_ds(p,ix,iy,rho)]*(p->g1))-grad_ds(wd,p,ix,iy,pressuret,0);
	break;
	case 1:
         return(w[fencode_ds(p,ix,iy,rho)]*(p->g2))-grad_ds(wd,p,ix,iy,pressuret,1);
	break;
	case 2:
         return(w[fencode_ds(p,ix,iy,rho)]*(p->g3))-grad_ds(wd,p,ix,iy,pressuret,2);
	break;
  }
  return 0;
}

__device__ __host__
float sourceb (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  //float src=0;
  switch(direction)
  {
	case 0:
         return(p->eta)*grad_ds(wd,p,ix,iy,current3,1);
	break;
	case 1:
         return -(p->eta)*grad_ds(wd,p,ix,iy,current3,0);
	break;
	case 2:
         return (p->eta)*(grad_ds(wd,p,ix,iy,current2,0)-grad_ds(wd,p,ix,iy,current1,1));
	break;
  }
  return 0;
}

__device__ __host__
float sourceenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

 // float src=0;
  float srcg,srcb;
  int field=energy;
  float ddcx,ddcy;
  float fi,fim1;//fip2,fim2;
      srcg=(p->g1)*w[fencode_ds(p,ix,iy,mom1)]+(p->g2)*w[fencode_ds(p,ix,iy,mom2)]+(p->g3)*w[fencode_ds(p,ix,iy,mom3)];

       fi=(w[fencode_ds(p,ix+1,iy,b2)]*wd[fencode_ds(p,ix+1,iy,current3)]-w[fencode_ds(p,ix+1,iy,b3)]*wd[fencode_ds(p,ix+1,iy,current2)]);
       fim1=(w[fencode_ds(p,ix-1,iy,b2)]*wd[fencode_ds(p,ix-1,iy,current3)]-w[fencode_ds(p,ix-1,iy,b3)]*wd[fencode_ds(p,ix-1,iy,current2)]);
      // fip2=(w[fencode_ds(p,ix+2,iy,b2)]*wd[fencode_ds(p,ix+2,iy,current3)]-w[fencode_ds(p,ix+2,iy,b3)]*wd[fencode_ds(p,ix+2,iy,current2)]);
     //  fim2=(w[fencode_ds(p,ix-2,iy,b2)]*wd[fencode_ds(p,ix-2,iy,current3)]-w[fencode_ds(p,ix-2,iy,b3)]*wd[fencode_ds(p,ix-2,iy,current2)]);
      // ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad_ds(fi,fim1,0,0,p,0);

       fi=(w[fencode_ds(p,ix+1,iy,b3)]*wd[fencode_ds(p,ix+1,iy,current1)]-w[fencode_ds(p,ix+1,iy,b1)]*wd[fencode_ds(p,ix+1,iy,current3)]);
       fim1=(w[fencode_ds(p,ix,iy-1,b3)]*wd[fencode_ds(p,ix,iy-1,current1)]-w[fencode_ds(p,ix,iy-1,b1)]*wd[fencode_ds(p,ix,iy-1,current3)]);
     //  fip2=(w[fencode_ds(p,ix,iy+2,b3)]*wd[fencode_ds(p,ix,iy+2,current1)]-w[fencode_ds(p,ix,iy+2,b1)]*wd[fencode_ds(p,ix,iy+2,current3)]);
     //  fim2=(w[fencode_ds(p,ix,iy-2,b3)]*wd[fencode_ds(p,ix,iy-2,current1)]-w[fencode_ds(p,ix,iy-2,b1)]*wd[fencode_ds(p,ix,iy-2,current3)]);
      // ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);
      ddcy=evalgrad_ds(fi,fim1,0,0,p,1);

      srcb=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

 // src=srcg+srcb;
  return ( srcg+srcb);
}


__device__ __host__
int derivsourcerho (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourcerho(dw,wd,w,p,ix,iy);
     	//dw[fencode_ds(p,ix,iy,field)]=w[fencode_ds(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivsourcemom (float *dw, float *wd, float *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode_ds(p,ix,iy,field)]=w[fencode_ds(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourcemom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_ds(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivsourceb (float *dw, float *wd, float *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourceb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivsourceenergy (float *dw, float *wd, float *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourceenergy(dw,wd,w,p,ix,iy);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void derivsource (float *dw, float *wd, float *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivsourcerho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      derivsourcemom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
      derivsourcemom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      derivsourcemom(dw,wd,w,p,ix,iy,field,2);
     break;
     case energy:
       derivsourceenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
      derivsourceb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      derivsourceb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      derivsourceb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}


__global__ void derivsource_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
    float *dwn1, float *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  float dt=p->dt;
  float dy=p->dy;
  float dx=p->dx;
  float g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               /*for(int f=rho; f<=b3; f++)               
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);*/
               for(int f=rho; f<=b3; f++)
               {              
                  derivsource(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_ds(p,i,j,f)]=1.0;
                  __syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+0.5*dt*dwn1[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+0.5*dt*dwn2[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+dt*dwn3[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_ds(p,i,j,f)]+2.0*dwn2[fencode_ds(p,i,j,f)]
                         +2.0*dwn3[fencode_ds(p,i,j,f)]+dwn4[fencode_ds(p,i,j,f)]);
               }*/
                __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+dt*dwn1[fencode_ds(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_ds(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuderivsource(struct params **p, float **w, float **wnew, float **b,struct params **d_p, float **d_w, float **d_wnew, float **d_b, float **d_wmod, float **d_dwn1, float **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, float *b, float *w, float *wnew, float *wmod, 
  //  float *dwn1, float *dwn2, float *dwn3, float *dwn4, float *wd)
     //init_parallel(struct params *p, float *b, float *u, float *v, float *h)
     derivsource_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(float), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(float), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







