#include "hip/hip_runtime.h"
#include "cudapars.h"
#include "paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "step.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////



__device__ __host__
int encode_ds (struct params *dp,int ix, int iy) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( iy * ((dp)->ni) + ix);
}

__device__ __host__
int fencode_ds (struct params *dp,int ix, int iy, int field) {

  //int kSizeX=(dp)->ni;
  //int kSizeY=(dp)->nj;
  
  return ( (iy * ((dp)->ni) + ix)+(field*((dp)->ni)*((dp)->nj)));
}

__device__ __host__
real evalgrad_ds(real fi, real fim1, real fip2, real fim2,struct params *p,int dir)
{
 //real valgrad_ds;

 if(dir == 0)
 {
     //valgrad=(2.0/(3.0*(p->dx)))*(fi-fim1)-(1.0/(12.0*(p->dx)))*(fip2-fim2);
   //return((1.0/(2.0*(p->dx)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dx)))*(fi-fim1)):((1.0/(12.0*(p->dx)))*((8*fi-8*fim1+fim2-fip2))));
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(fi-fim1)-(1.0/(12.0*(p->dy)))*(fip2-fim2);
     // return((2.0/(1.0*(p->dy)))*(fi-fim1));
   return(p->sodifon?((1.0/(2.0*(p->dy)))*(fi-fim1)):((1.0/(12.0*(p->dy)))*((8*fi-8*fim1+fim2-fip2))));
 }

 return -1;
}


__device__ __host__
real grad_ds(real *wmod,struct params *p,int i,int j,int field,int dir)
{
 //real valgrad_ds;

 if(dir == 0)
 {
    // valgrad=(2.0/(3.0*(p->dx)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i-1,j,field)])-(1.0/(12.0*(p->dx)))*(wmod[fencode(p,i+2,j,field)]-wmod[fencode(p,i-2,j,field)]);
//return((1.0/(2.0*(p->dx)))*(wmod[fencode_ds(p,i+1,j,field)]-wmod[fencode_ds(p,i-1,j,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_ds(p,i+1,j,field)]-8*wmod[fencode_ds(p,i-1,j,field)]+wmod[fencode_ds(p,i-1,j,field)]-wmod[fencode_ds(p,i+1,j,field)])/6.0):wmod[fencode_ds(p,i+1,j,field)]-wmod[fencode_ds(p,i-1,j,field)])/(2.0*(p->dx))    );
 }
 else if(dir == 1)
 {
    // valgrad=(2.0/(3.0*(p->dy)))*(wmod[fencode(p,i,j,field)]-wmod[fencode(p,i,j-1,field)])-(1.0/(12.0*(p->dy)))*(wmod[fencode(p,i,j+2,field)]-wmod[fencode(p,i,j-2,field)]);
// return((1.0/(2.0*(p->dy)))*(wmod[fencode_ds(p,i,j+1,field)]-wmod[fencode_ds(p,i,j-1,field)]));
 return(  ( (p->sodifon)?((8*wmod[fencode_ds(p,i,j+1,field)]-8*wmod[fencode_ds(p,i,j-1,field)]+wmod[fencode_ds(p,i,j-1,field)]-wmod[fencode_ds(p,i,j+1,field)])/6.0):wmod[fencode_ds(p,i,j+1,field)]-wmod[fencode_ds(p,i,j-1,field)])/(2.0*(p->dy))    );

 }

 return 0;
}

__device__ __host__
real sourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 // real src=0;
 // int field=rho;
 
  return 0;
}

__device__ __host__
real sourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src=(w[fencode_ds(p,ix,iy,rho)]*(p->g1))-grad_ds(wd,p,ix,iy,pressuret,0);
        // src=(w[fencode_ds(p,ix,iy,rho)]*(p->g1));
	break;
	case 1:
         src=(w[fencode_ds(p,ix,iy,rho)]*(p->g2))-grad_ds(wd,p,ix,iy,pressuret,1);
         //src=(w[fencode_ds(p,ix,iy,rho)]*(p->g2));
	break;
	case 2:
         //src=(w[fencode_ds(p,ix,iy,rho)]*(p->g3))-grad_ds(wd,p,ix,iy,pressuret,2);
         src=(w[fencode_ds(p,ix,iy,rho)]*(p->g3));
	break;
  }

  return(isnan(src)?0:src);


}

__device__ __host__
real sourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src=(p->eta)*grad_ds(wd,p,ix,iy,current3,1);
	break;
	case 1:
         src= -(p->eta)*grad_ds(wd,p,ix,iy,current3,0);
	break;
	case 2:
         src= (p->eta)*(grad_ds(wd,p,ix,iy,current2,0)-grad_ds(wd,p,ix,iy,current1,1));
	break;
  }
   return(isnan(src)?0:src);
}

__device__ __host__
real sourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

 // real src=0;
  real srcg,srcb;
  int field=energy;
  real ddcx,ddcy;
  real fi,fim1;//fip2,fim2;
      srcg=(p->g1)*w[fencode_ds(p,ix,iy,mom1)]+(p->g2)*w[fencode_ds(p,ix,iy,mom2)]+(p->g3)*w[fencode_ds(p,ix,iy,mom3)];

       fi=(w[fencode_ds(p,ix+1,iy,b2)]*wd[fencode_ds(p,ix+1,iy,current3)]-w[fencode_ds(p,ix+1,iy,b3)]*wd[fencode_ds(p,ix+1,iy,current2)]);
       fim1=(w[fencode_ds(p,ix-1,iy,b2)]*wd[fencode_ds(p,ix-1,iy,current3)]-w[fencode_ds(p,ix-1,iy,b3)]*wd[fencode_ds(p,ix-1,iy,current2)]);
      // fip2=(w[fencode_ds(p,ix+2,iy,b2)]*wd[fencode_ds(p,ix+2,iy,current3)]-w[fencode_ds(p,ix+2,iy,b3)]*wd[fencode_ds(p,ix+2,iy,current2)]);
     //  fim2=(w[fencode_ds(p,ix-2,iy,b2)]*wd[fencode_ds(p,ix-2,iy,current3)]-w[fencode_ds(p,ix-2,iy,b3)]*wd[fencode_ds(p,ix-2,iy,current2)]);
      // ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);
      ddcx=evalgrad_ds(fi,fim1,0,0,p,0);

       fi=(w[fencode_ds(p,ix,iy+1,b3)]*wd[fencode_ds(p,ix,iy+1,current1)]-w[fencode_ds(p,ix,iy+1,b1)]*wd[fencode_ds(p,ix,iy+1,current3)]);
       fim1=(w[fencode_ds(p,ix,iy-1,b3)]*wd[fencode_ds(p,ix,iy-1,current1)]-w[fencode_ds(p,ix,iy-1,b1)]*wd[fencode_ds(p,ix,iy-1,current3)]);
     //  fip2=(w[fencode_ds(p,ix,iy+2,b3)]*wd[fencode_ds(p,ix,iy+2,current1)]-w[fencode_ds(p,ix,iy+2,b1)]*wd[fencode_ds(p,ix,iy+2,current3)]);
     //  fim2=(w[fencode_ds(p,ix,iy-2,b3)]*wd[fencode_ds(p,ix,iy-2,current1)]-w[fencode_ds(p,ix,iy-2,b1)]*wd[fencode_ds(p,ix,iy-2,current3)]);
      // ddcx=evalgrad_ds(fi,fim1,fip2,fim2,p,0);
      ddcy=evalgrad_ds(fi,fim1,0,0,p,1);

      srcb=(isnan(ddcx)?0:ddcx)+(isnan(ddcy)?0:ddcy);

 // src=srcg+srcb;
  return ( srcg+srcb);
}


__device__ __host__
int derivsourcerho (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=rho;
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourcerho(dw,wd,w,p,ix,iy);
     	//dw[fencode_ds(p,ix,iy,field)]=w[fencode_ds(p,ix,iy,field)]+10;
  return ( status);
}

__device__ __host__
int derivsourcemom (real *dw, real *wd, real *w, struct params *p,int ix, int iy,int field, int direction) {

  int status=0;
     	//dw[fencode_ds(p,ix,iy,field)]=w[fencode_ds(p,ix,iy,field)]+20+5*(2*direction+1);
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourcemom(dw,wd,w,p,ix,iy,field,direction);
        //dw[fencode_ds(p,ix,iy,field)]=-ddotcurrentmom(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivsourceb (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field, int direction) {

  int status=0;
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourceb(dw,wd,w,p,ix,iy,field,direction);

  return ( status);
}

__device__ __host__
int derivsourceenergy (real *dw, real *wd, real *w, struct params *p,int ix, int iy) {

  int status=0;
  int field=energy;
        dw[fencode_ds(p,ix,iy,field)]=dw[fencode_ds(p,ix,iy,field)]+sourceenergy(dw,wd,w,p,ix,iy);

  return ( status);
}


//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void derivsource (real *dw, real *wd, real *w, struct params *p,int ix, int iy, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      derivsourcerho(dw,wd,w,p,ix,iy);
     break;
     case mom1:
      derivsourcemom(dw,wd,w,p,ix,iy,field,0);
     break;
     case mom2:
      derivsourcemom(dw,wd,w,p,ix,iy,field,1);
     break;
     case mom3:
      derivsourcemom(dw,wd,w,p,ix,iy,field,2);
     break;
     case energy:
       derivsourceenergy(dw,wd,w,p,ix,iy);
     break;
     case b1:
      derivsourceb(dw,wd,w,p,ix,iy,field,0);
     break;
     case b2:
      derivsourceb(dw,wd,w,p,ix,iy,field,1);
     break;
     case b3:
      derivsourceb(dw,wd,w,p,ix,iy,field,2);
     break;
  }
  //return ( status);
}


__global__ void derivsource_parallel(struct params *p, real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->ni;
  int nj=p->nj;
  real dt=p->dt;
  real dy=p->dy;
  real dx=p->dx;
  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
  if(i>1 && j >1 && i<((p->ni)-2) && j<((p->nj)-2))
	{		               
               /*for(int f=rho; f<=b3; f++)               
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               computebdotv(wmod,wd,p,i,j);*/
               for(int f=rho; f<=b3; f++)
               {              
                  //if( (f==mom2) && (j==2))
                  //   ;//derivsource(dwn1,wd,wmod,p,i,j,f);
                  //else
                    derivsource(dwn1,wd,wmod,p,i,j,f);
                  //dwn1[fencode_ds(p,i,j,f)]=1.0;
                 // __syncthreads();
               }
               
               /*for(int f=rho; f<=b3; f++) 
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+0.5*dt*dwn1[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn2,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+0.5*dt*dwn2[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn3,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  wmod[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+dt*dwn3[fencode_ds(p,i,j,f)];
               computej(wmod,wd,p,i,j);
               computepk(wmod,wd,p,i,j);
               computept(wmod,wd,p,i,j);
               for(int f=rho; f<=b3; f++) 
                  deriv(dwn4,wd,wmod,p,i,j,f);
               
               for(int f=rho; f<=b3; f++) 
                  {
                  wnew[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+(dt/6.0)*(
                     dwn1[fencode_ds(p,i,j,f)]+2.0*dwn2[fencode_ds(p,i,j,f)]
                         +2.0*dwn3[fencode_ds(p,i,j,f)]+dwn4[fencode_ds(p,i,j,f)]);
               }*/
              //  __syncthreads();
              /* for(int f=rho; f<=b3; f++)
                   wnew[fencode_ds(p,i,j,f)]=w[fencode_ds(p,i,j,f)]+dt*dwn1[fencode_ds(p,i,j,f)];
               computej(wnew,wd,p,i,j);
               computepk(wnew,wd,p,i,j);
               computept(wnew,wd,p,i,j);*/ 


	}
 __syncthreads();
  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_ds(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuderivsource(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew,  real **d_wmod, real **d_dwn1, real **d_wd)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
    dim3 dimGrid(((*p)->ni)/dimBlock.x,((*p)->nj)/dimBlock.y);
   int numBlocks = (((*p)->ni)*((*p)->nj)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     derivsource_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
// hipMemcpy(*w, *d_w, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->ni)* ((*p)->nj)*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->ni)* ((*p)->nj))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}







